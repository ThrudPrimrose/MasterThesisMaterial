#include "hip/hip_runtime.h"

#include <random>
#include <iostream>
#include <cstring>
#include <vector>
#include <unordered_map>

#include <hiptensor.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR(x)                                                  \
{                                                                        \
  const auto err = x;                                                    \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                   \
  {                                                                      \
    std::cout << "Error: " << hiptensorGetErrorString(err) << std::endl;  \
    std::cout << __FILE__ << " " << __LINE__ << std::endl;                      \
  }                                                                      \
}

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess) {
        std::cout << std::endl << File
                << ", line " << Line
                << ": " << hipGetErrorString(Error)
                << " (" << Error << ")"
                << std::endl;

        if (PrevLine > 0)
        std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
    }
    PrevFile = File;
    PrevLine = Line;
#endif
}


__global__ void 
__launch_bounds__(480)
 product1(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      {
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_B = &B[batchID][0 + B_extraOffset];
        const float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float reg0[10] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[145];
        float * localShrMem0 = &totalShrMem[145 * threadIdx.y];

        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 45) {
            shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
          }
        }

        float* shrRegion1 = &localShrMem0[45];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 100) {
            shrRegion1[threadIdx.x + 0] = glb_X[threadIdx.x + 0];
          }
        }
        __syncthreads();
        /*
        This is the product kernel created from the following YaTeTo description:
        Description(
        	alpha: 1.0
        	add: True
        	result: IndexedTensorDescription(name=A, indices=kpm, memoryLayout=DenseMemoryLayout(shape=(10, 10, 45), bbox=BoundingBox(Range(0, 10), Range(0, 10), Range(0, 45)), stride=(1, 10, 100), align=<yateto.arch.Architecture object at 0x7fbd1917ead0>), eqspp=dense(shape=(10, 10, 45), size=4500, ndim=3), is_compute_constant=False, is_temporary=False)
        	leftTerm: IndexedTensorDescription(name=B, indices=m, memoryLayout=DenseMemoryLayout(shape=(45,), bbox=BoundingBox(Range(0, 45)), stride=(1,), align=<yateto.arch.Architecture object at 0x7fbd1917ead0>), eqspp=dense(shape=(45,), size=45, ndim=1), is_compute_constant=False, is_temporary=False)
        	rightTerm: IndexedTensorDescription(name=X, indices=kp, memoryLayout=DenseMemoryLayout(shape=(10, 10), bbox=BoundingBox(Range(0, 10), Range(0, 10)), stride=(1, 10), align=<yateto.arch.Architecture object at 0x7fbd1917ead0>), eqspp=dense(shape=(10, 10), size=100, ndim=2), is_compute_constant=False, is_temporary=False)
        	isACsc: False
        	isBCsc: False
        	loopRanges: {'m': Range(0, 45), 'p': Range(0, 10), 'k': Range(0, 10)}
        )
        */
        if (threadIdx.x < 450) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 10;
          rows_left -= row_offset_1 * 10;
          const int dim_offset_m = row_offset_1;
          const int row_offset_0 = rows_left;
          const int dim_offset_k = row_offset_0;
          #pragma unroll
          for (int p = 0; p < 10; ++p) {
            reg0[p] = shrRegion0[dim_offset_m * 1] * shrRegion1[dim_offset_k * 1 + p * 10];
          }
        }
        if (threadIdx.x < 450) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 10;
          rows_left -= row_offset_1 * 10;
          const int row_offset_0 = rows_left;
          #pragma unroll
          for (int i = 0; i < 10; ++i) {
            glb_A[row_offset_0 * 1 + row_offset_1 * 100 + i * 10] = reg0[i] + 1.0 * glb_A[row_offset_0 * 1 + row_offset_1 * 100 + i * 10];
          }
        }
      }
    }
  }
}
void product_launcher1(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(480, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  product1<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}



__global__ void 
__launch_bounds__(480)
 product2(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      {
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_B = &B[batchID][0 + B_extraOffset];
        const float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float reg0[10] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[145+4500];
        float * localShrMem0 = &totalShrMem[(145+4500) * threadIdx.y];

        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 45) {
            shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
          }
        }

        float* shrRegion1 = &localShrMem0[45];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 100) {
            shrRegion1[threadIdx.x + 0] = glb_X[threadIdx.x + 0];
          }
        }

        float* shrRegion2 = &localShrMem0[145];
        // using ExtendedTensorLoader
        {
          for (int i = 0; i < 9; i++){
            shrRegion2[threadIdx.x + 480*i] = glb_A[threadIdx.x + 480*i];
          }
          if (threadIdx.x < 180) {
            shrRegion2[threadIdx.x + 480*9] = glb_A[threadIdx.x + 480*9];
          }
        }
        __syncthreads();
        /*
        This is the product kernel created from the following YaTeTo description:
        Description(
        	alpha: 1.0
        	add: True
        	result: IndexedTensorDescription(name=A, indices=kpm, memoryLayout=DenseMemoryLayout(shape=(10, 10, 45), bbox=BoundingBox(Range(0, 10), Range(0, 10), Range(0, 45)), stride=(1, 10, 100), align=<yateto.arch.Architecture object at 0x7fbd1917ead0>), eqspp=dense(shape=(10, 10, 45), size=4500, ndim=3), is_compute_constant=False, is_temporary=False)
        	leftTerm: IndexedTensorDescription(name=B, indices=m, memoryLayout=DenseMemoryLayout(shape=(45,), bbox=BoundingBox(Range(0, 45)), stride=(1,), align=<yateto.arch.Architecture object at 0x7fbd1917ead0>), eqspp=dense(shape=(45,), size=45, ndim=1), is_compute_constant=False, is_temporary=False)
        	rightTerm: IndexedTensorDescription(name=X, indices=kp, memoryLayout=DenseMemoryLayout(shape=(10, 10), bbox=BoundingBox(Range(0, 10), Range(0, 10)), stride=(1, 10), align=<yateto.arch.Architecture object at 0x7fbd1917ead0>), eqspp=dense(shape=(10, 10), size=100, ndim=2), is_compute_constant=False, is_temporary=False)
        	isACsc: False
        	isBCsc: False
        	loopRanges: {'m': Range(0, 45), 'p': Range(0, 10), 'k': Range(0, 10)}
        )
        */
        if (threadIdx.x < 450) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 10;
          rows_left -= row_offset_1 * 10;
          const int dim_offset_m = row_offset_1;
          const int row_offset_0 = rows_left;
          const int dim_offset_k = row_offset_0;
          #pragma unroll
          for (int p = 0; p < 10; ++p) {
            reg0[p] = shrRegion0[dim_offset_m * 1] * shrRegion1[dim_offset_k * 1 + p * 10];
          }

          #pragma unroll
          for (int i = 0; i < 10; ++i) {
            glb_A[row_offset_0 * 1 + row_offset_1 * 100 + i * 10] = reg0[i] + shrRegion2[row_offset_0 * 1 + row_offset_1 * 100 + i * 10];
          }
        }
      }
    }
  }
}
void product_launcher2(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(480, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  product2<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}


int main(){
  constexpr size_t num_els = 122554;
  float* A = new float[4784 * num_els]{0.f};
  float* B = new float[46 * num_els]{0.f};
  float* C = new float[120 * num_els]{0.f};
  float* D = new float[195 * num_els]{0.f};
  float* E = new float[1456 * num_els]{0.f};
  float* F = new float[644 * num_els]{0.f};
  float* X = new float[104 * num_els]{0.f};
  float* R1 = new float[4784 * num_els]{0.f};
  float* R2 = new float[4784 * num_els]{0.f};
  //float* Ri1 = new float[104 * num_els]{0.f};
  //float* Ri2 = new float[4784 * num_els]{0.f};
  //float* Ri1c = new float[104 * num_els]{0.f};
  //float* Ri2c = new float[4784 * num_els]{0.f};


  float* coreA = new float[4784];
  float* coreB = new float[46];
  float* coreC = new float[120];
  float* coreD = new float[195];
  float* coreE = new float[1456];
  float* coreF = new float[644];

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<int> distribution(1, 100);
  for (size_t i = 0; i < 4784; i++){
    coreA[i] = distribution(gen);
  }
  for (size_t i = 0; i < 46; i++){
    coreB[i] = distribution(gen);
  }
  for (size_t i = 0; i < 120; i++){
    coreC[i] = distribution(gen);
  }
  for (size_t i = 0; i < 195; i++){
    coreD[i] = distribution(gen);
  }
  for (size_t i = 0; i < 1456; i++){
    coreE[i] = distribution(gen);
  }
  for (size_t i = 0; i < 644; i++){
    coreF[i] = distribution(gen);
  }

  for (size_t i = 0; i < num_els; i++){
      std::memcpy(&A[i * 4784], &coreA[0], 4784 * sizeof(float));
      std::memcpy(&B[i * 46], &coreB[0], 46 * sizeof(float));
      std::memcpy(&C[i * 120], &coreC[0], 120 * sizeof(float));
      std::memcpy(&D[i * 195], &coreD[0], 195 * sizeof(float));
      std::memcpy(&E[i * 1456], &coreE[0], 1456 * sizeof(float));
      std::memcpy(&F[i * 644], &coreF[0], 644 * sizeof(float));
  }

  float* A_dev = nullptr;
  float* B_dev = nullptr;
  float* C_dev = nullptr;
  float* D_dev = nullptr;
  float* E_dev = nullptr;
  float* F_dev = nullptr;
  float* X_dev = nullptr;

  float** A_dev_begins = new float*[num_els];
  float** B_dev_begins = new float*[num_els];
  float** C_dev_begins = new float*[num_els];
  float** D_dev_begins = new float*[num_els];
  float** E_dev_begins = new float*[num_els];
  float** F_dev_begins = new float*[num_els];
  float** X_dev_begins = new float*[num_els];

  float** A_dev_begins_dev = nullptr;
  float** B_dev_begins_dev = nullptr;
  float** C_dev_begins_dev = nullptr;
  float** D_dev_begins_dev = nullptr;
  float** E_dev_begins_dev = nullptr;
  float** F_dev_begins_dev = nullptr;
  float** X_dev_begins_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 4784 * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev, sizeof(float) * 46 * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev, sizeof(float) * 120 * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev, sizeof(float) * 195 * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev, sizeof(float) * 1456 * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev, sizeof(float) * 644 * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev, sizeof(float) * 104 * num_els); CHECK_ERR;

  hipMalloc((void **)&A_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
 
  hipDeviceSynchronize(); CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev, (void *)B, sizeof(float) * 46 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev, (void *)C, sizeof(float) * 120 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev, (void *)D, sizeof(float) * 195 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev, (void *)E, sizeof(float) * 1456 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev, (void *)F, sizeof(float) * 644 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 104 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  for (size_t i = 0; i < num_els; i++){
    A_dev_begins[i] = A_dev + i * 4784;
    B_dev_begins[i] = B_dev + i * 46;
    C_dev_begins[i] = C_dev + i * 120;
    D_dev_begins[i] = D_dev + i * 195;
    E_dev_begins[i] = E_dev + i * 1456;
    F_dev_begins[i] = F_dev + i * 644;
    X_dev_begins[i] = X_dev + i * 104;
  }

  hipMemcpy((void *)A_dev_begins_dev, (void *)A_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev_begins_dev, (void *)B_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev_begins_dev, (void *)C_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev_begins_dev, (void *)D_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev_begins_dev, (void *)E_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev_begins_dev, (void *)F_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev_begins_dev, (void *)X_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  std::cout << "Dimensions: " << 8 << ", " << 14 << ", " << 46 << ", " << 13 << ", " << 15 << ", " << 14 << std::endl;

  float elapsedTimeT1 = 0.0;
  float elapsedTimeT2 = 0.0;
  float elapsedTimeT3 = 0.0; 
  float elapsedTimeT4 = 0.0;
  hipEvent_t startT1, stopT1;
  hipEvent_t startT2, stopT2;
  hipEvent_t startT3, stopT3;
  hipEvent_t startT4, stopT4;

  hipEventCreate(&startT3); CHECK_ERR;
  hipEventCreate(&stopT3); CHECK_ERR;
  hipEventRecord(startT3); CHECK_ERR;
  product_launcher1(A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT3); CHECK_ERR;
  hipEventSynchronize(stopT3); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT3, startT3, stopT3); CHECK_ERR;
  //double elapsedTime = elapsedTimeT1 + elapsedTimeT2 + elapsedTimeT3;
  hipDeviceSynchronize(); CHECK_ERR;
  
  //std::cout << "Gemmforge Tensor Contraction took: " << elapsedTime << " ms" << std::endl; 
  hipMemcpy(R1, A_dev, sizeof(float) * 4784 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;


  hipEventCreate(&startT2); CHECK_ERR;
  hipEventCreate(&stopT2); CHECK_ERR;
  hipEventRecord(startT2); CHECK_ERR;
  product_launcher1(A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT2); CHECK_ERR;
  hipEventSynchronize(stopT2); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT2, startT2, stopT2); CHECK_ERR;
  hipDeviceSynchronize(); CHECK_ERR;
  hipMemcpy(R1, A_dev, sizeof(float) * 4784 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;


  hipEventCreate(&startT4); CHECK_ERR;
  hipEventCreate(&stopT4); CHECK_ERR;
  hipEventRecord(startT4); CHECK_ERR;
  product_launcher2(A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT4); CHECK_ERR;
  hipEventSynchronize(stopT4); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT4, startT4, stopT4); CHECK_ERR;
  hipDeviceSynchronize(); CHECK_ERR;
  hipMemcpy(R2, A_dev, sizeof(float) * 4784 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;



  double fp_per_el = 156208;
  double ls_per_el = 48116;
  double fp_unfused_per_el = 156208;
  double ls_unfused_per_el = 87220;
  fp_per_el *= num_els;
  ls_per_el *= num_els;
  fp_unfused_per_el *= num_els;
  ls_unfused_per_el *= num_els;
  //std::cout << "Gemmforge Theoretical Fused Kernel GFLOPs/s: " << fp_per_el * 1e-6 / elapsedTime << std::endl;
  //std::cout << "Operational Theoretical Fused intensity: " << fp_per_el / ls_per_el << std::endl;
  //std::cout << "Gemmforge GFLOPs/s: " << fp_unfused_per_el * 1e-6 / elapsedTime << std::endl;
  //std::cout << "Operational intensity: " << fp_unfused_per_el / ls_unfused_per_el << std::endl;
  double peakFLOPGiven = 29767.7;
  double peakBandwidthGiven = 760.08;

  if (peakFLOPGiven > 0.1 && peakBandwidthGiven){
    double obtainable_unfused_peak_k3 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(14352) / static_cast<double>(38872)));
    std::cout << 100.0*(14352 * num_els * 1e-6 / elapsedTimeT3) / obtainable_unfused_peak_k3 << " % of roof w. respect to Kernel3 intensity achieved with Gemmforge" << std::endl;
    std::cout << 100.0*(14352 * num_els * 1e-6 / elapsedTimeT2) / obtainable_unfused_peak_k3 << " % of roof w. respect to Kernel3 (Optimization Idea 1) intensity achieved with Gemmforge" << std::endl;
    std::cout << 100.0*(14352 * num_els * 1e-6 / elapsedTimeT4) / obtainable_unfused_peak_k3 << " % of roof w. respect to Kernel3 (Optimization Idea 2) intensity achieved with Gemmforge" << std::endl;
  }

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 104 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  bool results_wrong = false;
  for (size_t i = 0; i < 4784 * num_els; i++){
    if (std::abs(R1[i] - R2[i]) > 5.0f) {
      std::cout << "Results do not match, problem first at offset " << i << " :_(" << std::endl;
      results_wrong = true;
      break;
    }
  }
  if (!results_wrong){
    std::cout << "Gemmforge and Gemmforge Optimized contraction results match! :)" << std::endl;
  }

  hipFree(A_dev_begins_dev);
  hipFree(B_dev_begins_dev);
  hipFree(C_dev_begins_dev);
  hipFree(D_dev_begins_dev);
  hipFree(E_dev_begins_dev);
  hipFree(F_dev_begins_dev);
  hipFree(X_dev_begins_dev);

  delete[] A;
  delete[] B;
  delete[] C;
  delete[] D;
  delete[] E;
  delete[] F;
  delete[] X;
  delete[] A_dev_begins;
  delete[] B_dev_begins;
  delete[] C_dev_begins;
  delete[] D_dev_begins;
  delete[] E_dev_begins;
  delete[] F_dev_begins;
  delete[] X_dev_begins;
  delete[] R1;
  delete[] R2;

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  hipFree(D_dev);
  hipFree(E_dev);
  hipFree(F_dev);
  hipFree(X_dev);

  delete[] coreA;
  delete[] coreB;
  delete[] coreC;
  delete[] coreD;
  delete[] coreE;
  delete[] coreF;

  return 0;
}

