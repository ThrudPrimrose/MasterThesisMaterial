#include "hip/hip_runtime.h"

#include <random>
#include <iostream>
#include <cstring>
#include <vector>
#include <unordered_map>

#include <hiptensor.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR(x)                                                  \
{                                                                        \
  const auto err = x;                                                    \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                   \
  {                                                                      \
    std::cout << "Error: " << hiptensorGetErrorString(err) << std::endl;  \
    std::cout << __FILE__ << " " << __LINE__ << std::endl;                      \
  }                                                                      \
}

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess) {
        std::cout << std::endl << File
                << ", line " << Line
                << ": " << hipGetErrorString(Error)
                << " (" << Error << ")"
                << std::endl;

        if (PrevLine > 0)
        std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
    }
    PrevFile = File;
    PrevLine = Line;
#endif
}

__global__ void 
__launch_bounds__(32)
 kernel_sloopOverGEMM_NT_NT_NT__d89_26_d9_89_d9_26__alpha_1_0_beta_0_0_p_p_p__ef07740(const float * const * C, int C_extraOffset, const float * const * D, int D_extraOffset, float ** X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=X,	  memoryLayout=DenseMemoryLayout(shape=(9, 26), bbox=BoundingBox(Range(0, 9), Range(0, 26)), stride=(1, 9), align=<yateto.arch.Architecture object at 0x7f0cf1523ad0>),	  eqspp=dense(shape=(9, 26), size=234, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=C,	  memoryLayout=DenseMemoryLayout(shape=(9, 89), bbox=BoundingBox(Range(0, 9), Range(0, 89)), stride=(1, 9), align=<yateto.arch.Architecture object at 0x7f0cf1523ad0>),	  eqspp=dense(shape=(9, 89), size=801, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=D,	  memoryLayout=DenseMemoryLayout(shape=(89, 26), bbox=BoundingBox(Range(0, 89), Range(0, 26)), stride=(1, 89), align=<yateto.arch.Architecture object at 0x7f0cf1523ad0>),	  eqspp=dense(shape=(89, 26), size=2314, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=0.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 9), Range(0, 26), Range(0, 89))), 'matrix_a': DenseMatrix{name = C, num. rows = 9, num. columns = 89, leading dimension = 9, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 9, 89]}, 'matrix_b': DenseMatrix{name = D, num. rows = 89, num. columns = 26, leading dimension = 89, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 89, 26]}, 'matrix_c': DenseMatrix{name = X, num. rows = 9, num. columns = 26, leading dimension = 9, direction = DataFlowDirection.SINK, bbox = [0, 0, 9, 26]}, 'args': ['C, extraOffset_C', 'D, extraOffset_D', 'X, extraOffset_X', 'numElements', 'flags', 'streamPtr']})
      */
      {
    //('gemm', {'descr': Description(  result=TensorDescription(  name=X,	  memoryLayout=DenseMemoryLayout(shape=(9, 26), bbox=BoundingBox(Range(0, 9), Range(0, 26)), stride=(1, 9), align=<yateto.arch.Architecture object at 0x7f0cf1523ad0>),	  eqspp=dense(shape=(9, 26), size=234, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=C,	  memoryLayout=DenseMemoryLayout(shape=(9, 89), bbox=BoundingBox(Range(0, 9), Range(0, 89)), stride=(1, 9), align=<yateto.arch.Architecture object at 0x7f0cf1523ad0>),	  eqspp=dense(shape=(9, 89), size=801, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=D,	  memoryLayout=DenseMemoryLayout(shape=(89, 26), bbox=BoundingBox(Range(0, 89), Range(0, 26)), stride=(1, 89), align=<yateto.arch.Architecture object at 0x7f0cf1523ad0>),	  eqspp=dense(shape=(89, 26), size=2314, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=0.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 9), Range(0, 26), Range(0, 89))), 'matrix_a': DenseMatrix{name = C, num. rows = 9, num. columns = 89, leading dimension = 9, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 9, 89]}, 'matrix_b': DenseMatrix{name = D, num. rows = 89, num. columns = 26, leading dimension = 89, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 89, 26]}, 'matrix_c': DenseMatrix{name = X, num. rows = 9, num. columns = 26, leading dimension = 9, direction = DataFlowDirection.SINK, bbox = [0, 0, 9, 26]}, 'args': ['C, extraOffset_C', 'D, extraOffset_D', 'X, extraOffset_X', 'numElements', 'flags', 'streamPtr']})
        const float * const __restrict__ glb_D = &D[batchID][0 + D_extraOffset];
        const float * const __restrict__ glb_C = &C[batchID][0 + C_extraOffset];
        float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float reg0[9] = {0.0f};
        __shared__  __align__(8) float totalShrMem[3115];
        float * localShrMem0 = &totalShrMem[3115 * threadIdx.y];
        __shared__ __align__(8) float shrC[26*9];
        
        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 25; ++i) {
            shrRegion0[threadIdx.x + i * 32] = glb_C[threadIdx.x + i * 32];
          }
          if (threadIdx.x < 1) {
            shrRegion0[threadIdx.x + 800] = glb_C[threadIdx.x + 800];
          }
        }
        
        float* shrRegion1 = &localShrMem0[801];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 72; ++i) {
            shrRegion1[threadIdx.x + i * 32] = glb_D[threadIdx.x + i * 32];
          }
          if (threadIdx.x < 10) {
            shrRegion1[threadIdx.x + 2304] = glb_D[threadIdx.x + 2304];
          }
        }
        __syncwarp();
        if (threadIdx.x < 26) {
          float value;
        
          #pragma unroll 45
          for (int k = 0; k < 89; ++k) {
            value = shrRegion1[threadIdx.x * 89 + k];
        
            #pragma unroll
            for (int m = 0; m < 9; ++m) {
              reg0[m] += value * shrRegion0[m + 9 * k];
            }
          }
        }
        if (threadIdx.x < 26) {
          #pragma unroll
          for (int m = 0; m < 9; ++m) {
            shrC[threadIdx.x * 9 + m] = reg0[m];
          }
        }
        __syncwarp();

        #pragma unroll
        for (int i = 0; i < 7; ++i) {
          glb_X[threadIdx.x + i*32] = shrC[threadIdx.x + i*32];
        }
        if (threadIdx.x < 10) {
          glb_X[threadIdx.x + 7*32] = shrC[threadIdx.x + 7*32];
        }

        
      }
    }
  }
}
void sloopOverGEMM_NT_NT_NT__d89_26_d9_89_d9_26__alpha_1_0_beta_0_0_p_p_p__ef07740(const float * const * C, int C_extraOffset, const float * const * D, int D_extraOffset, float ** X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(32, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d89_26_d9_89_d9_26__alpha_1_0_beta_0_0_p_p_p__ef07740<<<grid,block,0,stream>>>(C, C_extraOffset, D, D_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}


__global__ void 
__launch_bounds__(256)
 kernel_sloopOverGEMM_NT_NT_NT__d234_17_d234_46_d46_17__alpha_1_0_beta_1_0_p_p_p__4e8177d(float ** A, int A_extraOffset, const float * const * E, int E_extraOffset, const float * const * F, int F_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=A,	  memoryLayout=DenseMemoryLayout(shape=(234, 17), bbox=BoundingBox(Range(0, 234), Range(0, 17)), stride=(1, 234), align=<yateto.arch.Architecture object at 0x7f0cf155e710>),	  eqspp=dense(shape=(234, 17), size=3978, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=E,	  memoryLayout=DenseMemoryLayout(shape=(234, 46), bbox=BoundingBox(Range(0, 234), Range(0, 46)), stride=(1, 234), align=<yateto.arch.Architecture object at 0x7f0cf155e710>),	  eqspp=dense(shape=(234, 46), size=10764, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=F,	  memoryLayout=DenseMemoryLayout(shape=(46, 17), bbox=BoundingBox(Range(0, 46), Range(0, 17)), stride=(1, 46), align=<yateto.arch.Architecture object at 0x7f0cf155e710>),	  eqspp=dense(shape=(46, 17), size=782, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=1.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 234), Range(0, 17), Range(0, 46))), 'matrix_a': DenseMatrix{name = E, num. rows = 234, num. columns = 46, leading dimension = 234, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 234, 46]}, 'matrix_b': DenseMatrix{name = F, num. rows = 46, num. columns = 17, leading dimension = 46, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 46, 17]}, 'matrix_c': DenseMatrix{name = A, num. rows = 234, num. columns = 17, leading dimension = 234, direction = DataFlowDirection.SINK, bbox = [0, 0, 234, 17]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F', 'A, extraOffset_A', 'numElements', 'flags', 'streamPtr']})
      */
      {
    //('gemm', {'descr': Description(  result=TensorDescription(  name=A,	  memoryLayout=DenseMemoryLayout(shape=(234, 17), bbox=BoundingBox(Range(0, 234), Range(0, 17)), stride=(1, 234), align=<yateto.arch.Architecture object at 0x7f0cf155e710>),	  eqspp=dense(shape=(234, 17), size=3978, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=E,	  memoryLayout=DenseMemoryLayout(shape=(234, 46), bbox=BoundingBox(Range(0, 234), Range(0, 46)), stride=(1, 234), align=<yateto.arch.Architecture object at 0x7f0cf155e710>),	  eqspp=dense(shape=(234, 46), size=10764, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=F,	  memoryLayout=DenseMemoryLayout(shape=(46, 17), bbox=BoundingBox(Range(0, 46), Range(0, 17)), stride=(1, 46), align=<yateto.arch.Architecture object at 0x7f0cf155e710>),	  eqspp=dense(shape=(46, 17), size=782, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=1.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 234), Range(0, 17), Range(0, 46))), 'matrix_a': DenseMatrix{name = E, num. rows = 234, num. columns = 46, leading dimension = 234, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 234, 46]}, 'matrix_b': DenseMatrix{name = F, num. rows = 46, num. columns = 17, leading dimension = 46, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 46, 17]}, 'matrix_c': DenseMatrix{name = A, num. rows = 234, num. columns = 17, leading dimension = 234, direction = DataFlowDirection.SINK, bbox = [0, 0, 234, 17]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F', 'A, extraOffset_A', 'numElements', 'flags', 'streamPtr']})
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_E = &E[batchID][0 + E_extraOffset];
        const float * const __restrict__ glb_F = &F[batchID][0 + F_extraOffset];
        float reg0[17] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[11546];
        float * localShrMem0 = &totalShrMem[11546 * threadIdx.y];
        
        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 42; ++i) {
            shrRegion0[threadIdx.x + i * 256] = glb_E[threadIdx.x + i * 256];
          }
          if (threadIdx.x < 12) {
            shrRegion0[threadIdx.x + 10752] = glb_E[threadIdx.x + 10752];
          }
        }
        
        float* shrRegion1 = &localShrMem0[10764];
        // using ExtendedPatchLoader
        {
          shrRegion1[threadIdx.x + 0] = glb_F[threadIdx.x + 0];
          shrRegion1[threadIdx.x + 256] = glb_F[threadIdx.x + 256];
          shrRegion1[threadIdx.x + 512] = glb_F[threadIdx.x + 512];
          if (threadIdx.x < 14) {
            shrRegion1[threadIdx.x + 768] = glb_F[threadIdx.x + 768];
          }
        }
        __syncthreads();
        if (threadIdx.x < 234) {
          float value;
        
          #pragma unroll
          for (int k = 0; k < 46; ++k) {
            value = shrRegion0[threadIdx.x + k * 234];
        
            #pragma unroll
            for (int n = 0; n < 17; ++n) {
              reg0[n] += value * shrRegion1[k + 46 * n];
            }
          }
        }
        if (threadIdx.x < 234) {
          #pragma unroll
          for (int n = 0; n < 17; ++n) {
            glb_A[threadIdx.x + 234 * n] = reg0[n] + glb_A[threadIdx.x + 234 * n];
          }
        }
        
      }
    }
  }
}
void sloopOverGEMM_NT_NT_NT__d234_17_d234_46_d46_17__alpha_1_0_beta_1_0_p_p_p__4e8177d(float ** A, int A_extraOffset, const float * const * E, int E_extraOffset, const float * const * F, int F_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(256, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d234_17_d234_46_d46_17__alpha_1_0_beta_1_0_p_p_p__4e8177d<<<grid,block,0,stream>>>(A, A_extraOffset, E, E_extraOffset, F, F_extraOffset, numElements, flags);
  CHECK_ERR;
}


__global__ void 
__launch_bounds__(160)
 kernel_sproduct_NT_NT_NT__d9_26_d9_26_17_d17__alpha_1_0_p_p_p__d554f24(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      {
        const float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_B = &B[batchID][0 + B_extraOffset];
        float reg0[26] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[251];
        float * localShrMem0 = &totalShrMem[251 * threadIdx.y];

        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 17) {
            shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
          }
        }

        float* shrRegion1 = &localShrMem0[17];
        // using ExtendedTensorLoader
        {
          shrRegion1[threadIdx.x + 0] = glb_X[threadIdx.x + 0];
          if (threadIdx.x < 74) {
            shrRegion1[threadIdx.x + 160] = glb_X[threadIdx.x + 160];
          }
        }
        __syncthreads();
        /*
        This is the product kernel created from the following YaTeTo description:
        Description(
        	alpha: 1.0
        	add: True
        	result: IndexedTensorDescription(name=A, indices=kpm, memoryLayout=DenseMemoryLayout(shape=(9, 26, 17), bbox=BoundingBox(Range(0, 9), Range(0, 26), Range(0, 17)), stride=(1, 9, 234), align=<yateto.arch.Architecture object at 0x7f0cf4133ad0>), eqspp=dense(shape=(9, 26, 17), size=3978, ndim=3), is_compute_constant=False, is_temporary=False)
        	leftTerm: IndexedTensorDescription(name=B, indices=m, memoryLayout=DenseMemoryLayout(shape=(17,), bbox=BoundingBox(Range(0, 17)), stride=(1,), align=<yateto.arch.Architecture object at 0x7f0cf4133ad0>), eqspp=dense(shape=(17,), size=17, ndim=1), is_compute_constant=False, is_temporary=False)
        	rightTerm: IndexedTensorDescription(name=X, indices=kp, memoryLayout=DenseMemoryLayout(shape=(9, 26), bbox=BoundingBox(Range(0, 9), Range(0, 26)), stride=(1, 9), align=<yateto.arch.Architecture object at 0x7f0cf4133ad0>), eqspp=dense(shape=(9, 26), size=234, ndim=2), is_compute_constant=False, is_temporary=False)
        	isACsc: False
        	isBCsc: False
        	loopRanges: {'m': Range(0, 17), 'k': Range(0, 9), 'p': Range(0, 26)}
        )
        */
        if (threadIdx.x < 153) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 9;
          rows_left -= row_offset_1 * 9;
          const int dim_offset_m = row_offset_1;
          const int row_offset_0 = rows_left;
          const int dim_offset_k = row_offset_0;
          #pragma unroll
          for (int p = 0; p < 26; ++p) {
            reg0[p] = shrRegion0[dim_offset_m * 1] * shrRegion1[dim_offset_k * 1 + p * 9];
          }
        }
        if (threadIdx.x < 153) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 9;
          rows_left -= row_offset_1 * 9;
          const int row_offset_0 = rows_left;
          #pragma unroll
          for (int i = 0; i < 26; ++i) {
            glb_A[row_offset_0 * 1 + row_offset_1 * 234 + i * 9] = reg0[i] + 1.0 * glb_A[row_offset_0 * 1 + row_offset_1 * 234 + i * 9];
          }
        }
      }
    }
  }
}
void sproduct_NT_NT_NT__d9_26_d9_26_17_d17__alpha_1_0_p_p_p__d554f24(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(160, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sproduct_NT_NT_NT__d9_26_d9_26_17_d17__alpha_1_0_p_p_p__d554f24<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}



int main(){
  constexpr size_t num_els = 47706;
  float* A = new float[3978 * num_els]{0.f};
  float* B = new float[17 * num_els]{0.f};
  float* C = new float[801 * num_els]{0.f};
  float* D = new float[2314 * num_els]{0.f};
  float* E = new float[10764 * num_els]{0.f};
  float* F = new float[782 * num_els]{0.f};
  float* X = new float[234 * num_els]{0.f};
  float* R1 = new float[3978 * num_els]{0.f};
  float* R2 = new float[3978 * num_els]{0.f};
  //float* Ri1 = new float[234 * num_els]{0.f};
  //float* Ri2 = new float[3978 * num_els]{0.f};
  //float* Ri1c = new float[234 * num_els]{0.f};
  //float* Ri2c = new float[3978 * num_els]{0.f};


  float* coreA = new float[3978];
  float* coreB = new float[17];
  float* coreC = new float[801];
  float* coreD = new float[2314];
  float* coreE = new float[10764];
  float* coreF = new float[782];

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<int> distribution(1, 100);
  for (size_t i = 0; i < 3978; i++){
    coreA[i] = distribution(gen);
  }
  for (size_t i = 0; i < 17; i++){
    coreB[i] = distribution(gen);
  }
  for (size_t i = 0; i < 801; i++){
    coreC[i] = distribution(gen);
  }
  for (size_t i = 0; i < 2314; i++){
    coreD[i] = distribution(gen);
  }
  for (size_t i = 0; i < 10764; i++){
    coreE[i] = distribution(gen);
  }
  for (size_t i = 0; i < 782; i++){
    coreF[i] = distribution(gen);
  }

  for (size_t i = 0; i < num_els; i++){
      std::memcpy(&A[i * 3978], &coreA[0], 3978 * sizeof(float));
      std::memcpy(&B[i * 17], &coreB[0], 17 * sizeof(float));
      std::memcpy(&C[i * 801], &coreC[0], 801 * sizeof(float));
      std::memcpy(&D[i * 2314], &coreD[0], 2314 * sizeof(float));
      std::memcpy(&E[i * 10764], &coreE[0], 10764 * sizeof(float));
      std::memcpy(&F[i * 782], &coreF[0], 782 * sizeof(float));
  }

  float* A_dev = nullptr;
  float* B_dev = nullptr;
  float* C_dev = nullptr;
  float* D_dev = nullptr;
  float* E_dev = nullptr;
  float* F_dev = nullptr;
  float* X_dev = nullptr;

  float** A_dev_begins = new float*[num_els];
  float** B_dev_begins = new float*[num_els];
  float** C_dev_begins = new float*[num_els];
  float** D_dev_begins = new float*[num_els];
  float** E_dev_begins = new float*[num_els];
  float** F_dev_begins = new float*[num_els];
  float** X_dev_begins = new float*[num_els];

  float** A_dev_begins_dev = nullptr;
  float** B_dev_begins_dev = nullptr;
  float** C_dev_begins_dev = nullptr;
  float** D_dev_begins_dev = nullptr;
  float** E_dev_begins_dev = nullptr;
  float** F_dev_begins_dev = nullptr;
  float** X_dev_begins_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 3978 * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev, sizeof(float) * 17 * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev, sizeof(float) * 801 * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev, sizeof(float) * 2314 * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev, sizeof(float) * 10764 * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev, sizeof(float) * 782 * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev, sizeof(float) * 234 * num_els); CHECK_ERR;

  hipMalloc((void **)&A_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
 
  hipDeviceSynchronize(); CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 3978 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev, (void *)B, sizeof(float) * 17 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev, (void *)C, sizeof(float) * 801 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev, (void *)D, sizeof(float) * 2314 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev, (void *)E, sizeof(float) * 10764 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev, (void *)F, sizeof(float) * 782 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 234 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  for (size_t i = 0; i < num_els; i++){
    A_dev_begins[i] = A_dev + i * 3978;
    B_dev_begins[i] = B_dev + i * 17;
    C_dev_begins[i] = C_dev + i * 801;
    D_dev_begins[i] = D_dev + i * 2314;
    E_dev_begins[i] = E_dev + i * 10764;
    F_dev_begins[i] = F_dev + i * 782;
    X_dev_begins[i] = X_dev + i * 234;
  }

  hipMemcpy((void *)A_dev_begins_dev, (void *)A_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev_begins_dev, (void *)B_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev_begins_dev, (void *)C_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev_begins_dev, (void *)D_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev_begins_dev, (void *)E_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev_begins_dev, (void *)F_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev_begins_dev, (void *)X_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  sloopOverGEMM_NT_NT_NT__d89_26_d9_89_d9_26__alpha_1_0_beta_0_0_p_p_p__ef07740(C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipDeviceSynchronize(); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 234 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  std::cout << "Dimensions: " << 9 << ", " << 46 << ", " << 17 << ", " << 26 << ", " << 89 << std::endl;

  float elapsedTimeT1 = 0.0;
  float elapsedTimeT2 = 0.0;
  float elapsedTimeT3 = 0.0; 
  hipEvent_t startT1, stopT1;
  hipEvent_t startT2, stopT2;
  hipEvent_t startT3, stopT3;
  hipEventCreate(&startT1); CHECK_ERR;
  hipEventCreate(&stopT1); CHECK_ERR;
  hipEventRecord(startT1); CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d89_26_d9_89_d9_26__alpha_1_0_beta_0_0_p_p_p__ef07740(C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT1); CHECK_ERR;
  hipEventSynchronize(stopT1); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT1, startT1, stopT1); CHECK_ERR;
  //hipDeviceSynchronize(); CHECK_ERR;

  //hipMemcpy(Ri1, X_dev, sizeof(float) * 234 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  
  hipEventCreate(&startT2); CHECK_ERR;
  hipEventCreate(&stopT2); CHECK_ERR;
  hipEventRecord(startT2); CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d234_17_d234_46_d46_17__alpha_1_0_beta_1_0_p_p_p__4e8177d(A_dev_begins_dev, 0, E_dev_begins_dev, 0, F_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT2); CHECK_ERR;
  hipEventSynchronize(stopT2); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT2, startT2, stopT2); CHECK_ERR;
  //hipDeviceSynchronize(); CHECK_ERR;

  //hipMemcpy(Ri2, A_dev, sizeof(float) * 3978 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

  hipEventCreate(&startT3); CHECK_ERR;
  hipEventCreate(&stopT3); CHECK_ERR;
  hipEventRecord(startT3); CHECK_ERR;
  sproduct_NT_NT_NT__d9_26_d9_26_17_d17__alpha_1_0_p_p_p__d554f24(A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT3); CHECK_ERR;
  hipEventSynchronize(stopT3); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT3, startT3, stopT3); CHECK_ERR;
  double elapsedTime = elapsedTimeT1 + elapsedTimeT2 + elapsedTimeT3;
  hipDeviceSynchronize(); CHECK_ERR;
  
  std::cout << "Gemmforge Tensor Contraction took: " << elapsedTime << " ms" << std::endl; 
  hipMemcpy(R1, A_dev, sizeof(float) * 3978 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 3978 * num_els, hipMemcpyHostToDevice); CHECK_ERR;


  double fp_per_el = 419562;
  double ls_per_el = 90536;
  double fp_unfused_per_el = 419562;
  double ls_unfused_per_el = 124232;
  fp_per_el *= num_els;
  ls_per_el *= num_els;
  fp_unfused_per_el *= num_els;
  ls_unfused_per_el *= num_els;
  std::cout << "Gemmforge Theoretical Fused Kernel GFLOPs/s: " << fp_per_el * 1e-6 / elapsedTime << std::endl;
  std::cout << "Operational Theoretical Fused intensity: " << fp_per_el / ls_per_el << std::endl;
  std::cout << "Gemmforge GFLOPs/s: " << fp_unfused_per_el * 1e-6 / elapsedTime << std::endl;
  std::cout << "Operational intensity: " << fp_unfused_per_el / ls_unfused_per_el << std::endl;
  double peakFLOPGiven = 29767.7;
  double peakBandwidthGiven = 760.08;

  if (peakFLOPGiven > 0.1 && peakBandwidthGiven){
    double obtainable_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_per_el) / static_cast<double>(ls_per_el)));
    std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTime) / obtainable_peak << " % of roof w. respect to operational intensity achieved with Gemmforge" << std::endl;
    //std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTime) / obtainable_peak << " % of roof w. respect to operational intensity achieved with cuTensor" << std::endl;
    double obtainable_unfused_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_unfused_per_el) / static_cast<double>(ls_unfused_per_el)));
    std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTime) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with Gemmforge" << std::endl;
    //std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTime) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with cuTensor" << std::endl;
    double obtainable_unfused_peak_k1 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(41652) / static_cast<double>(13396)));
    std::cout << 100.0*(41652 * num_els  * 1e-6 / elapsedTimeT1) / obtainable_unfused_peak_k1 << " % of roof w. respect to Kernel1 intensity achieved with Gemmforge" << std::endl;
    double obtainable_unfused_peak_k2 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(369954) / static_cast<double>(78008)));
    std::cout << 100.0*(369954 * num_els  * 1e-6 / elapsedTimeT2) / obtainable_unfused_peak_k2 << " % of roof w. respect to Kernel2 intensity achieved with Gemmforge" << std::endl;
    double obtainable_unfused_peak_k3 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(7956) / static_cast<double>(32828)));
    std::cout << 100.0*(7956 * num_els * 1e-6 / elapsedTimeT3) / obtainable_unfused_peak_k3 << " % of roof w. respect to Kernel3 intensity achieved with Gemmforge" << std::endl;
  }

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 3978 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 234 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  if constexpr (!false){
  hiptensorHandle_t* handle;
  HANDLE_ERROR(hiptensorCreate(&handle));

  hipEvent_t startCT1, stopCT1;
  hipEvent_t startCT2, stopCT2;
  hipEvent_t startCT3, stopCT3;
  hipEventCreate(&startCT1); CHECK_ERR;
  hipEventCreate(&stopCT1); CHECK_ERR;
  hipEventCreate(&startCT2); CHECK_ERR;
  hipEventCreate(&stopCT2); CHECK_ERR;
  hipEventCreate(&startCT3); CHECK_ERR;
  hipEventCreate(&stopCT3); CHECK_ERR;
  float elapsedTimeCT1 = 0.f;
  float elapsedTimeCT2 = 0.f;
  float elapsedTimeCT3 = 0.f;

  // Kernel 1
  std::cout << "cuTensor Kernel 1" << std::endl;
  {
    float alphaK1 = 1.0f;
    float betaK1 = 0.0f;
    float alphaK2 = 1.0f;
    float betaK2 = 1.0;
    float alphaK3 = 1.0f;
    float betaK3 = 1.0;

    std::vector<int> modeA{'k', 'p', 'm', 'b'};
    std::vector<int> modeB{'m', 'b'};
    std::vector<int> modeC{'k', 'q', 'b'};
    std::vector<int> modeD{'q', 'p', 'b'};
    std::vector<int> modeE{'k', 'p', 'l', 'b'};
    std::vector<int> modeF{'l', 'm', 'b'};
    std::vector<int> modeX{'k', 'p', 'b'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();
    int nmodeD = modeD.size();
    int nmodeE = modeE.size();
    int nmodeF = modeF.size();
    int nmodeX = modeX.size();

    std::unordered_map<int, int64_t> extent;
    // Derived from the kernel
    extent['k'] = 9;
    extent['l'] = 46;
    extent['m'] = 17;
    extent['p'] = 26;
    extent['q'] = 89;
    extent['b'] = num_els;

    std::vector<int64_t> extentA;
    for (auto mode : modeA) {
        extentA.push_back(extent[mode]);
    }
    std::vector<int64_t> extentB;
    for (auto mode : modeB) {
        extentB.push_back(extent[mode]);
    }
    std::vector<int64_t> extentC;
    for (auto mode : modeC) {
        extentC.push_back(extent[mode]);
    }
    std::vector<int64_t> extentD;
    for (auto mode : modeD) {
        extentD.push_back(extent[mode]);
    }
    std::vector<int64_t> extentE;
    for (auto mode : modeE) {
        extentE.push_back(extent[mode]);
    }
    std::vector<int64_t> extentF;
    for (auto mode : modeF) {
        extentF.push_back(extent[mode]);
    }
    std::vector<int64_t> extentX;
    for (auto mode : modeX) {
        extentX.push_back(extent[mode]);
    }
    
    hipDataType typeA = HIP_R_32F;
    hipDataType typeB = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hipDataType typeD = HIP_R_32F;
    hipDataType typeE = HIP_R_32F;
    hipDataType typeF = HIP_R_32F;
    hipDataType typeX = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descA,
                    nmodeA,
                    extentA.data(),
                    NULL,
                    typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descB,
                    nmodeB,
                    extentB.data(),
                    NULL,
                    typeB, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descC,
                    nmodeC,
                    extentC.data(),
                    NULL,
                    typeC, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descD;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descD,
                    nmodeD,
                    extentD.data(),
                    NULL,
                    typeD, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descE;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descE,
                    nmodeE,
                    extentE.data(),
                    NULL,
                    typeE, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descF;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descF,
                    nmodeF,
                    extentF.data(),
                    NULL,
                    typeF, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descX;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descX,
                    nmodeX,
                    extentX.data(),
                    NULL,
                    typeX, HIPTENSOR_OP_IDENTITY));


    uint32_t alignmentRequirementA;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    A_dev,
                    &descA,
                    &alignmentRequirementA));

    uint32_t alignmentRequirementB;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    B_dev,
                    &descB,
                    &alignmentRequirementB));

    uint32_t alignmentRequirementC;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    C_dev,
                    &descC, 
                    &alignmentRequirementC));

    uint32_t alignmentRequirementD;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    D_dev,
                    &descD,
                    &alignmentRequirementD));

    uint32_t alignmentRequirementE;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    E_dev,
                    &descE,
                    &alignmentRequirementE));

    uint32_t alignmentRequirementF;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    F_dev,
                    &descF, 
                    &alignmentRequirementF));

    uint32_t alignmentRequirementX;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    X_dev,
                    &descX, 
                    &alignmentRequirementX));

    cutensorContractionDescriptor_t desc1;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc1,
                  &descC, modeC.data(), alignmentRequirementC,
                  &descD, modeD.data(), alignmentRequirementD,
                  &descX, modeX.data(), alignmentRequirementX,
                  &descX, modeX.data(), alignmentRequirementX,
                  typeCompute));

    cutensorContractionFind_t find1;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find1, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize1 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc1,
                 &find1,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize1));

    cutensorContractionDescriptor_t desc2;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc2,
                  &descF, modeF.data(), alignmentRequirementF,
                  &descE, modeE.data(), alignmentRequirementE,
                  &descA, modeA.data(), alignmentRequirementA,
                  &descA, modeA.data(), alignmentRequirementA,
                  typeCompute));

    cutensorContractionFind_t find2;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find2, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize2 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc2,
                 &find2,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize2));


    cutensorContractionDescriptor_t desc3;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc3,
                  &descB, modeB.data(), alignmentRequirementB,
                  &descX, modeX.data(), alignmentRequirementX,
                  &descA, modeA.data(), alignmentRequirementA,
                  &descA, modeA.data(), alignmentRequirementA,
                  typeCompute));

    cutensorContractionFind_t find3;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find3, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize3 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc3,
                 &find3,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize3));

    uint64_t maxWorkSize = std::max(std::max(worksize1, worksize2), worksize3);
    void *work = nullptr;
    if (maxWorkSize > 0)
    {
        if (hipSuccess != hipMalloc(&work, maxWorkSize))
        {
            work = nullptr;
            maxWorkSize = 0;
            worksize1 = 0;
            worksize2 = 0;
            worksize3 = 0;
            hipGetLastError(); // Clear last error to save CHECK_ERR;
        } else {
            worksize1 = maxWorkSize;
            worksize2 = maxWorkSize;
            worksize3 = maxWorkSize;
        }
    }


    hiptensorContractionPlan_t plan1;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan1,
                 &desc1,
                 &find1,
                 worksize1));

    hiptensorContractionPlan_t plan2;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan2,
                 &desc2,
                 &find2,
                 worksize2));

    hiptensorContractionPlan_t plan3;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan3,
                 &desc3,
                 &find3,
                 worksize3));

    hipDeviceSynchronize(); CHECK_ERR;

    hipEventRecord(startCT1); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan1,
                              (void*) &alphaK1, C_dev, D_dev,
                              (void*) &betaK1,  X_dev, X_dev, 
                              work, worksize1, 0);
    hipEventRecord(stopCT1); CHECK_ERR;
    hipEventSynchronize(stopCT1); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT1, startCT1, stopCT1); CHECK_ERR;

    //hipDeviceSynchronize(); CHECK_ERR;
    //hipMemcpy(Ri1c, X_dev, sizeof(float) * 234 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipEventRecord(startCT2); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan2,
                              (void*) &alphaK2, F_dev, E_dev,
                              (void*) &betaK2,  A_dev, A_dev, 
                              work, worksize2, 0);
    hipEventRecord(stopCT2); CHECK_ERR;
    hipEventSynchronize(stopCT2); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT2, startCT2, stopCT2); CHECK_ERR;

    //hipDeviceSynchronize(); CHECK_ERR;
    //hipMemcpy(Ri2c, A_dev, sizeof(float) * 3978 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipEventRecord(startCT3); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan3,
                              (void*) &alphaK3, B_dev, X_dev,
                              (void*) &betaK3,  A_dev, A_dev, 
                              work, worksize3, 0);
    hipEventRecord(stopCT3); CHECK_ERR;
    hipEventSynchronize(stopCT3); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT3, startCT3, stopCT3); CHECK_ERR;

    hipDeviceSynchronize(); CHECK_ERR;
    
    hipMemcpy(R2, A_dev, sizeof(float) * 3978 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipFree(work);
  }

  float elapsedTimeCuTensor = elapsedTimeCT1 + elapsedTimeCT2 + elapsedTimeCT2;
  if (peakFLOPGiven > 0.1 && peakBandwidthGiven){
    double obtainable_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_per_el) / static_cast<double>(ls_per_el)));
    std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTimeCuTensor) / obtainable_peak << " % of roof w. respect to operational intensity achieved with cuTensor" << std::endl;

    double obtainable_unfused_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_unfused_per_el) / static_cast<double>(ls_unfused_per_el)));
    std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTimeCuTensor) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with cuTensor" << std::endl;
  }

  /*
  bool i1results_wrong = false;
  for (size_t i = 0; i < 234 * num_els; i++){
    if (std::abs(Ri1[i] - Ri1c[i]) > 1.0f) {
      std::cout << "Intermediate Results 1 do not match, problem first at offset " << i << " :_(" << std::endl;
      i1results_wrong = true;
      break;
    }
  }
  if (!i1results_wrong){
    std::cout << "Gemmforge and cuTensor contraction intermediate results 1 match! :)" << std::endl;
  }
  
  bool i2results_wrong = false;
  for (size_t i = 0; i < 3978 * num_els; i++){
    if (std::abs(Ri2[i] - Ri2c[i]) > 1.0f) {
      std::cout << "Intermediate Results 2 do not match, problem first at offset " << i << " :_(" << std::endl;
      i2results_wrong = true;
      break;
    }
  }
  if (!i2results_wrong){
    std::cout << "Gemmforge and cuTensor contraction intermediate results 2 match! :)" << std::endl;
  }
  */

  bool results_wrong = false;
  for (size_t i = 0; i < 3978 * num_els; i++){
    if (std::abs(R1[i] - R2[i]) > 5.0f) {
      std::cout << "Results do not match, problem first at offset " << i << " :_(" << std::endl;
      results_wrong = true;
      break;
    }
  }
  if (!results_wrong){
    std::cout << "Gemmforge and cuTensor contraction results match! :)" << std::endl;
  }
  }

  hipFree(A_dev_begins_dev);
  hipFree(B_dev_begins_dev);
  hipFree(C_dev_begins_dev);
  hipFree(D_dev_begins_dev);
  hipFree(E_dev_begins_dev);
  hipFree(F_dev_begins_dev);
  hipFree(X_dev_begins_dev);

  delete[] A;
  delete[] B;
  delete[] C;
  delete[] D;
  delete[] E;
  delete[] F;
  delete[] X;
  delete[] A_dev_begins;
  delete[] B_dev_begins;
  delete[] C_dev_begins;
  delete[] D_dev_begins;
  delete[] E_dev_begins;
  delete[] F_dev_begins;
  delete[] X_dev_begins;
  delete[] R1;
  delete[] R2;

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  hipFree(D_dev);
  hipFree(E_dev);
  hipFree(F_dev);
  hipFree(X_dev);

  delete[] coreA;
  delete[] coreB;
  delete[] coreC;
  delete[] coreD;
  delete[] coreE;
  delete[] coreF;

  return 0;
}

