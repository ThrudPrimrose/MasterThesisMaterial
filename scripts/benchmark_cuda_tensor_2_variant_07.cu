#include "hip/hip_runtime.h"

#include <random>
#include <iostream>
#include <cstring>
#include <vector>
#include <unordered_map>

#include <hiptensor.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR(x)                                                  \
{                                                                        \
  const auto err = x;                                                    \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                   \
  {                                                                      \
    std::cout << "Error: " << hiptensorGetErrorString(err) << std::endl;  \
    std::cout << __FILE__ << " " << __LINE__ << std::endl;                      \
  }                                                                      \
}

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess) {
        std::cout << std::endl << File
                << ", line " << Line
                << ": " << hipGetErrorString(Error)
                << " (" << Error << ")"
                << std::endl;

        if (PrevLine > 0)
        std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
    }
    PrevFile = File;
    PrevLine = Line;
#endif
}

__global__ void 
__launch_bounds__(32)
 kernel_sloopOverGEMM_NT_NT_NT__d10_32_d32_10_d10_10__alpha_1_0_beta_0_0_p_p_p__6a7ea95(const float * const * C, int C_extraOffset, const float * const * D, int D_extraOffset, float ** X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=X,	  memoryLayout=DenseMemoryLayout(shape=(10, 10), bbox=BoundingBox(Range(0, 10), Range(0, 10)), stride=(1, 10), align=<yateto.arch.Architecture object at 0x7f3bda32fb50>),	  eqspp=dense(shape=(10, 10), size=100, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=C,	  memoryLayout=DenseMemoryLayout(shape=(10, 32), bbox=BoundingBox(Range(0, 10), Range(0, 32)), stride=(1, 10), align=<yateto.arch.Architecture object at 0x7f3bda32fb50>),	  eqspp=dense(shape=(10, 32), size=320, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=D,	  memoryLayout=DenseMemoryLayout(shape=(32, 10), bbox=BoundingBox(Range(0, 32), Range(0, 10)), stride=(1, 32), align=<yateto.arch.Architecture object at 0x7f3bda32fb50>),	  eqspp=dense(shape=(32, 10), size=320, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=0.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 10), Range(0, 10), Range(0, 32))), 'matrix_a': DenseMatrix{name = C, num. rows = 10, num. columns = 32, leading dimension = 10, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 10, 32]}, 'matrix_b': DenseMatrix{name = D, num. rows = 32, num. columns = 10, leading dimension = 32, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 32, 10]}, 'matrix_c': DenseMatrix{name = X, num. rows = 10, num. columns = 10, leading dimension = 10, direction = DataFlowDirection.SINK, bbox = [0, 0, 10, 10]}, 'args': ['C, extraOffset_C', 'D, extraOffset_D', 'X, extraOffset_X', 'numElements', 'flags', 'streamPtr']})
      */
      {
    //('gemm', {'descr': Description(  result=TensorDescription(  name=X,	  memoryLayout=DenseMemoryLayout(shape=(10, 10), bbox=BoundingBox(Range(0, 10), Range(0, 10)), stride=(1, 10), align=<yateto.arch.Architecture object at 0x7f3bda32fb50>),	  eqspp=dense(shape=(10, 10), size=100, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=C,	  memoryLayout=DenseMemoryLayout(shape=(10, 32), bbox=BoundingBox(Range(0, 10), Range(0, 32)), stride=(1, 10), align=<yateto.arch.Architecture object at 0x7f3bda32fb50>),	  eqspp=dense(shape=(10, 32), size=320, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=D,	  memoryLayout=DenseMemoryLayout(shape=(32, 10), bbox=BoundingBox(Range(0, 32), Range(0, 10)), stride=(1, 32), align=<yateto.arch.Architecture object at 0x7f3bda32fb50>),	  eqspp=dense(shape=(32, 10), size=320, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=0.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 10), Range(0, 10), Range(0, 32))), 'matrix_a': DenseMatrix{name = C, num. rows = 10, num. columns = 32, leading dimension = 10, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 10, 32]}, 'matrix_b': DenseMatrix{name = D, num. rows = 32, num. columns = 10, leading dimension = 32, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 32, 10]}, 'matrix_c': DenseMatrix{name = X, num. rows = 10, num. columns = 10, leading dimension = 10, direction = DataFlowDirection.SINK, bbox = [0, 0, 10, 10]}, 'args': ['C, extraOffset_C', 'D, extraOffset_D', 'X, extraOffset_X', 'numElements', 'flags', 'streamPtr']})
        const float * const __restrict__ glb_C = &C[batchID][0 + C_extraOffset];
        const float * const __restrict__ glb_D = &D[batchID][0 + D_extraOffset];
        float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float reg0[10] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[640];
        float * localShrMem0 = &totalShrMem[640 * threadIdx.y];
        
        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 10; ++i) {
            shrRegion0[threadIdx.x + i * 32] = glb_C[threadIdx.x + i * 32];
          }
        }
        
        float* shrRegion1 = &localShrMem0[320];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 10; ++i) {
            shrRegion1[threadIdx.x + i * 32] = glb_D[threadIdx.x + i * 32];
          }
        }
        __syncwarp();
        if (threadIdx.x < 10) {
          float value;
        
          #pragma unroll
          for (int k = 0; k < 32; ++k) {
            value = shrRegion0[threadIdx.x + k * 10];
        
            #pragma unroll
            for (int n = 0; n < 10; ++n) {
              reg0[n] += value * shrRegion1[k + 32 * n];
            }
          }
        }
        if (threadIdx.x < 10) {
          #pragma unroll
          for (int n = 0; n < 10; ++n) {
            glb_X[threadIdx.x + 10 * n] = reg0[n];
          }
        }
        
      }
    }
  }
}
void sloopOverGEMM_NT_NT_NT__d10_32_d32_10_d10_10__alpha_1_0_beta_0_0_p_p_p__6a7ea95(const float * const * C, int C_extraOffset, const float * const * D, int D_extraOffset, float ** X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(32, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d10_32_d32_10_d10_10__alpha_1_0_beta_0_0_p_p_p__6a7ea95<<<grid,block,0,stream>>>(C, C_extraOffset, D, D_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}


__global__ void 
__launch_bounds__(128)
 kernel_sloopOverGEMM_NT_NT_NT__d17_45_d100_45_d100_17__alpha_1_0_beta_1_0_p_p_p__4972d7a(float ** A, int A_extraOffset, const float * const * E, int E_extraOffset, const float * const * F, int F_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=A,	  memoryLayout=DenseMemoryLayout(shape=(100, 45), bbox=BoundingBox(Range(0, 100), Range(0, 45)), stride=(1, 100), align=<yateto.arch.Architecture object at 0x7f3bda2fff90>),	  eqspp=dense(shape=(100, 45), size=4500, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=E,	  memoryLayout=DenseMemoryLayout(shape=(100, 17), bbox=BoundingBox(Range(0, 100), Range(0, 17)), stride=(1, 100), align=<yateto.arch.Architecture object at 0x7f3bda2fff90>),	  eqspp=dense(shape=(100, 17), size=1700, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=F,	  memoryLayout=DenseMemoryLayout(shape=(17, 45), bbox=BoundingBox(Range(0, 17), Range(0, 45)), stride=(1, 17), align=<yateto.arch.Architecture object at 0x7f3bda2fff90>),	  eqspp=dense(shape=(17, 45), size=765, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=1.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 100), Range(0, 45), Range(0, 17))), 'matrix_a': DenseMatrix{name = E, num. rows = 100, num. columns = 17, leading dimension = 100, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 100, 17]}, 'matrix_b': DenseMatrix{name = F, num. rows = 17, num. columns = 45, leading dimension = 17, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 17, 45]}, 'matrix_c': DenseMatrix{name = A, num. rows = 100, num. columns = 45, leading dimension = 100, direction = DataFlowDirection.SINK, bbox = [0, 0, 100, 45]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F', 'A, extraOffset_A', 'numElements', 'flags', 'streamPtr']})
      */
      {
    //('gemm', {'descr': Description(  result=TensorDescription(  name=A,	  memoryLayout=DenseMemoryLayout(shape=(100, 45), bbox=BoundingBox(Range(0, 100), Range(0, 45)), stride=(1, 100), align=<yateto.arch.Architecture object at 0x7f3bda2fff90>),	  eqspp=dense(shape=(100, 45), size=4500, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=E,	  memoryLayout=DenseMemoryLayout(shape=(100, 17), bbox=BoundingBox(Range(0, 100), Range(0, 17)), stride=(1, 100), align=<yateto.arch.Architecture object at 0x7f3bda2fff90>),	  eqspp=dense(shape=(100, 17), size=1700, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=F,	  memoryLayout=DenseMemoryLayout(shape=(17, 45), bbox=BoundingBox(Range(0, 17), Range(0, 45)), stride=(1, 17), align=<yateto.arch.Architecture object at 0x7f3bda2fff90>),	  eqspp=dense(shape=(17, 45), size=765, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=1.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 100), Range(0, 45), Range(0, 17))), 'matrix_a': DenseMatrix{name = E, num. rows = 100, num. columns = 17, leading dimension = 100, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 100, 17]}, 'matrix_b': DenseMatrix{name = F, num. rows = 17, num. columns = 45, leading dimension = 17, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 17, 45]}, 'matrix_c': DenseMatrix{name = A, num. rows = 100, num. columns = 45, leading dimension = 100, direction = DataFlowDirection.SINK, bbox = [0, 0, 100, 45]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F', 'A, extraOffset_A', 'numElements', 'flags', 'streamPtr']})
        const float * const __restrict__ glb_F = &F[batchID][0 + F_extraOffset];
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_E = &E[batchID][0 + E_extraOffset];
        float reg0[45] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[2465+4500];
        float * localShrMem0 = &totalShrMem[(2465+4500) * threadIdx.y];
        
        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 13; ++i) {
            shrRegion0[threadIdx.x + i * 128] = glb_E[threadIdx.x + i * 128];
          }
          if (threadIdx.x < 36) {
            shrRegion0[threadIdx.x + 1664] = glb_E[threadIdx.x + 1664];
          }
        }
        
        float* shrRegion1 = &localShrMem0[1700];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 5; ++i) {
            shrRegion1[threadIdx.x + i * 128] = glb_F[threadIdx.x + i * 128];
          }
          if (threadIdx.x < 125) {
            shrRegion1[threadIdx.x + 640] = glb_F[threadIdx.x + 640];
          }
        }

        float* shrRegion2 = &localShrMem0[2465];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 35; ++i) {
            shrRegion2[threadIdx.x + i * 128] = glb_A[threadIdx.x + i * 128];
          }
          if (threadIdx.x < 20) {
            shrRegion2[threadIdx.x + 35*128] = glb_A[threadIdx.x + 35*128];
          }
        }
       
        __syncthreads();
        if (threadIdx.x < 100) {
          float value;
        
          #pragma unroll
          for (int k = 0; k < 17; ++k) {
            value = shrRegion0[threadIdx.x + k * 100];
        
            #pragma unroll
            for (int n = 0; n < 45; ++n) {
              reg0[n] += value * shrRegion1[k + 17 * n];
            }
          }
        }
        if (threadIdx.x < 100) {
          #pragma unroll
          for (int n = 0; n < 45; ++n) {
            glb_A[threadIdx.x + 100 * n] = reg0[n] + shrRegion2[threadIdx.x + 100 * n];
          }
        }
        
      }
    }
  }
}
void sloopOverGEMM_NT_NT_NT__d17_45_d100_45_d100_17__alpha_1_0_beta_1_0_p_p_p__4972d7a(float ** A, int A_extraOffset, const float * const * E, int E_extraOffset, const float * const * F, int F_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(128, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d17_45_d100_45_d100_17__alpha_1_0_beta_1_0_p_p_p__4972d7a<<<grid,block,0,stream>>>(A, A_extraOffset, E, E_extraOffset, F, F_extraOffset, numElements, flags);
  CHECK_ERR;
}


__global__ void 
__launch_bounds__(480)
 kernel_sproduct_NT_NT_NT__d45_d10_10_d10_10_45__alpha_1_0_p_p_p__1e400ba(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      {
        const float * const __restrict__ glb_B = &B[batchID][0 + B_extraOffset];
        const float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        float reg0[10] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[145];
        float * localShrMem0 = &totalShrMem[145 * threadIdx.y];

        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 45) {
            shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
          }
        }

        float* shrRegion1 = &localShrMem0[45];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 100) {
            shrRegion1[threadIdx.x + 0] = glb_X[threadIdx.x + 0];
          }
        }
        __syncthreads();
        /*
        This is the product kernel created from the following YaTeTo description:
        Description(
        	alpha: 1.0
        	add: True
        	result: IndexedTensorDescription(name=A, indices=kpm, memoryLayout=DenseMemoryLayout(shape=(10, 10, 45), bbox=BoundingBox(Range(0, 10), Range(0, 10), Range(0, 45)), stride=(1, 10, 100), align=<yateto.arch.Architecture object at 0x7f3bda32f2d0>), eqspp=dense(shape=(10, 10, 45), size=4500, ndim=3), is_compute_constant=False, is_temporary=False)
        	leftTerm: IndexedTensorDescription(name=B, indices=m, memoryLayout=DenseMemoryLayout(shape=(45,), bbox=BoundingBox(Range(0, 45)), stride=(1,), align=<yateto.arch.Architecture object at 0x7f3bda32f2d0>), eqspp=dense(shape=(45,), size=45, ndim=1), is_compute_constant=False, is_temporary=False)
        	rightTerm: IndexedTensorDescription(name=X, indices=kp, memoryLayout=DenseMemoryLayout(shape=(10, 10), bbox=BoundingBox(Range(0, 10), Range(0, 10)), stride=(1, 10), align=<yateto.arch.Architecture object at 0x7f3bda32f2d0>), eqspp=dense(shape=(10, 10), size=100, ndim=2), is_compute_constant=False, is_temporary=False)
        	isACsc: False
        	isBCsc: False
        	loopRanges: {'m': Range(0, 45), 'k': Range(0, 10), 'p': Range(0, 10)}
        )
        */
        if (threadIdx.x < 450) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 10;
          rows_left -= row_offset_1 * 10;
          const int dim_offset_m = row_offset_1;
          const int row_offset_0 = rows_left;
          const int dim_offset_k = row_offset_0;
          #pragma unroll
          for (int p = 0; p < 10; ++p) {
            reg0[p] = shrRegion0[dim_offset_m * 1] * shrRegion1[dim_offset_k * 1 + p * 10];
          }
        }
        if (threadIdx.x < 450) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 10;
          rows_left -= row_offset_1 * 10;
          const int row_offset_0 = rows_left;
          #pragma unroll
          for (int i = 0; i < 10; ++i) {
            glb_A[row_offset_0 * 1 + row_offset_1 * 100 + i * 10] = reg0[i] + 1.0 * glb_A[row_offset_0 * 1 + row_offset_1 * 100 + i * 10];
          }
        }
      }
    }
  }
}
void sproduct_NT_NT_NT__d45_d10_10_d10_10_45__alpha_1_0_p_p_p__1e400ba(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(480, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sproduct_NT_NT_NT__d45_d10_10_d10_10_45__alpha_1_0_p_p_p__1e400ba<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}



int main(){
  constexpr size_t num_els = 116141;
  float* A = new float[4500 * num_els]{0.f};
  float* B = new float[45 * num_els]{0.f};
  float* C = new float[320 * num_els]{0.f};
  float* D = new float[320 * num_els]{0.f};
  float* E = new float[1700 * num_els]{0.f};
  float* F = new float[765 * num_els]{0.f};
  float* X = new float[100 * num_els]{0.f};
  float* R1 = new float[4500 * num_els]{0.f};
  float* R2 = new float[4500 * num_els]{0.f};
  //float* Ri1 = new float[100 * num_els]{0.f};
  //float* Ri2 = new float[4500 * num_els]{0.f};
  //float* Ri1c = new float[100 * num_els]{0.f};
  //float* Ri2c = new float[4500 * num_els]{0.f};


  float* coreA = new float[4500];
  float* coreB = new float[45];
  float* coreC = new float[320];
  float* coreD = new float[320];
  float* coreE = new float[1700];
  float* coreF = new float[765];

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<int> distribution(1, 100);
  for (size_t i = 0; i < 4500; i++){
    coreA[i] = distribution(gen);
  }
  for (size_t i = 0; i < 45; i++){
    coreB[i] = distribution(gen);
  }
  for (size_t i = 0; i < 320; i++){
    coreC[i] = distribution(gen);
  }
  for (size_t i = 0; i < 320; i++){
    coreD[i] = distribution(gen);
  }
  for (size_t i = 0; i < 1700; i++){
    coreE[i] = distribution(gen);
  }
  for (size_t i = 0; i < 765; i++){
    coreF[i] = distribution(gen);
  }

  for (size_t i = 0; i < num_els; i++){
      std::memcpy(&A[i * 4500], &coreA[0], 4500 * sizeof(float));
      std::memcpy(&B[i * 45], &coreB[0], 45 * sizeof(float));
      std::memcpy(&C[i * 320], &coreC[0], 320 * sizeof(float));
      std::memcpy(&D[i * 320], &coreD[0], 320 * sizeof(float));
      std::memcpy(&E[i * 1700], &coreE[0], 1700 * sizeof(float));
      std::memcpy(&F[i * 765], &coreF[0], 765 * sizeof(float));
  }

  float* A_dev = nullptr;
  float* B_dev = nullptr;
  float* C_dev = nullptr;
  float* D_dev = nullptr;
  float* E_dev = nullptr;
  float* F_dev = nullptr;
  float* X_dev = nullptr;

  float** A_dev_begins = new float*[num_els];
  float** B_dev_begins = new float*[num_els];
  float** C_dev_begins = new float*[num_els];
  float** D_dev_begins = new float*[num_els];
  float** E_dev_begins = new float*[num_els];
  float** F_dev_begins = new float*[num_els];
  float** X_dev_begins = new float*[num_els];

  float** A_dev_begins_dev = nullptr;
  float** B_dev_begins_dev = nullptr;
  float** C_dev_begins_dev = nullptr;
  float** D_dev_begins_dev = nullptr;
  float** E_dev_begins_dev = nullptr;
  float** F_dev_begins_dev = nullptr;
  float** X_dev_begins_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 4500 * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev, sizeof(float) * 45 * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev, sizeof(float) * 320 * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev, sizeof(float) * 320 * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev, sizeof(float) * 1700 * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev, sizeof(float) * 765 * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev, sizeof(float) * 100 * num_els); CHECK_ERR;

  hipMalloc((void **)&A_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
 
  hipDeviceSynchronize(); CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4500 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev, (void *)B, sizeof(float) * 45 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev, (void *)C, sizeof(float) * 320 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev, (void *)D, sizeof(float) * 320 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev, (void *)E, sizeof(float) * 1700 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev, (void *)F, sizeof(float) * 765 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 100 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  for (size_t i = 0; i < num_els; i++){
    A_dev_begins[i] = A_dev + i * 4500;
    B_dev_begins[i] = B_dev + i * 45;
    C_dev_begins[i] = C_dev + i * 320;
    D_dev_begins[i] = D_dev + i * 320;
    E_dev_begins[i] = E_dev + i * 1700;
    F_dev_begins[i] = F_dev + i * 765;
    X_dev_begins[i] = X_dev + i * 100;
  }

  hipMemcpy((void *)A_dev_begins_dev, (void *)A_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev_begins_dev, (void *)B_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev_begins_dev, (void *)C_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev_begins_dev, (void *)D_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev_begins_dev, (void *)E_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev_begins_dev, (void *)F_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev_begins_dev, (void *)X_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  sloopOverGEMM_NT_NT_NT__d10_32_d32_10_d10_10__alpha_1_0_beta_0_0_p_p_p__6a7ea95(C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipDeviceSynchronize(); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 100 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  std::cout << "Dimensions: " << 10 << ", " << 17 << ", " << 45 << ", " << 10 << ", " << 32 << std::endl;

  float elapsedTimeT1 = 0.0;
  float elapsedTimeT2 = 0.0;
  float elapsedTimeT3 = 0.0; 
  hipEvent_t startT1, stopT1;
  hipEvent_t startT2, stopT2;
  hipEvent_t startT3, stopT3;
  hipEventCreate(&startT1); CHECK_ERR;
  hipEventCreate(&stopT1); CHECK_ERR;
  hipEventRecord(startT1); CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d10_32_d32_10_d10_10__alpha_1_0_beta_0_0_p_p_p__6a7ea95(C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT1); CHECK_ERR;
  hipEventSynchronize(stopT1); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT1, startT1, stopT1); CHECK_ERR;
  //hipDeviceSynchronize(); CHECK_ERR;

  //hipMemcpy(Ri1, X_dev, sizeof(float) * 100 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  
  hipEventCreate(&startT2); CHECK_ERR;
  hipEventCreate(&stopT2); CHECK_ERR;
  hipEventRecord(startT2); CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d17_45_d100_45_d100_17__alpha_1_0_beta_1_0_p_p_p__4972d7a(A_dev_begins_dev, 0, E_dev_begins_dev, 0, F_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT2); CHECK_ERR;
  hipEventSynchronize(stopT2); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT2, startT2, stopT2); CHECK_ERR;
  //hipDeviceSynchronize(); CHECK_ERR;

  //hipMemcpy(Ri2, A_dev, sizeof(float) * 4500 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

  hipEventCreate(&startT3); CHECK_ERR;
  hipEventCreate(&stopT3); CHECK_ERR;
  hipEventRecord(startT3); CHECK_ERR;
  sproduct_NT_NT_NT__d45_d10_10_d10_10_45__alpha_1_0_p_p_p__1e400ba(A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT3); CHECK_ERR;
  hipEventSynchronize(stopT3); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT3, startT3, stopT3); CHECK_ERR;
  double elapsedTime = elapsedTimeT1 + elapsedTimeT2 + elapsedTimeT3;
  hipDeviceSynchronize(); CHECK_ERR;
  
  std::cout << "Gemmforge Tensor Contraction took: " << elapsedTime << " ms" << std::endl; 
  hipMemcpy(R1, A_dev, sizeof(float) * 4500 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4500 * num_els, hipMemcpyHostToDevice); CHECK_ERR;


  double fp_per_el = 172900;
  double ls_per_el = 48600;
  double fp_unfused_per_el = 172900;
  double ls_unfused_per_el = 85400;
  fp_per_el *= num_els;
  ls_per_el *= num_els;
  fp_unfused_per_el *= num_els;
  ls_unfused_per_el *= num_els;
  std::cout << "Gemmforge Theoretical Fused Kernel GFLOPs/s: " << fp_per_el * 1e-6 / elapsedTime << std::endl;
  std::cout << "Operational Theoretical Fused intensity: " << fp_per_el / ls_per_el << std::endl;
  std::cout << "Gemmforge GFLOPs/s: " << fp_unfused_per_el * 1e-6 / elapsedTime << std::endl;
  std::cout << "Operational intensity: " << fp_unfused_per_el / ls_unfused_per_el << std::endl;
  double peakFLOPGiven = 29767.7;
  double peakBandwidthGiven = 760.08;

  if (peakFLOPGiven > 0.1 && peakBandwidthGiven){
    double obtainable_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_per_el) / static_cast<double>(ls_per_el)));
    std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTime) / obtainable_peak << " % of roof w. respect to operational intensity achieved with Gemmforge" << std::endl;
    //std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTime) / obtainable_peak << " % of roof w. respect to operational intensity achieved with cuTensor" << std::endl;
    double obtainable_unfused_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_unfused_per_el) / static_cast<double>(ls_unfused_per_el)));
    std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTime) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with Gemmforge" << std::endl;
    //std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTime) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with cuTensor" << std::endl;
    double obtainable_unfused_peak_k1 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(6400) / static_cast<double>(2960)));
    std::cout << 100.0*(6400 * num_els  * 1e-6 / elapsedTimeT1) / obtainable_unfused_peak_k1 << " % of roof w. respect to Kernel1 intensity achieved with Gemmforge" << std::endl;
    double obtainable_unfused_peak_k2 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(157500) / static_cast<double>(45860)));
    std::cout << 100.0*(157500 * num_els  * 1e-6 / elapsedTimeT2) / obtainable_unfused_peak_k2 << " % of roof w. respect to Kernel2 intensity achieved with Gemmforge" << std::endl;
    double obtainable_unfused_peak_k3 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(9000) / static_cast<double>(36580)));
    std::cout << 100.0*(9000 * num_els * 1e-6 / elapsedTimeT3) / obtainable_unfused_peak_k3 << " % of roof w. respect to Kernel3 intensity achieved with Gemmforge" << std::endl;
  }

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4500 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 100 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  if constexpr (!false){
  hiptensorHandle_t* handle;
  HANDLE_ERROR(hiptensorCreate(&handle));

  hipEvent_t startCT1, stopCT1;
  hipEvent_t startCT2, stopCT2;
  hipEvent_t startCT3, stopCT3;
  hipEventCreate(&startCT1); CHECK_ERR;
  hipEventCreate(&stopCT1); CHECK_ERR;
  hipEventCreate(&startCT2); CHECK_ERR;
  hipEventCreate(&stopCT2); CHECK_ERR;
  hipEventCreate(&startCT3); CHECK_ERR;
  hipEventCreate(&stopCT3); CHECK_ERR;
  float elapsedTimeCT1 = 0.f;
  float elapsedTimeCT2 = 0.f;
  float elapsedTimeCT3 = 0.f;

  // Kernel 1
  std::cout << "cuTensor Kernel 1" << std::endl;
  {
    float alphaK1 = 1.0f;
    float betaK1 = 0.0f;
    float alphaK2 = 1.0f;
    float betaK2 = 1.0;
    float alphaK3 = 1.0f;
    float betaK3 = 1.0;

    std::vector<int> modeA{'k', 'p', 'm', 'b'};
    std::vector<int> modeB{'m', 'b'};
    std::vector<int> modeC{'k', 'q', 'b'};
    std::vector<int> modeD{'q', 'p', 'b'};
    std::vector<int> modeE{'k', 'p', 'l', 'b'};
    std::vector<int> modeF{'l', 'm', 'b'};
    std::vector<int> modeX{'k', 'p', 'b'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();
    int nmodeD = modeD.size();
    int nmodeE = modeE.size();
    int nmodeF = modeF.size();
    int nmodeX = modeX.size();

    std::unordered_map<int, int64_t> extent;
    // Derived from the kernel
    extent['k'] = 10;
    extent['l'] = 17;
    extent['m'] = 45;
    extent['p'] = 10;
    extent['q'] = 32;
    extent['b'] = num_els;

    std::vector<int64_t> extentA;
    for (auto mode : modeA) {
        extentA.push_back(extent[mode]);
    }
    std::vector<int64_t> extentB;
    for (auto mode : modeB) {
        extentB.push_back(extent[mode]);
    }
    std::vector<int64_t> extentC;
    for (auto mode : modeC) {
        extentC.push_back(extent[mode]);
    }
    std::vector<int64_t> extentD;
    for (auto mode : modeD) {
        extentD.push_back(extent[mode]);
    }
    std::vector<int64_t> extentE;
    for (auto mode : modeE) {
        extentE.push_back(extent[mode]);
    }
    std::vector<int64_t> extentF;
    for (auto mode : modeF) {
        extentF.push_back(extent[mode]);
    }
    std::vector<int64_t> extentX;
    for (auto mode : modeX) {
        extentX.push_back(extent[mode]);
    }
    
    hipDataType typeA = HIP_R_32F;
    hipDataType typeB = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hipDataType typeD = HIP_R_32F;
    hipDataType typeE = HIP_R_32F;
    hipDataType typeF = HIP_R_32F;
    hipDataType typeX = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descA,
                    nmodeA,
                    extentA.data(),
                    NULL,
                    typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descB,
                    nmodeB,
                    extentB.data(),
                    NULL,
                    typeB, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descC,
                    nmodeC,
                    extentC.data(),
                    NULL,
                    typeC, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descD;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descD,
                    nmodeD,
                    extentD.data(),
                    NULL,
                    typeD, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descE;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descE,
                    nmodeE,
                    extentE.data(),
                    NULL,
                    typeE, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descF;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descF,
                    nmodeF,
                    extentF.data(),
                    NULL,
                    typeF, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descX;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descX,
                    nmodeX,
                    extentX.data(),
                    NULL,
                    typeX, HIPTENSOR_OP_IDENTITY));


    uint32_t alignmentRequirementA;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    A_dev,
                    &descA,
                    &alignmentRequirementA));

    uint32_t alignmentRequirementB;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    B_dev,
                    &descB,
                    &alignmentRequirementB));

    uint32_t alignmentRequirementC;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    C_dev,
                    &descC, 
                    &alignmentRequirementC));

    uint32_t alignmentRequirementD;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    D_dev,
                    &descD,
                    &alignmentRequirementD));

    uint32_t alignmentRequirementE;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    E_dev,
                    &descE,
                    &alignmentRequirementE));

    uint32_t alignmentRequirementF;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    F_dev,
                    &descF, 
                    &alignmentRequirementF));

    uint32_t alignmentRequirementX;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    X_dev,
                    &descX, 
                    &alignmentRequirementX));

    cutensorContractionDescriptor_t desc1;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc1,
                  &descC, modeC.data(), alignmentRequirementC,
                  &descD, modeD.data(), alignmentRequirementD,
                  &descX, modeX.data(), alignmentRequirementX,
                  &descX, modeX.data(), alignmentRequirementX,
                  typeCompute));

    cutensorContractionFind_t find1;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find1, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize1 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc1,
                 &find1,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize1));

    cutensorContractionDescriptor_t desc2;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc2,
                  &descF, modeF.data(), alignmentRequirementF,
                  &descE, modeE.data(), alignmentRequirementE,
                  &descA, modeA.data(), alignmentRequirementA,
                  &descA, modeA.data(), alignmentRequirementA,
                  typeCompute));

    cutensorContractionFind_t find2;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find2, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize2 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc2,
                 &find2,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize2));


    cutensorContractionDescriptor_t desc3;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc3,
                  &descB, modeB.data(), alignmentRequirementB,
                  &descX, modeX.data(), alignmentRequirementX,
                  &descA, modeA.data(), alignmentRequirementA,
                  &descA, modeA.data(), alignmentRequirementA,
                  typeCompute));

    cutensorContractionFind_t find3;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find3, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize3 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc3,
                 &find3,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize3));

    uint64_t maxWorkSize = std::max(std::max(worksize1, worksize2), worksize3);
    void *work = nullptr;
    if (maxWorkSize > 0)
    {
        if (hipSuccess != hipMalloc(&work, maxWorkSize))
        {
            work = nullptr;
            maxWorkSize = 0;
            worksize1 = 0;
            worksize2 = 0;
            worksize3 = 0;
            hipGetLastError(); // Clear last error to save CHECK_ERR;
        } else {
            worksize1 = maxWorkSize;
            worksize2 = maxWorkSize;
            worksize3 = maxWorkSize;
        }
    }


    hiptensorContractionPlan_t plan1;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan1,
                 &desc1,
                 &find1,
                 worksize1));

    hiptensorContractionPlan_t plan2;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan2,
                 &desc2,
                 &find2,
                 worksize2));

    hiptensorContractionPlan_t plan3;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan3,
                 &desc3,
                 &find3,
                 worksize3));

    hipDeviceSynchronize(); CHECK_ERR;

    hipEventRecord(startCT1); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan1,
                              (void*) &alphaK1, C_dev, D_dev,
                              (void*) &betaK1,  X_dev, X_dev, 
                              work, worksize1, 0);
    hipEventRecord(stopCT1); CHECK_ERR;
    hipEventSynchronize(stopCT1); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT1, startCT1, stopCT1); CHECK_ERR;

    //hipDeviceSynchronize(); CHECK_ERR;
    //hipMemcpy(Ri1c, X_dev, sizeof(float) * 100 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipEventRecord(startCT2); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan2,
                              (void*) &alphaK2, F_dev, E_dev,
                              (void*) &betaK2,  A_dev, A_dev, 
                              work, worksize2, 0);
    hipEventRecord(stopCT2); CHECK_ERR;
    hipEventSynchronize(stopCT2); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT2, startCT2, stopCT2); CHECK_ERR;

    //hipDeviceSynchronize(); CHECK_ERR;
    //hipMemcpy(Ri2c, A_dev, sizeof(float) * 4500 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipEventRecord(startCT3); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan3,
                              (void*) &alphaK3, B_dev, X_dev,
                              (void*) &betaK3,  A_dev, A_dev, 
                              work, worksize3, 0);
    hipEventRecord(stopCT3); CHECK_ERR;
    hipEventSynchronize(stopCT3); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT3, startCT3, stopCT3); CHECK_ERR;

    hipDeviceSynchronize(); CHECK_ERR;
    
    hipMemcpy(R2, A_dev, sizeof(float) * 4500 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipFree(work);
  }

  float elapsedTimeCuTensor = elapsedTimeCT1 + elapsedTimeCT2 + elapsedTimeCT2;
  if (peakFLOPGiven > 0.1 && peakBandwidthGiven){
    double obtainable_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_per_el) / static_cast<double>(ls_per_el)));
    std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTimeCuTensor) / obtainable_peak << " % of roof w. respect to operational intensity achieved with cuTensor" << std::endl;

    double obtainable_unfused_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_unfused_per_el) / static_cast<double>(ls_unfused_per_el)));
    std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTimeCuTensor) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with cuTensor" << std::endl;
  }

  /*
  bool i1results_wrong = false;
  for (size_t i = 0; i < 100 * num_els; i++){
    if (std::abs(Ri1[i] - Ri1c[i]) > 1.0f) {
      std::cout << "Intermediate Results 1 do not match, problem first at offset " << i << " :_(" << std::endl;
      i1results_wrong = true;
      break;
    }
  }
  if (!i1results_wrong){
    std::cout << "Gemmforge and cuTensor contraction intermediate results 1 match! :)" << std::endl;
  }
  
  bool i2results_wrong = false;
  for (size_t i = 0; i < 4500 * num_els; i++){
    if (std::abs(Ri2[i] - Ri2c[i]) > 1.0f) {
      std::cout << "Intermediate Results 2 do not match, problem first at offset " << i << " :_(" << std::endl;
      i2results_wrong = true;
      break;
    }
  }
  if (!i2results_wrong){
    std::cout << "Gemmforge and cuTensor contraction intermediate results 2 match! :)" << std::endl;
  }
  */

  bool results_wrong = false;
  for (size_t i = 0; i < 4500 * num_els; i++){
    if (std::abs(R1[i] - R2[i]) > 5.0f) {
      std::cout << "Results do not match, problem first at offset " << i << " :_(" << std::endl;
      results_wrong = true;
      break;
    }
  }
  if (!results_wrong){
    std::cout << "Gemmforge and cuTensor contraction results match! :)" << std::endl;
  }
  }

  hipFree(A_dev_begins_dev);
  hipFree(B_dev_begins_dev);
  hipFree(C_dev_begins_dev);
  hipFree(D_dev_begins_dev);
  hipFree(E_dev_begins_dev);
  hipFree(F_dev_begins_dev);
  hipFree(X_dev_begins_dev);

  delete[] A;
  delete[] B;
  delete[] C;
  delete[] D;
  delete[] E;
  delete[] F;
  delete[] X;
  delete[] A_dev_begins;
  delete[] B_dev_begins;
  delete[] C_dev_begins;
  delete[] D_dev_begins;
  delete[] E_dev_begins;
  delete[] F_dev_begins;
  delete[] X_dev_begins;
  delete[] R1;
  delete[] R2;

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  hipFree(D_dev);
  hipFree(E_dev);
  hipFree(F_dev);
  hipFree(X_dev);

  delete[] coreA;
  delete[] coreB;
  delete[] coreC;
  delete[] coreD;
  delete[] coreE;
  delete[] coreF;

  return 0;
}

