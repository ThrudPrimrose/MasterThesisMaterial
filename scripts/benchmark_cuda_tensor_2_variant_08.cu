#include "hip/hip_runtime.h"

#include <cstring>
#include <iostream>
#include <random>
#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                                        \
  {                                                                            \
    const auto err = x;                                                        \
    if (err != HIPTENSOR_STATUS_SUCCESS) {                                      \
      std::cout << "Error: " << hiptensorGetErrorString(err) << std::endl;      \
      std::cout << __FILE__ << " " << __LINE__ << std::endl;                   \
    }                                                                          \
  }

#define CHECK_ERR checkErr(__FILE__, __LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    // We don't exit when we encounter CUDA errors in this example.
    // std::exit(EXIT_FAILURE);
  }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
  hipError_t Error = hipGetLastError();
  if (Error != hipSuccess) {
    std::cout << std::endl
              << File << ", line " << Line << ": " << hipGetErrorString(Error)
              << " (" << Error << ")" << std::endl;

    if (PrevLine > 0)
      std::cout << "Previous CUDA call:" << std::endl
                << PrevFile << ", line " << PrevLine << std::endl;
  }
  PrevFile = File;
  PrevLine = Line;
#endif
}

__global__ void __launch_bounds__(32)
    kernel_sloopOverGEMM_NT_NT_NT__d21_61_d21_22_d61_22__alpha_1_0_beta_0_0_p_p_p__7610af5(
        const float *const *C, int C_extraOffset, const float *const *D,
        int D_extraOffset, float **X, int X_extraOffset, unsigned numElements,
        unsigned *flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=X,
      memoryLayout=DenseMemoryLayout(shape=(21, 22), bbox=BoundingBox(Range(0,
      21), Range(0, 22)), stride=(1, 21), align=<yateto.arch.Architecture object
      at 0x7f0cf15817d0>),	  eqspp=dense(shape=(21, 22), size=462, ndim=2),
      is_compute_constant=False,	  is_temporary=False),
      leftTerm=TensorDescription(  name=C,
      memoryLayout=DenseMemoryLayout(shape=(21, 61), bbox=BoundingBox(Range(0,
      21), Range(0, 61)), stride=(1, 21), align=<yateto.arch.Architecture object
      at 0x7f0cf15817d0>),	  eqspp=dense(shape=(21, 61), size=1281,
      ndim=2), is_compute_constant=False,	  is_temporary=False),
      rightTerm=TensorDescription(  name=D,
      memoryLayout=DenseMemoryLayout(shape=(61, 22), bbox=BoundingBox(Range(0,
      61), Range(0, 22)), stride=(1, 61), align=<yateto.arch.Architecture object
      at 0x7f0cf15817d0>),	  eqspp=dense(shape=(61, 22), size=1342,
      ndim=2), is_compute_constant=False,	  is_temporary=False),
      transA=False, transB=False,	  alpha=1.0,	  beta=0.0,
      prefetchName=None, isACsc=False,	  isBCsc=False, alignedA=False,
      alignedC=False, mnk=(Range(0, 21), Range(0, 22), Range(0, 61))),
      'matrix_a': DenseMatrix{name = C, num. rows = 21, num. columns = 61,
      leading dimension = 21, direction = DataFlowDirection.SOURCE, bbox = [0,
      0, 21, 61]}, 'matrix_b': DenseMatrix{name = D, num. rows = 61, num.
      columns = 22, leading dimension = 61, direction =
      DataFlowDirection.SOURCE, bbox = [0, 0, 61, 22]}, 'matrix_c':
      DenseMatrix{name = X, num. rows = 21, num. columns = 22, leading dimension
      = 21, direction = DataFlowDirection.SINK, bbox = [0, 0, 21, 22]}, 'args':
      ['C, extraOffset_C', 'D, extraOffset_D', 'X, extraOffset_X',
      'numElements', 'flags', 'streamPtr']})
      */
      {
        //('gemm', {'descr': Description(  result=TensorDescription(  name=X,
        // memoryLayout=DenseMemoryLayout(shape=(21, 22),
        // bbox=BoundingBox(Range(0, 21), Range(0, 22)), stride=(1, 21),
        // align=<yateto.arch.Architecture object at 0x7f0cf15817d0>),
        // eqspp=dense(shape=(21, 22), size=462, ndim=2),
        // is_compute_constant=False,	  is_temporary=False),
        // leftTerm=TensorDescription(  name=C,
        // memoryLayout=DenseMemoryLayout(shape=(21, 61),
        // bbox=BoundingBox(Range(0, 21), Range(0, 61)), stride=(1, 21),
        // align=<yateto.arch.Architecture object at 0x7f0cf15817d0>),
        // eqspp=dense(shape=(21, 61), size=1281, ndim=2),
        // is_compute_constant=False,	  is_temporary=False),
        // rightTerm=TensorDescription(  name=D,
        // memoryLayout=DenseMemoryLayout(shape=(61, 22),
        // bbox=BoundingBox(Range(0, 61), Range(0, 22)), stride=(1, 61),
        // align=<yateto.arch.Architecture object at 0x7f0cf15817d0>),
        // eqspp=dense(shape=(61, 22), size=1342, ndim=2),
        // is_compute_constant=False,	  is_temporary=False), transA=False,
        // transB=False,	  alpha=1.0,	  beta=0.0, prefetchName=None,
        // isACsc=False,	  isBCsc=False,	  alignedA=False,
        // alignedC=False, mnk=(Range(0, 21), Range(0, 22), Range(0, 61))),
        // 'matrix_a': DenseMatrix{name = C, num. rows = 21, num. columns = 61,
        // leading dimension = 21, direction = DataFlowDirection.SOURCE, bbox =
        // [0, 0, 21, 61]}, 'matrix_b': DenseMatrix{name = D, num. rows = 61,
        // num. columns = 22, leading dimension = 61, direction =
        // DataFlowDirection.SOURCE, bbox = [0, 0, 61, 22]}, 'matrix_c':
        // DenseMatrix{name = X, num. rows = 21, num. columns = 22, leading
        // dimension = 21, direction = DataFlowDirection.SINK, bbox = [0, 0, 21,
        // 22]}, 'args': ['C, extraOffset_C', 'D, extraOffset_D', 'X,
        // extraOffset_X', 'numElements', 'flags', 'streamPtr']})
        const float *const __restrict__ glb_C = &C[batchID][0 + C_extraOffset];
        float *const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        const float *const __restrict__ glb_D = &D[batchID][0 + D_extraOffset];
        float reg0[21] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                          0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                          0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__ __align__(16) float totalShrMem1[1952];
        float *localShrMem0 = &totalShrMem1[1952 * threadIdx.y];
        __shared__ __align__(16) float totalShrMem2[1408];
        float *localShrMem1 = &totalShrMem2[1408 * threadIdx.y];
        __shared__ __align__(16) float shrC[22 * 21];

        float *shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
#pragma unroll
          for (int i = 0; i < 61; ++i) {
            if (threadIdx.x < 21) {
              shrRegion0[threadIdx.x + i * 32] = glb_C[threadIdx.x + i * 21];
            }
          }
        }

        float *shrRegion1 = &localShrMem1[0];
        // using ExtendedPatchLoader
        {
#pragma unroll
          for (int i = 0; i < 22; ++i) {
            shrRegion1[threadIdx.x + i * 64] = glb_D[threadIdx.x + i * 61];
            if (threadIdx.x < 29) {
              shrRegion1[threadIdx.x + 32 + i * 64] =
                  glb_D[threadIdx.x + 32 + i * 61];
            }
          }
        }
        __syncwarp();
        if (threadIdx.x < 22) {

          // float value;

          
          float4 values;
          #pragma unroll
          for (int k = 0; k < 60; k += 4) {
            values.x = shrRegion1[threadIdx.x * 64 + 0 + k];
            values.y = shrRegion1[threadIdx.x * 64 + 1 + k];
            values.z = shrRegion1[threadIdx.x * 64 + 2 + k];
            values.w = shrRegion1[threadIdx.x * 64 + 3 + k];

          #pragma unroll
            for (int n = 0; n < 20; n += 4) {
              reg0[n + 0] += values.x * shrRegion0[k*32 + 0 + n];
              reg0[n + 1] += values.x * shrRegion0[k*32 + 1 + n];
              reg0[n + 2] += values.x * shrRegion0[k*32 + 2 + n];
              reg0[n + 3] += values.x * shrRegion0[k*32 + 3 + n];
              reg0[n + 0] += values.y * shrRegion0[(k+1)*32 + 0 + n];
              reg0[n + 1] += values.y * shrRegion0[(k+1)*32 + 1 + n];
              reg0[n + 2] += values.y * shrRegion0[(k+1)*32 + 2 + n];
              reg0[n + 3] += values.y * shrRegion0[(k+1)*32 + 3 + n];
              reg0[n + 0] += values.z * shrRegion0[(k+2)*32 + 0 + n];
              reg0[n + 1] += values.z * shrRegion0[(k+2)*32 + 1 + n];
              reg0[n + 2] += values.z * shrRegion0[(k+2)*32 + 2 + n];
              reg0[n + 3] += values.z * shrRegion0[(k+2)*32 + 3 + n];
              reg0[n + 0] += values.w * shrRegion0[(k+3)*32 + 0 + n];
              reg0[n + 1] += values.w * shrRegion0[(k+3)*32 + 1 + n];
              reg0[n + 2] += values.w * shrRegion0[(k+3)*32 + 2 + n];
              reg0[n + 3] += values.w * shrRegion0[(k+3)*32 + 3 + n];
            }
            reg0[20] += values.x * shrRegion0[20 + k*32];
            reg0[20] += values.y * shrRegion0[20 + (k+1)*32];
            reg0[20] += values.z * shrRegion0[20 + (k+2)*32];
            reg0[20] += values.w * shrRegion0[20 + (k+3)*32];
          }
          values.x = shrRegion1[threadIdx.x * 64 + 0 + 60];
          #pragma unroll
          for (int n = 0; n < 21; n += 1) {
            reg0[n] += values.x * shrRegion0[n + 60*32];
          }
          /*
          float value;
#pragma unroll
          for (int k = 0; k < 61; k++) {
            value = shrRegion1[threadIdx.x * 64 + k];
#pragma unroll
            for (int m = 0; m < 21; m++) {
              reg0[m] += value * shrRegion0[m + k * 32];
            }
          }
          */
        }

        if (threadIdx.x < 22) {
#pragma unroll
          for (int m = 0; m < 21; ++m) {
            shrC[threadIdx.x * 21 + m] = reg0[m];
          }
        }
        __syncwarp();
#pragma unroll
        for (int i = 0; i < 14; i++) {
          glb_X[threadIdx.x + i * 32] = shrC[threadIdx.x + i * 32];
        }
        if (threadIdx.x < 14) {
          glb_X[threadIdx.x + 14 * 32] = shrC[threadIdx.x + 14 * 32];
        }
      }
    }
  }
}
void sloopOverGEMM_NT_NT_NT__d21_61_d21_22_d61_22__alpha_1_0_beta_0_0_p_p_p__7610af5(
    const float *const *C, int C_extraOffset, const float *const *D,
    int D_extraOffset, float **X, int X_extraOffset, unsigned numElements,
    unsigned *flags, void *streamPtr) {
  dim3 block(32, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream =
      (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d21_61_d21_22_d61_22__alpha_1_0_beta_0_0_p_p_p__7610af5<<<
      grid, block, 0, stream>>>(C, C_extraOffset, D, D_extraOffset, X,
                                X_extraOffset, numElements, flags);
  CHECK_ERR;
}

__global__ void __launch_bounds__(480)
    kernel_sloopOverGEMM_NT_NT_NT__d462_13_d462_10_d13_10__alpha_1_0_beta_1_0_p_p_p__419e9f1(
        float **A, int A_extraOffset, const float *const *E, int E_extraOffset,
        const float *const *F, int F_extraOffset, unsigned numElements,
        unsigned *flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=A,
      memoryLayout=DenseMemoryLayout(shape=(462, 10), bbox=BoundingBox(Range(0,
      462), Range(0, 10)), stride=(1, 462), align=<yateto.arch.Architecture
      object at 0x7f0cfd758d90>),	  eqspp=dense(shape=(462, 10),
      size=4620, ndim=2),	  is_compute_constant=False,
      is_temporary=False), leftTerm=TensorDescription(  name=E,
      memoryLayout=DenseMemoryLayout(shape=(462, 13), bbox=BoundingBox(Range(0,
      462), Range(0, 13)), stride=(1, 462), align=<yateto.arch.Architecture
      object at 0x7f0cfd758d90>),	  eqspp=dense(shape=(462, 13),
      size=6006, ndim=2),	  is_compute_constant=False,
      is_temporary=False), rightTerm=TensorDescription(  name=F,
      memoryLayout=DenseMemoryLayout(shape=(13, 10), bbox=BoundingBox(Range(0,
      13), Range(0, 10)), stride=(1, 13), align=<yateto.arch.Architecture object
      at 0x7f0cfd758d90>),	  eqspp=dense(shape=(13, 10), size=130, ndim=2),
      is_compute_constant=False,	  is_temporary=False),	  transA=False,
      transB=False,	  alpha=1.0,	  beta=1.0,	  prefetchName=None,
      isACsc=False,	  isBCsc=False,	  alignedA=False, alignedC=False,
      mnk=(Range(0, 462), Range(0, 10), Range(0, 13))), 'matrix_a':
      DenseMatrix{name = E, num. rows = 462, num. columns = 13, leading
      dimension = 462, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 462,
      13]}, 'matrix_b': DenseMatrix{name = F, num. rows = 13, num. columns = 10,
      leading dimension = 13, direction = DataFlowDirection.SOURCE, bbox = [0,
      0, 13, 10]}, 'matrix_c': DenseMatrix{name = A, num. rows = 462, num.
      columns = 10, leading dimension = 462, direction = DataFlowDirection.SINK,
      bbox = [0, 0, 462, 10]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F',
      'A, extraOffset_A', 'numElements', 'flags', 'streamPtr']})
      */
      {
        //('gemm', {'descr': Description(  result=TensorDescription(  name=A,
        // memoryLayout=DenseMemoryLayout(shape=(462, 10),
        // bbox=BoundingBox(Range(0, 462), Range(0, 10)), stride=(1, 462),
        // align=<yateto.arch.Architecture object at 0x7f0cfd758d90>),
        // eqspp=dense(shape=(462, 10), size=4620, ndim=2),
        // is_compute_constant=False,	  is_temporary=False),
        // leftTerm=TensorDescription(  name=E,
        // memoryLayout=DenseMemoryLayout(shape=(462, 13),
        // bbox=BoundingBox(Range(0, 462), Range(0, 13)), stride=(1, 462),
        // align=<yateto.arch.Architecture object at 0x7f0cfd758d90>),
        // eqspp=dense(shape=(462, 13), size=6006, ndim=2),
        // is_compute_constant=False,	  is_temporary=False),
        // rightTerm=TensorDescription(  name=F,
        // memoryLayout=DenseMemoryLayout(shape=(13, 10),
        // bbox=BoundingBox(Range(0, 13), Range(0, 10)), stride=(1, 13),
        // align=<yateto.arch.Architecture object at 0x7f0cfd758d90>),
        // eqspp=dense(shape=(13, 10), size=130, ndim=2),
        // is_compute_constant=False,	  is_temporary=False), transA=False,
        // transB=False,	  alpha=1.0,	  beta=1.0, prefetchName=None,
        // isACsc=False,	  isBCsc=False,	  alignedA=False,
        // alignedC=False, mnk=(Range(0, 462), Range(0, 10), Range(0, 13))),
        // 'matrix_a': DenseMatrix{name = E, num. rows = 462, num. columns = 13,
        // leading dimension = 462, direction = DataFlowDirection.SOURCE, bbox =
        // [0, 0, 462, 13]}, 'matrix_b': DenseMatrix{name = F, num. rows = 13,
        // num. columns = 10, leading dimension = 13, direction =
        // DataFlowDirection.SOURCE, bbox = [0, 0, 13, 10]}, 'matrix_c':
        // DenseMatrix{name = A, num. rows = 462, num. columns = 10, leading
        // dimension = 462, direction = DataFlowDirection.SINK, bbox = [0, 0,
        // 462, 10]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F', 'A,
        // extraOffset_A', 'numElements', 'flags', 'streamPtr']})
        const float *const __restrict__ glb_E = &E[batchID][0 + E_extraOffset];
        float *const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float *const __restrict__ glb_F = &F[batchID][0 + F_extraOffset];
        float reg0[10] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                          0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__ __align__(8) float totalShrMem[6136];
        float *localShrMem0 = &totalShrMem[6136 * threadIdx.y];

        float *shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
#pragma unroll
          for (int i = 0; i < 12; ++i) {
            shrRegion0[threadIdx.x + i * 480] = glb_E[threadIdx.x + i * 480];
          }
          if (threadIdx.x < 246) {
            shrRegion0[threadIdx.x + 5760] = glb_E[threadIdx.x + 5760];
          }
        }

        float *shrRegion1 = &localShrMem0[6006];
        // using ExtendedPatchLoader
        {
          if (threadIdx.x < 130) {
            shrRegion1[threadIdx.x + 0] = glb_F[threadIdx.x + 0];
          }
        }
        __syncthreads();
        if (threadIdx.x < 462) {
          float value;

#pragma unroll
          for (int k = 0; k < 13; ++k) {
            value = shrRegion0[threadIdx.x + k * 462];

#pragma unroll
            for (int n = 0; n < 10; ++n) {
              reg0[n] += value * shrRegion1[k + 13 * n];
            }
          }
        }
        if (threadIdx.x < 462) {
#pragma unroll
          for (int n = 0; n < 10; ++n) {
            glb_A[threadIdx.x + 462 * n] =
                reg0[n] + glb_A[threadIdx.x + 462 * n];
          }
        }
      }
    }
  }
}

void sloopOverGEMM_NT_NT_NT__d462_13_d462_10_d13_10__alpha_1_0_beta_1_0_p_p_p__419e9f1(
    float **A, int A_extraOffset, const float *const *E, int E_extraOffset,
    const float *const *F, int F_extraOffset, unsigned numElements,
    unsigned *flags, void *streamPtr) {
  dim3 block(480, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream =
      (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d462_13_d462_10_d13_10__alpha_1_0_beta_1_0_p_p_p__419e9f1<<<
      grid, block, 0, stream>>>(A, A_extraOffset, E, E_extraOffset, F,
                                F_extraOffset, numElements, flags);
  CHECK_ERR;
}

__global__ void __launch_bounds__(224)
    kernel_sproduct_NT_NT_NT__d10_d21_22_10_d21_22__alpha_1_0_p_p_p__9329d0c(
        float **A, int A_extraOffset, const float *const *B, int B_extraOffset,
        const float *const *X, int X_extraOffset, unsigned numElements,
        unsigned *flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      {
        const float *const __restrict__ glb_B = &B[batchID][0 + B_extraOffset];
        float *const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float *const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float reg0[22] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                          0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                          0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__ __align__(8) float totalShrMem[472];
        float *localShrMem0 = &totalShrMem[472 * threadIdx.y];

        float *shrRegion0 = &localShrMem0[0];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 10) {
            shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
          }
        }

        float *shrRegion1 = &localShrMem0[10];
        // using ExtendedTensorLoader
        {
          shrRegion1[threadIdx.x + 0] = glb_X[threadIdx.x + 0];
          shrRegion1[threadIdx.x + 224] = glb_X[threadIdx.x + 224];
          if (threadIdx.x < 14) {
            shrRegion1[threadIdx.x + 448] = glb_X[threadIdx.x + 448];
          }
        }
        __syncthreads();
        /*
        This is the product kernel created from the following YaTeTo
        description: Description( alpha: 1.0 add: True result:
        IndexedTensorDescription(name=A, indices=kpm,
        memoryLayout=DenseMemoryLayout(shape=(21, 22, 10),
        bbox=BoundingBox(Range(0, 21), Range(0, 22), Range(0, 10)), stride=(1,
        21, 462), align=<yateto.arch.Architecture object at 0x7f0cf15675d0>),
        eqspp=dense(shape=(21, 22, 10), size=4620, ndim=3),
        is_compute_constant=False, is_temporary=False) leftTerm:
        IndexedTensorDescription(name=B, indices=m,
        memoryLayout=DenseMemoryLayout(shape=(10,), bbox=BoundingBox(Range(0,
        10)), stride=(1,), align=<yateto.arch.Architecture object at
        0x7f0cf15675d0>), eqspp=dense(shape=(10,), size=10, ndim=1),
        is_compute_constant=False, is_temporary=False) rightTerm:
        IndexedTensorDescription(name=X, indices=kp,
        memoryLayout=DenseMemoryLayout(shape=(21, 22), bbox=BoundingBox(Range(0,
        21), Range(0, 22)), stride=(1, 21), align=<yateto.arch.Architecture
        object at 0x7f0cf15675d0>), eqspp=dense(shape=(21, 22), size=462,
        ndim=2), is_compute_constant=False, is_temporary=False) isACsc: False
                isBCsc: False
                loopRanges: {'m': Range(0, 10), 'k': Range(0, 21), 'p': Range(0,
        22)}
        )
        */
        if (threadIdx.x < 210) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 21;
          rows_left -= row_offset_1 * 21;
          const int dim_offset_m = row_offset_1;
          const int row_offset_0 = rows_left;
          const int dim_offset_k = row_offset_0;
#pragma unroll
          for (int p = 0; p < 22; ++p) {
            reg0[p] = shrRegion0[dim_offset_m * 1] *
                      shrRegion1[dim_offset_k * 1 + p * 21];
          }
        }
        if (threadIdx.x < 210) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 21;
          rows_left -= row_offset_1 * 21;
          const int row_offset_0 = rows_left;
#pragma unroll
          for (int i = 0; i < 22; ++i) {
            glb_A[row_offset_0 * 1 + row_offset_1 * 462 + i * 21] =
                reg0[i] +
                1.0 * glb_A[row_offset_0 * 1 + row_offset_1 * 462 + i * 21];
          }
        }
      }
    }
  }
}
void sproduct_NT_NT_NT__d10_d21_22_10_d21_22__alpha_1_0_p_p_p__9329d0c(
    float **A, int A_extraOffset, const float *const *B, int B_extraOffset,
    const float *const *X, int X_extraOffset, unsigned numElements,
    unsigned *flags, void *streamPtr) {
  dim3 block(224, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream =
      (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sproduct_NT_NT_NT__d10_d21_22_10_d21_22__alpha_1_0_p_p_p__9329d0c<<<
      grid, block, 0, stream>>>(A, A_extraOffset, B, B_extraOffset, X,
                                X_extraOffset, numElements, flags);
  CHECK_ERR;
}

int main() {
  constexpr size_t num_els = 65053;
  float *A = new float[4620 * num_els]{0.f};
  float *B = new float[10 * num_els]{0.f};
  float *C = new float[1281 * num_els]{0.f};
  float *D = new float[1342 * num_els]{0.f};
  float *E = new float[6006 * num_els]{0.f};
  float *F = new float[130 * num_els]{0.f};
  float *X = new float[462 * num_els]{0.f};
  float *R1 = new float[4620 * num_els]{0.f};
  float *R2 = new float[4620 * num_els]{0.f};
  // float* Ri1 = new float[462 * num_els]{0.f};
  // float* Ri2 = new float[4620 * num_els]{0.f};
  // float* Ri1c = new float[462 * num_els]{0.f};
  // float* Ri2c = new float[4620 * num_els]{0.f};

  float *coreA = new float[4620];
  float *coreB = new float[10];
  float *coreC = new float[1281];
  float *coreD = new float[1342];
  float *coreE = new float[6006];
  float *coreF = new float[130];

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<int> distribution(1, 100);
  for (size_t i = 0; i < 4620; i++) {
    coreA[i] = distribution(gen);
  }
  for (size_t i = 0; i < 10; i++) {
    coreB[i] = distribution(gen);
  }
  for (size_t i = 0; i < 1281; i++) {
    coreC[i] = distribution(gen);
  }
  for (size_t i = 0; i < 1342; i++) {
    coreD[i] = distribution(gen);
  }
  for (size_t i = 0; i < 6006; i++) {
    coreE[i] = distribution(gen);
  }
  for (size_t i = 0; i < 130; i++) {
    coreF[i] = distribution(gen);
  }

  for (size_t i = 0; i < num_els; i++) {
    std::memcpy(&A[i * 4620], &coreA[0], 4620 * sizeof(float));
    std::memcpy(&B[i * 10], &coreB[0], 10 * sizeof(float));
    std::memcpy(&C[i * 1281], &coreC[0], 1281 * sizeof(float));
    std::memcpy(&D[i * 1342], &coreD[0], 1342 * sizeof(float));
    std::memcpy(&E[i * 6006], &coreE[0], 6006 * sizeof(float));
    std::memcpy(&F[i * 130], &coreF[0], 130 * sizeof(float));
  }

  float *A_dev = nullptr;
  float *B_dev = nullptr;
  float *C_dev = nullptr;
  float *D_dev = nullptr;
  float *E_dev = nullptr;
  float *F_dev = nullptr;
  float *X_dev = nullptr;

  float **A_dev_begins = new float *[num_els];
  float **B_dev_begins = new float *[num_els];
  float **C_dev_begins = new float *[num_els];
  float **D_dev_begins = new float *[num_els];
  float **E_dev_begins = new float *[num_els];
  float **F_dev_begins = new float *[num_els];
  float **X_dev_begins = new float *[num_els];

  float **A_dev_begins_dev = nullptr;
  float **B_dev_begins_dev = nullptr;
  float **C_dev_begins_dev = nullptr;
  float **D_dev_begins_dev = nullptr;
  float **E_dev_begins_dev = nullptr;
  float **F_dev_begins_dev = nullptr;
  float **X_dev_begins_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 4620 * num_els);
  CHECK_ERR;
  hipMalloc((void **)&B_dev, sizeof(float) * 10 * num_els);
  CHECK_ERR;
  hipMalloc((void **)&C_dev, sizeof(float) * 1281 * num_els);
  CHECK_ERR;
  hipMalloc((void **)&D_dev, sizeof(float) * 1342 * num_els);
  CHECK_ERR;
  hipMalloc((void **)&E_dev, sizeof(float) * 6006 * num_els);
  CHECK_ERR;
  hipMalloc((void **)&F_dev, sizeof(float) * 130 * num_els);
  CHECK_ERR;
  hipMalloc((void **)&X_dev, sizeof(float) * 462 * num_els);
  CHECK_ERR;

  hipMalloc((void **)&A_dev_begins_dev, sizeof(float *) * num_els);
  CHECK_ERR;
  hipMalloc((void **)&B_dev_begins_dev, sizeof(float *) * num_els);
  CHECK_ERR;
  hipMalloc((void **)&C_dev_begins_dev, sizeof(float *) * num_els);
  CHECK_ERR;
  hipMalloc((void **)&D_dev_begins_dev, sizeof(float *) * num_els);
  CHECK_ERR;
  hipMalloc((void **)&E_dev_begins_dev, sizeof(float *) * num_els);
  CHECK_ERR;
  hipMalloc((void **)&F_dev_begins_dev, sizeof(float *) * num_els);
  CHECK_ERR;
  hipMalloc((void **)&X_dev_begins_dev, sizeof(float *) * num_els);
  CHECK_ERR;

  hipDeviceSynchronize();
  CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4620 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)B_dev, (void *)B, sizeof(float) * 10 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)C_dev, (void *)C, sizeof(float) * 1281 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)D_dev, (void *)D, sizeof(float) * 1342 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)E_dev, (void *)E, sizeof(float) * 6006 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)F_dev, (void *)F, sizeof(float) * 130 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 462 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;

  for (size_t i = 0; i < num_els; i++) {
    A_dev_begins[i] = A_dev + i * 4620;
    B_dev_begins[i] = B_dev + i * 10;
    C_dev_begins[i] = C_dev + i * 1281;
    D_dev_begins[i] = D_dev + i * 1342;
    E_dev_begins[i] = E_dev + i * 6006;
    F_dev_begins[i] = F_dev + i * 130;
    X_dev_begins[i] = X_dev + i * 462;
  }

  hipMemcpy((void *)A_dev_begins_dev, (void *)A_dev_begins,
             sizeof(float *) * num_els, hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)B_dev_begins_dev, (void *)B_dev_begins,
             sizeof(float *) * num_els, hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)C_dev_begins_dev, (void *)C_dev_begins,
             sizeof(float *) * num_els, hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)D_dev_begins_dev, (void *)D_dev_begins,
             sizeof(float *) * num_els, hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)E_dev_begins_dev, (void *)E_dev_begins,
             sizeof(float *) * num_els, hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)F_dev_begins_dev, (void *)F_dev_begins,
             sizeof(float *) * num_els, hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)X_dev_begins_dev, (void *)X_dev_begins,
             sizeof(float *) * num_els, hipMemcpyHostToDevice);
  CHECK_ERR;

  sloopOverGEMM_NT_NT_NT__d21_61_d21_22_d61_22__alpha_1_0_beta_0_0_p_p_p__7610af5(
      C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els,
      nullptr, nullptr);
  CHECK_ERR;
  hipDeviceSynchronize();
  CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 462 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;

  std::cout << "Dimensions: " << 21 << ", " << 13 << ", " << 10 << ", " << 22
            << ", " << 61 << std::endl;

  float elapsedTimeT1 = 0.0;
  float elapsedTimeT2 = 0.0;
  float elapsedTimeT3 = 0.0;
  hipEvent_t startT1, stopT1;
  hipEvent_t startT2, stopT2;
  hipEvent_t startT3, stopT3;
  hipEventCreate(&startT1);
  CHECK_ERR;
  hipEventCreate(&stopT1);
  CHECK_ERR;
  hipEventRecord(startT1);
  CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d21_61_d21_22_d61_22__alpha_1_0_beta_0_0_p_p_p__7610af5(
      C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els,
      nullptr, nullptr);
  CHECK_ERR;
  hipEventRecord(stopT1);
  CHECK_ERR;
  hipEventSynchronize(stopT1);
  CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT1, startT1, stopT1);
  CHECK_ERR;
  // hipDeviceSynchronize(); CHECK_ERR;

  // hipMemcpy(Ri1, X_dev, sizeof(float) * 462 * num_els,
  // hipMemcpyDeviceToHost); CHECK_ERR;

  hipEventCreate(&startT2);
  CHECK_ERR;
  hipEventCreate(&stopT2);
  CHECK_ERR;
  hipEventRecord(startT2);
  CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d462_13_d462_10_d13_10__alpha_1_0_beta_1_0_p_p_p__419e9f1(
      A_dev_begins_dev, 0, E_dev_begins_dev, 0, F_dev_begins_dev, 0, num_els,
      nullptr, nullptr);
  CHECK_ERR;
  hipEventRecord(stopT2);
  CHECK_ERR;
  hipEventSynchronize(stopT2);
  CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT2, startT2, stopT2);
  CHECK_ERR;
  // hipDeviceSynchronize(); CHECK_ERR;

  // hipMemcpy(Ri2, A_dev, sizeof(float) * 4620 * num_els,
  // hipMemcpyDeviceToHost); CHECK_ERR;

  hipEventCreate(&startT3);
  CHECK_ERR;
  hipEventCreate(&stopT3);
  CHECK_ERR;
  hipEventRecord(startT3);
  CHECK_ERR;
  sproduct_NT_NT_NT__d10_d21_22_10_d21_22__alpha_1_0_p_p_p__9329d0c(
      A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els,
      nullptr, nullptr);
  CHECK_ERR;
  hipEventRecord(stopT3);
  CHECK_ERR;
  hipEventSynchronize(stopT3);
  CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT3, startT3, stopT3);
  CHECK_ERR;
  double elapsedTime = elapsedTimeT1 + elapsedTimeT2 + elapsedTimeT3;
  hipDeviceSynchronize();
  CHECK_ERR;

  std::cout << "Gemmforge Tensor Contraction took: " << elapsedTime << " ms"
            << std::endl;
  hipMemcpy(R1, A_dev, sizeof(float) * 4620 * num_els, hipMemcpyDeviceToHost);
  CHECK_ERR;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4620 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;

  double fp_per_el = 190344;
  double ls_per_el = 72036;
  double fp_unfused_per_el = 190344;
  double ls_unfused_per_el = 112692;
  fp_per_el *= num_els;
  ls_per_el *= num_els;
  fp_unfused_per_el *= num_els;
  ls_unfused_per_el *= num_els;
  std::cout << "Gemmforge Theoretical Fused Kernel GFLOPs/s: "
            << fp_per_el * 1e-6 / elapsedTime << std::endl;
  std::cout << "Operational Theoretical Fused intensity: "
            << fp_per_el / ls_per_el << std::endl;
  std::cout << "Gemmforge GFLOPs/s: " << fp_unfused_per_el * 1e-6 / elapsedTime
            << std::endl;
  std::cout << "Operational intensity: "
            << fp_unfused_per_el / ls_unfused_per_el << std::endl;
  double peakFLOPGiven = 29767.7;
  double peakBandwidthGiven = 760.08;

  if (peakFLOPGiven > 0.1 && peakBandwidthGiven) {
    double obtainable_peak =
        std::min(static_cast<double>(peakFLOPGiven),
                 static_cast<double>(peakBandwidthGiven *
                                     static_cast<double>(fp_per_el) /
                                     static_cast<double>(ls_per_el)));
    std::cout << 100.0 * (fp_per_el * 1e-6 / elapsedTime) / obtainable_peak
              << " % of roof w. respect to operational intensity achieved with "
                 "Gemmforge"
              << std::endl;
    // std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTime) / obtainable_peak <<
    // " % of roof w. respect to operational intensity achieved with cuTensor"
    // << std::endl;
    double obtainable_unfused_peak =
        std::min(static_cast<double>(peakFLOPGiven),
                 static_cast<double>(peakBandwidthGiven *
                                     static_cast<double>(fp_unfused_per_el) /
                                     static_cast<double>(ls_unfused_per_el)));
    std::cout << 100.0 * (fp_unfused_per_el * 1e-6 / elapsedTime) /
                     obtainable_unfused_peak
              << " % of roof w. respect to unfused operational intensity "
                 "achieved with Gemmforge"
              << std::endl;
    // std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTime) /
    // obtainable_unfused_peak << " % of roof w. respect to unfused operational
    // intensity achieved with cuTensor" << std::endl;
    double obtainable_unfused_peak_k1 = std::min(
        static_cast<double>(peakFLOPGiven),
        static_cast<double>(peakBandwidthGiven * static_cast<double>(56364) /
                            static_cast<double>(12340)));
    std::cout
        << 100.0 * (56364 * num_els * 1e-6 / elapsedTimeT1) /
               obtainable_unfused_peak_k1
        << " % of roof w. respect to Kernel1 intensity achieved with Gemmforge"
        << std::endl;
    double obtainable_unfused_peak_k2 = std::min(
        static_cast<double>(peakFLOPGiven),
        static_cast<double>(peakBandwidthGiven * static_cast<double>(124740) /
                            static_cast<double>(61504)));
    std::cout
        << 100.0 * (124740 * num_els * 1e-6 / elapsedTimeT2) /
               obtainable_unfused_peak_k2
        << " % of roof w. respect to Kernel2 intensity achieved with Gemmforge"
        << std::endl;
    double obtainable_unfused_peak_k3 = std::min(
        static_cast<double>(peakFLOPGiven),
        static_cast<double>(peakBandwidthGiven * static_cast<double>(9240) /
                            static_cast<double>(38848)));
    std::cout
        << 100.0 * (9240 * num_els * 1e-6 / elapsedTimeT3) /
               obtainable_unfused_peak_k3
        << " % of roof w. respect to Kernel3 intensity achieved with Gemmforge"
        << std::endl;
  }

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4620 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 462 * num_els,
             hipMemcpyHostToDevice);
  CHECK_ERR;

  if constexpr (!false) {
    hiptensorHandle_t *handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    hipEvent_t startCT1, stopCT1;
    hipEvent_t startCT2, stopCT2;
    hipEvent_t startCT3, stopCT3;
    hipEventCreate(&startCT1);
    CHECK_ERR;
    hipEventCreate(&stopCT1);
    CHECK_ERR;
    hipEventCreate(&startCT2);
    CHECK_ERR;
    hipEventCreate(&stopCT2);
    CHECK_ERR;
    hipEventCreate(&startCT3);
    CHECK_ERR;
    hipEventCreate(&stopCT3);
    CHECK_ERR;
    float elapsedTimeCT1 = 0.f;
    float elapsedTimeCT2 = 0.f;
    float elapsedTimeCT3 = 0.f;

    // Kernel 1
    std::cout << "cuTensor Kernel 1" << std::endl;
    {
      float alphaK1 = 1.0f;
      float betaK1 = 0.0f;
      float alphaK2 = 1.0f;
      float betaK2 = 1.0;
      float alphaK3 = 1.0f;
      float betaK3 = 1.0;

      std::vector<int> modeA{'k', 'p', 'm', 'b'};
      std::vector<int> modeB{'m', 'b'};
      std::vector<int> modeC{'k', 'q', 'b'};
      std::vector<int> modeD{'q', 'p', 'b'};
      std::vector<int> modeE{'k', 'p', 'l', 'b'};
      std::vector<int> modeF{'l', 'm', 'b'};
      std::vector<int> modeX{'k', 'p', 'b'};
      int nmodeA = modeA.size();
      int nmodeB = modeB.size();
      int nmodeC = modeC.size();
      int nmodeD = modeD.size();
      int nmodeE = modeE.size();
      int nmodeF = modeF.size();
      int nmodeX = modeX.size();

      std::unordered_map<int, int64_t> extent;
      // Derived from the kernel
      extent['k'] = 21;
      extent['l'] = 13;
      extent['m'] = 10;
      extent['p'] = 22;
      extent['q'] = 61;
      extent['b'] = num_els;

      std::vector<int64_t> extentA;
      for (auto mode : modeA) {
        extentA.push_back(extent[mode]);
      }
      std::vector<int64_t> extentB;
      for (auto mode : modeB) {
        extentB.push_back(extent[mode]);
      }
      std::vector<int64_t> extentC;
      for (auto mode : modeC) {
        extentC.push_back(extent[mode]);
      }
      std::vector<int64_t> extentD;
      for (auto mode : modeD) {
        extentD.push_back(extent[mode]);
      }
      std::vector<int64_t> extentE;
      for (auto mode : modeE) {
        extentE.push_back(extent[mode]);
      }
      std::vector<int64_t> extentF;
      for (auto mode : modeF) {
        extentF.push_back(extent[mode]);
      }
      std::vector<int64_t> extentX;
      for (auto mode : modeX) {
        extentX.push_back(extent[mode]);
      }

      hipDataType typeA = HIP_R_32F;
      hipDataType typeB = HIP_R_32F;
      hipDataType typeC = HIP_R_32F;
      hipDataType typeD = HIP_R_32F;
      hipDataType typeE = HIP_R_32F;
      hipDataType typeF = HIP_R_32F;
      hipDataType typeX = HIP_R_32F;
      hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

      hiptensorTensorDescriptor_t descA;
      HANDLE_ERROR(hiptensorInitTensorDescriptor(handle, &descA, nmodeA,
                                                extentA.data(), NULL, typeA,
                                                HIPTENSOR_OP_IDENTITY));

      hiptensorTensorDescriptor_t descB;
      HANDLE_ERROR(hiptensorInitTensorDescriptor(handle, &descB, nmodeB,
                                                extentB.data(), NULL, typeB,
                                                HIPTENSOR_OP_IDENTITY));

      hiptensorTensorDescriptor_t descC;
      HANDLE_ERROR(hiptensorInitTensorDescriptor(handle, &descC, nmodeC,
                                                extentC.data(), NULL, typeC,
                                                HIPTENSOR_OP_IDENTITY));

      hiptensorTensorDescriptor_t descD;
      HANDLE_ERROR(hiptensorInitTensorDescriptor(handle, &descD, nmodeD,
                                                extentD.data(), NULL, typeD,
                                                HIPTENSOR_OP_IDENTITY));

      hiptensorTensorDescriptor_t descE;
      HANDLE_ERROR(hiptensorInitTensorDescriptor(handle, &descE, nmodeE,
                                                extentE.data(), NULL, typeE,
                                                HIPTENSOR_OP_IDENTITY));

      hiptensorTensorDescriptor_t descF;
      HANDLE_ERROR(hiptensorInitTensorDescriptor(handle, &descF, nmodeF,
                                                extentF.data(), NULL, typeF,
                                                HIPTENSOR_OP_IDENTITY));

      hiptensorTensorDescriptor_t descX;
      HANDLE_ERROR(hiptensorInitTensorDescriptor(handle, &descX, nmodeX,
                                                extentX.data(), NULL, typeX,
                                                HIPTENSOR_OP_IDENTITY));

      uint32_t alignmentRequirementA;
      HANDLE_ERROR(cutensorGetAlignmentRequirement(handle, A_dev, &descA,
                                                   &alignmentRequirementA));

      uint32_t alignmentRequirementB;
      HANDLE_ERROR(cutensorGetAlignmentRequirement(handle, B_dev, &descB,
                                                   &alignmentRequirementB));

      uint32_t alignmentRequirementC;
      HANDLE_ERROR(cutensorGetAlignmentRequirement(handle, C_dev, &descC,
                                                   &alignmentRequirementC));

      uint32_t alignmentRequirementD;
      HANDLE_ERROR(cutensorGetAlignmentRequirement(handle, D_dev, &descD,
                                                   &alignmentRequirementD));

      uint32_t alignmentRequirementE;
      HANDLE_ERROR(cutensorGetAlignmentRequirement(handle, E_dev, &descE,
                                                   &alignmentRequirementE));

      uint32_t alignmentRequirementF;
      HANDLE_ERROR(cutensorGetAlignmentRequirement(handle, F_dev, &descF,
                                                   &alignmentRequirementF));

      uint32_t alignmentRequirementX;
      HANDLE_ERROR(cutensorGetAlignmentRequirement(handle, X_dev, &descX,
                                                   &alignmentRequirementX));

      cutensorContractionDescriptor_t desc1;
      HANDLE_ERROR(cutensorInitContractionDescriptor(
          handle, &desc1, &descC, modeC.data(), alignmentRequirementC, &descD,
          modeD.data(), alignmentRequirementD, &descX, modeX.data(),
          alignmentRequirementX, &descX, modeX.data(), alignmentRequirementX,
          typeCompute));

      cutensorContractionFind_t find1;
      HANDLE_ERROR(
          cutensorInitContractionFind(handle, &find1, HIPTENSOR_ALGO_DEFAULT));

      uint64_t worksize1 = 0;
      HANDLE_ERROR(cutensorContractionGetWorkspaceSize(
          handle, &desc1, &find1, CUTENSOR_WORKSPACE_RECOMMENDED, &worksize1));

      cutensorContractionDescriptor_t desc2;
      HANDLE_ERROR(cutensorInitContractionDescriptor(
          handle, &desc2, &descF, modeF.data(), alignmentRequirementF, &descE,
          modeE.data(), alignmentRequirementE, &descA, modeA.data(),
          alignmentRequirementA, &descA, modeA.data(), alignmentRequirementA,
          typeCompute));

      cutensorContractionFind_t find2;
      HANDLE_ERROR(
          cutensorInitContractionFind(handle, &find2, HIPTENSOR_ALGO_DEFAULT));

      uint64_t worksize2 = 0;
      HANDLE_ERROR(cutensorContractionGetWorkspaceSize(
          handle, &desc2, &find2, CUTENSOR_WORKSPACE_RECOMMENDED, &worksize2));

      cutensorContractionDescriptor_t desc3;
      HANDLE_ERROR(cutensorInitContractionDescriptor(
          handle, &desc3, &descB, modeB.data(), alignmentRequirementB, &descX,
          modeX.data(), alignmentRequirementX, &descA, modeA.data(),
          alignmentRequirementA, &descA, modeA.data(), alignmentRequirementA,
          typeCompute));

      cutensorContractionFind_t find3;
      HANDLE_ERROR(
          cutensorInitContractionFind(handle, &find3, HIPTENSOR_ALGO_DEFAULT));

      uint64_t worksize3 = 0;
      HANDLE_ERROR(cutensorContractionGetWorkspaceSize(
          handle, &desc3, &find3, CUTENSOR_WORKSPACE_RECOMMENDED, &worksize3));

      uint64_t maxWorkSize =
          std::max(std::max(worksize1, worksize2), worksize3);
      void *work = nullptr;
      if (maxWorkSize > 0) {
        if (hipSuccess != hipMalloc(&work, maxWorkSize)) {
          work = nullptr;
          maxWorkSize = 0;
          worksize1 = 0;
          worksize2 = 0;
          worksize3 = 0;
          hipGetLastError(); // Clear last error to save CHECK_ERR;
        } else {
          worksize1 = maxWorkSize;
          worksize2 = maxWorkSize;
          worksize3 = maxWorkSize;
        }
      }

      hiptensorContractionPlan_t plan1;
      HANDLE_ERROR(cutensorInitContractionPlan(handle, &plan1, &desc1, &find1,
                                               worksize1));

      hiptensorContractionPlan_t plan2;
      HANDLE_ERROR(cutensorInitContractionPlan(handle, &plan2, &desc2, &find2,
                                               worksize2));

      hiptensorContractionPlan_t plan3;
      HANDLE_ERROR(cutensorInitContractionPlan(handle, &plan3, &desc3, &find3,
                                               worksize3));

      hipDeviceSynchronize();
      CHECK_ERR;

      hipEventRecord(startCT1);
      CHECK_ERR;
      hiptensorContraction(handle, &plan1, (void *)&alphaK1, C_dev, D_dev,
                          (void *)&betaK1, X_dev, X_dev, work, worksize1, 0);
      hipEventRecord(stopCT1);
      CHECK_ERR;
      hipEventSynchronize(stopCT1);
      CHECK_ERR;
      hipEventElapsedTime(&elapsedTimeCT1, startCT1, stopCT1);
      CHECK_ERR;

      // hipDeviceSynchronize(); CHECK_ERR;
      // hipMemcpy(Ri1c, X_dev, sizeof(float) * 462 * num_els,
      // hipMemcpyDeviceToHost); CHECK_ERR;

      hipEventRecord(startCT2);
      CHECK_ERR;
      hiptensorContraction(handle, &plan2, (void *)&alphaK2, F_dev, E_dev,
                          (void *)&betaK2, A_dev, A_dev, work, worksize2, 0);
      hipEventRecord(stopCT2);
      CHECK_ERR;
      hipEventSynchronize(stopCT2);
      CHECK_ERR;
      hipEventElapsedTime(&elapsedTimeCT2, startCT2, stopCT2);
      CHECK_ERR;

      // hipDeviceSynchronize(); CHECK_ERR;
      // hipMemcpy(Ri2c, A_dev, sizeof(float) * 4620 * num_els,
      // hipMemcpyDeviceToHost); CHECK_ERR;

      hipEventRecord(startCT3);
      CHECK_ERR;
      hiptensorContraction(handle, &plan3, (void *)&alphaK3, B_dev, X_dev,
                          (void *)&betaK3, A_dev, A_dev, work, worksize3, 0);
      hipEventRecord(stopCT3);
      CHECK_ERR;
      hipEventSynchronize(stopCT3);
      CHECK_ERR;
      hipEventElapsedTime(&elapsedTimeCT3, startCT3, stopCT3);
      CHECK_ERR;

      hipDeviceSynchronize();
      CHECK_ERR;

      hipMemcpy(R2, A_dev, sizeof(float) * 4620 * num_els,
                 hipMemcpyDeviceToHost);
      CHECK_ERR;

      hipFree(work);
    }

    float elapsedTimeCuTensor =
        elapsedTimeCT1 + elapsedTimeCT2 + elapsedTimeCT2;
    if (peakFLOPGiven > 0.1 && peakBandwidthGiven) {
      double obtainable_peak =
          std::min(static_cast<double>(peakFLOPGiven),
                   static_cast<double>(peakBandwidthGiven *
                                       static_cast<double>(fp_per_el) /
                                       static_cast<double>(ls_per_el)));
      std::cout << 100.0 * (fp_per_el * 1e-6 / elapsedTimeCuTensor) /
                       obtainable_peak
                << " % of roof w. respect to operational intensity achieved "
                   "with cuTensor"
                << std::endl;

      double obtainable_unfused_peak =
          std::min(static_cast<double>(peakFLOPGiven),
                   static_cast<double>(peakBandwidthGiven *
                                       static_cast<double>(fp_unfused_per_el) /
                                       static_cast<double>(ls_unfused_per_el)));
      std::cout << 100.0 * (fp_unfused_per_el * 1e-6 / elapsedTimeCuTensor) /
                       obtainable_unfused_peak
                << " % of roof w. respect to unfused operational intensity "
                   "achieved with cuTensor"
                << std::endl;
    }

    /*
    bool i1results_wrong = false;
    for (size_t i = 0; i < 462 * num_els; i++){
      if (std::abs(Ri1[i] - Ri1c[i]) > 1.0f) {
        std::cout << "Intermediate Results 1 do not match, problem first at
    offset " << i << " :_(" << std::endl; i1results_wrong = true; break;
      }
    }
    if (!i1results_wrong){
      std::cout << "Gemmforge and cuTensor contraction intermediate results 1
    match! :)" << std::endl;
    }

    bool i2results_wrong = false;
    for (size_t i = 0; i < 4620 * num_els; i++){
      if (std::abs(Ri2[i] - Ri2c[i]) > 1.0f) {
        std::cout << "Intermediate Results 2 do not match, problem first at
    offset " << i << " :_(" << std::endl; i2results_wrong = true; break;
      }
    }
    if (!i2results_wrong){
      std::cout << "Gemmforge and cuTensor contraction intermediate results 2
    match! :)" << std::endl;
    }
    */

    bool results_wrong = false;
    for (size_t i = 0; i < 4620 * num_els; i++) {
      if (std::abs(R1[i] - R2[i]) > 5.0f) {
        std::cout << "Results do not match, problem first at offset " << i
                  << " :_(" << std::endl;
        results_wrong = true;
        break;
      }
    }
    if (!results_wrong) {
      std::cout << "Gemmforge and cuTensor contraction results match! :)"
                << std::endl;
    }
  }

  hipFree(A_dev_begins_dev);
  hipFree(B_dev_begins_dev);
  hipFree(C_dev_begins_dev);
  hipFree(D_dev_begins_dev);
  hipFree(E_dev_begins_dev);
  hipFree(F_dev_begins_dev);
  hipFree(X_dev_begins_dev);

  delete[] A;
  delete[] B;
  delete[] C;
  delete[] D;
  delete[] E;
  delete[] F;
  delete[] X;
  delete[] A_dev_begins;
  delete[] B_dev_begins;
  delete[] C_dev_begins;
  delete[] D_dev_begins;
  delete[] E_dev_begins;
  delete[] F_dev_begins;
  delete[] X_dev_begins;
  delete[] R1;
  delete[] R2;

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  hipFree(D_dev);
  hipFree(E_dev);
  hipFree(F_dev);
  hipFree(X_dev);

  delete[] coreA;
  delete[] coreB;
  delete[] coreC;
  delete[] coreD;
  delete[] coreE;
  delete[] coreF;

  return 0;
}
