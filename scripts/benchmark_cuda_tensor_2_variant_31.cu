#include "hip/hip_runtime.h"

#include <random>
#include <iostream>
#include <cstring>
#include <vector>
#include <unordered_map>

#include <hiptensor.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR(x)                                                  \
{                                                                        \
  const auto err = x;                                                    \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                   \
  {                                                                      \
    std::cout << "Error: " << hiptensorGetErrorString(err) << std::endl;  \
    std::cout << __FILE__ << " " << __LINE__ << std::endl;                      \
  }                                                                      \
}

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess) {
        std::cout << std::endl << File
                << ", line " << Line
                << ": " << hipGetErrorString(Error)
                << " (" << Error << ")"
                << std::endl;

        if (PrevLine > 0)
        std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
    }
    PrevFile = File;
    PrevLine = Line;
#endif
}

__global__ void 
__launch_bounds__(32)
 kernel_sloopOverGEMM_NT_NT_NT__d8_15_d8_13_d15_13__alpha_1_0_beta_0_0_p_p_p__63aca98(const float * const * C, int C_extraOffset, const float * const * D, int D_extraOffset, float ** X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=X,	  memoryLayout=DenseMemoryLayout(shape=(8, 13), bbox=BoundingBox(Range(0, 8), Range(0, 13)), stride=(1, 8), align=<yateto.arch.Architecture object at 0x7fbd191e6410>),	  eqspp=dense(shape=(8, 13), size=104, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=C,	  memoryLayout=DenseMemoryLayout(shape=(8, 15), bbox=BoundingBox(Range(0, 8), Range(0, 15)), stride=(1, 8), align=<yateto.arch.Architecture object at 0x7fbd191e6410>),	  eqspp=dense(shape=(8, 15), size=120, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=D,	  memoryLayout=DenseMemoryLayout(shape=(15, 13), bbox=BoundingBox(Range(0, 15), Range(0, 13)), stride=(1, 15), align=<yateto.arch.Architecture object at 0x7fbd191e6410>),	  eqspp=dense(shape=(15, 13), size=195, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=0.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 8), Range(0, 13), Range(0, 15))), 'matrix_a': DenseMatrix{name = C, num. rows = 8, num. columns = 15, leading dimension = 8, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 8, 15]}, 'matrix_b': DenseMatrix{name = D, num. rows = 15, num. columns = 13, leading dimension = 15, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 15, 13]}, 'matrix_c': DenseMatrix{name = X, num. rows = 8, num. columns = 13, leading dimension = 8, direction = DataFlowDirection.SINK, bbox = [0, 0, 8, 13]}, 'args': ['C, extraOffset_C', 'D, extraOffset_D', 'X, extraOffset_X', 'numElements', 'flags', 'streamPtr']})
      */
      {
    //('gemm', {'descr': Description(  result=TensorDescription(  name=X,	  memoryLayout=DenseMemoryLayout(shape=(8, 13), bbox=BoundingBox(Range(0, 8), Range(0, 13)), stride=(1, 8), align=<yateto.arch.Architecture object at 0x7fbd191e6410>),	  eqspp=dense(shape=(8, 13), size=104, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=C,	  memoryLayout=DenseMemoryLayout(shape=(8, 15), bbox=BoundingBox(Range(0, 8), Range(0, 15)), stride=(1, 8), align=<yateto.arch.Architecture object at 0x7fbd191e6410>),	  eqspp=dense(shape=(8, 15), size=120, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=D,	  memoryLayout=DenseMemoryLayout(shape=(15, 13), bbox=BoundingBox(Range(0, 15), Range(0, 13)), stride=(1, 15), align=<yateto.arch.Architecture object at 0x7fbd191e6410>),	  eqspp=dense(shape=(15, 13), size=195, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=0.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 8), Range(0, 13), Range(0, 15))), 'matrix_a': DenseMatrix{name = C, num. rows = 8, num. columns = 15, leading dimension = 8, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 8, 15]}, 'matrix_b': DenseMatrix{name = D, num. rows = 15, num. columns = 13, leading dimension = 15, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 15, 13]}, 'matrix_c': DenseMatrix{name = X, num. rows = 8, num. columns = 13, leading dimension = 8, direction = DataFlowDirection.SINK, bbox = [0, 0, 8, 13]}, 'args': ['C, extraOffset_C', 'D, extraOffset_D', 'X, extraOffset_X', 'numElements', 'flags', 'streamPtr']})
        const float * const __restrict__ glb_C = &C[batchID][0 + C_extraOffset];
        float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        const float * const __restrict__ glb_D = &D[batchID][0 + D_extraOffset];
        float reg0[13] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[195];
        float * localShrMem0 = &totalShrMem[195 * threadIdx.y];
        
        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 6; ++i) {
            shrRegion0[threadIdx.x + i * 32] = glb_D[threadIdx.x + i * 32];
          }
          if (threadIdx.x < 3) {
            shrRegion0[threadIdx.x + 192] = glb_D[threadIdx.x + 192];
          }
        }
        __syncwarp();
        if (threadIdx.x < 8) {
          float value;
        
          #pragma unroll
          for (int k = 0; k < 15; ++k) {
            value = glb_C[threadIdx.x + k * 8];
        
            #pragma unroll
            for (int n = 0; n < 13; ++n) {
              reg0[n] += value * shrRegion0[k + 15 * n];
            }
          }
        }
        if (threadIdx.x < 8) {
          #pragma unroll
          for (int n = 0; n < 13; ++n) {
            glb_X[threadIdx.x + 8 * n] = reg0[n];
          }
        }
        
      }
    }
  }
}
void sloopOverGEMM_NT_NT_NT__d8_15_d8_13_d15_13__alpha_1_0_beta_0_0_p_p_p__63aca98(const float * const * C, int C_extraOffset, const float * const * D, int D_extraOffset, float ** X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(32, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d8_15_d8_13_d15_13__alpha_1_0_beta_0_0_p_p_p__63aca98<<<grid,block,0,stream>>>(C, C_extraOffset, D, D_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}


__global__ void 
__launch_bounds__(128)
 kernel_sloopOverGEMM_NT_NT_NT__d104_46_d14_46_d104_14__alpha_1_0_beta_1_0_p_p_p__5ce9ba2(float ** A, int A_extraOffset, const float * const * E, int E_extraOffset, const float * const * F, int F_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      /*
      This is the LoG created from the following YaTeTo description:
      ('gemm', {'descr': Description(  result=TensorDescription(  name=A,	  memoryLayout=DenseMemoryLayout(shape=(104, 46), bbox=BoundingBox(Range(0, 104), Range(0, 46)), stride=(1, 104), align=<yateto.arch.Architecture object at 0x7fbd191d77d0>),	  eqspp=dense(shape=(104, 46), size=4784, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=E,	  memoryLayout=DenseMemoryLayout(shape=(104, 14), bbox=BoundingBox(Range(0, 104), Range(0, 14)), stride=(1, 104), align=<yateto.arch.Architecture object at 0x7fbd191d77d0>),	  eqspp=dense(shape=(104, 14), size=1456, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=F,	  memoryLayout=DenseMemoryLayout(shape=(14, 46), bbox=BoundingBox(Range(0, 14), Range(0, 46)), stride=(1, 14), align=<yateto.arch.Architecture object at 0x7fbd191d77d0>),	  eqspp=dense(shape=(14, 46), size=644, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=1.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 104), Range(0, 46), Range(0, 14))), 'matrix_a': DenseMatrix{name = E, num. rows = 104, num. columns = 14, leading dimension = 104, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 104, 14]}, 'matrix_b': DenseMatrix{name = F, num. rows = 14, num. columns = 46, leading dimension = 14, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 14, 46]}, 'matrix_c': DenseMatrix{name = A, num. rows = 104, num. columns = 46, leading dimension = 104, direction = DataFlowDirection.SINK, bbox = [0, 0, 104, 46]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F', 'A, extraOffset_A', 'numElements', 'flags', 'streamPtr']})
      */
      {
    //('gemm', {'descr': Description(  result=TensorDescription(  name=A,	  memoryLayout=DenseMemoryLayout(shape=(104, 46), bbox=BoundingBox(Range(0, 104), Range(0, 46)), stride=(1, 104), align=<yateto.arch.Architecture object at 0x7fbd191d77d0>),	  eqspp=dense(shape=(104, 46), size=4784, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  leftTerm=TensorDescription(  name=E,	  memoryLayout=DenseMemoryLayout(shape=(104, 14), bbox=BoundingBox(Range(0, 104), Range(0, 14)), stride=(1, 104), align=<yateto.arch.Architecture object at 0x7fbd191d77d0>),	  eqspp=dense(shape=(104, 14), size=1456, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  rightTerm=TensorDescription(  name=F,	  memoryLayout=DenseMemoryLayout(shape=(14, 46), bbox=BoundingBox(Range(0, 14), Range(0, 46)), stride=(1, 14), align=<yateto.arch.Architecture object at 0x7fbd191d77d0>),	  eqspp=dense(shape=(14, 46), size=644, ndim=2),	  is_compute_constant=False,	  is_temporary=False),	  transA=False,	  transB=False,	  alpha=1.0,	  beta=1.0,	  prefetchName=None,	  isACsc=False,	  isBCsc=False,	  alignedA=False,	  alignedC=False,	  mnk=(Range(0, 104), Range(0, 46), Range(0, 14))), 'matrix_a': DenseMatrix{name = E, num. rows = 104, num. columns = 14, leading dimension = 104, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 104, 14]}, 'matrix_b': DenseMatrix{name = F, num. rows = 14, num. columns = 46, leading dimension = 14, direction = DataFlowDirection.SOURCE, bbox = [0, 0, 14, 46]}, 'matrix_c': DenseMatrix{name = A, num. rows = 104, num. columns = 46, leading dimension = 104, direction = DataFlowDirection.SINK, bbox = [0, 0, 104, 46]}, 'args': ['E, extraOffset_E', 'F, extraOffset_F', 'A, extraOffset_A', 'numElements', 'flags', 'streamPtr']})
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_F = &F[batchID][0 + F_extraOffset];
        const float * const __restrict__ glb_E = &E[batchID][0 + E_extraOffset];
        float reg0[46] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[644];
        float * localShrMem0 = &totalShrMem[644 * threadIdx.y];
        
        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedPatchLoader
        {
          #pragma unroll
          for (int i = 0; i < 5; ++i) {
            shrRegion0[threadIdx.x + i * 128] = glb_F[threadIdx.x + i * 128];
          }
          if (threadIdx.x < 4) {
            shrRegion0[threadIdx.x + 640] = glb_F[threadIdx.x + 640];
          }
        }
        __syncthreads();
        if (threadIdx.x < 104) {
          float value;
        
          #pragma unroll
          for (int k = 0; k < 14; ++k) {
            value = glb_E[threadIdx.x + k * 104];
        
            #pragma unroll
            for (int n = 0; n < 46; ++n) {
              reg0[n] += value * shrRegion0[k + 14 * n];
            }
          }
        }
        if (threadIdx.x < 104) {
          #pragma unroll
          for (int n = 0; n < 46; ++n) {
            glb_A[threadIdx.x + 104 * n] = reg0[n] + glb_A[threadIdx.x + 104 * n];
          }
        }
        
      }
    }
  }
}
void sloopOverGEMM_NT_NT_NT__d104_46_d14_46_d104_14__alpha_1_0_beta_1_0_p_p_p__5ce9ba2(float ** A, int A_extraOffset, const float * const * E, int E_extraOffset, const float * const * F, int F_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(128, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sloopOverGEMM_NT_NT_NT__d104_46_d14_46_d104_14__alpha_1_0_beta_1_0_p_p_p__5ce9ba2<<<grid,block,0,stream>>>(A, A_extraOffset, E, E_extraOffset, F, F_extraOffset, numElements, flags);
  CHECK_ERR;
}


__global__ void 
__launch_bounds__(384)
 kernel_sproduct_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      {
        const float * const __restrict__ glb_B = &B[batchID][0 + B_extraOffset];
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float reg0[13] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[150];
        float * localShrMem0 = &totalShrMem[150 * threadIdx.y];

        float* shrRegion0 = &localShrMem0[0];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 46) {
            shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
          }
        }

        float* shrRegion1 = &localShrMem0[46];
        // using ExtendedTensorLoader
        {
          if (threadIdx.x < 104) {
            shrRegion1[threadIdx.x + 0] = glb_X[threadIdx.x + 0];
          }
        }
        __syncthreads();
        /*
        This is the product kernel created from the following YaTeTo description:
        Description(
        	alpha: 1.0
        	add: True
        	result: IndexedTensorDescription(name=A, indices=kpm, memoryLayout=DenseMemoryLayout(shape=(8, 13, 46), bbox=BoundingBox(Range(0, 8), Range(0, 13), Range(0, 46)), stride=(1, 8, 104), align=<yateto.arch.Architecture object at 0x7fbd1919e910>), eqspp=dense(shape=(8, 13, 46), size=4784, ndim=3), is_compute_constant=False, is_temporary=False)
        	leftTerm: IndexedTensorDescription(name=B, indices=m, memoryLayout=DenseMemoryLayout(shape=(46,), bbox=BoundingBox(Range(0, 46)), stride=(1,), align=<yateto.arch.Architecture object at 0x7fbd1919e910>), eqspp=dense(shape=(46,), size=46, ndim=1), is_compute_constant=False, is_temporary=False)
        	rightTerm: IndexedTensorDescription(name=X, indices=kp, memoryLayout=DenseMemoryLayout(shape=(8, 13), bbox=BoundingBox(Range(0, 8), Range(0, 13)), stride=(1, 8), align=<yateto.arch.Architecture object at 0x7fbd1919e910>), eqspp=dense(shape=(8, 13), size=104, ndim=2), is_compute_constant=False, is_temporary=False)
        	isACsc: False
        	isBCsc: False
        	loopRanges: {'m': Range(0, 46), 'p': Range(0, 13), 'k': Range(0, 8)}
        )
        */
        if (threadIdx.x < 368) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 8;
          rows_left -= row_offset_1 * 8;
          const int dim_offset_m = row_offset_1;
          const int row_offset_0 = rows_left;
          const int dim_offset_k = row_offset_0;
          #pragma unroll
          for (int p = 0; p < 13; ++p) {
            reg0[p] = shrRegion0[dim_offset_m * 1] * shrRegion1[dim_offset_k * 1 + p * 8];
          }
        }
        if (threadIdx.x < 368) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 8;
          rows_left -= row_offset_1 * 8;
          const int row_offset_0 = rows_left;
          #pragma unroll
          for (int i = 0; i < 13; ++i) {
            glb_A[row_offset_0 * 1 + row_offset_1 * 104 + i * 8] = reg0[i] + 1.0 * glb_A[row_offset_0 * 1 + row_offset_1 * 104 + i * 8];
          }
        }
      }
    }
  }
}
void sproduct_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(384, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sproduct_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}

__global__ void 
__launch_bounds__(384)
 kernel_sproduct2_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      {
        const float * const __restrict__ glb_B = &B[batchID][0 + B_extraOffset];
        float * const __restrict__ glb_A = &A[batchID][0 + A_extraOffset];
        const float * const __restrict__ glb_X = &X[batchID][0 + X_extraOffset];
        float reg0[13] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        __shared__  __align__(8) float totalShrMem[150];
        float * localShrMem0 = &totalShrMem[150 * threadIdx.y];

        float* shrRegion0 = &localShrMem0[0];
        float* shrRegion1 = &localShrMem0[46];
        // using ExtendedTensorLoader
        float* loadRegion = (threadIdx.x < 46)? shrRegion0 : shrRegion1;
        const float* globalregion = (threadIdx.x < 46)? glb_B : glb_X;
        int loadSubstract = (threadIdx.x < 46)? 0 : -46;
        {
          if (threadIdx.x < 150) {
            loadRegion[threadIdx.x + loadSubstract] = globalregion[threadIdx.x + loadSubstract];
          }
        }

        __syncthreads();
        /*
        This is the product kernel created from the following YaTeTo description:
        Description(
        	alpha: 1.0
        	add: True
        	result: IndexedTensorDescription(name=A, indices=kpm, memoryLayout=DenseMemoryLayout(shape=(8, 13, 46), bbox=BoundingBox(Range(0, 8), Range(0, 13), Range(0, 46)), stride=(1, 8, 104), align=<yateto.arch.Architecture object at 0x7fbd1919e910>), eqspp=dense(shape=(8, 13, 46), size=4784, ndim=3), is_compute_constant=False, is_temporary=False)
        	leftTerm: IndexedTensorDescription(name=B, indices=m, memoryLayout=DenseMemoryLayout(shape=(46,), bbox=BoundingBox(Range(0, 46)), stride=(1,), align=<yateto.arch.Architecture object at 0x7fbd1919e910>), eqspp=dense(shape=(46,), size=46, ndim=1), is_compute_constant=False, is_temporary=False)
        	rightTerm: IndexedTensorDescription(name=X, indices=kp, memoryLayout=DenseMemoryLayout(shape=(8, 13), bbox=BoundingBox(Range(0, 8), Range(0, 13)), stride=(1, 8), align=<yateto.arch.Architecture object at 0x7fbd1919e910>), eqspp=dense(shape=(8, 13), size=104, ndim=2), is_compute_constant=False, is_temporary=False)
        	isACsc: False
        	isBCsc: False
        	loopRanges: {'m': Range(0, 46), 'p': Range(0, 13), 'k': Range(0, 8)}
        )
        */
        if (threadIdx.x < 368) {
          int rows_left = threadIdx.x;
          const int row_offset_1 = rows_left / 8;
          rows_left -= row_offset_1 * 8;
          const int row_offset_0 = rows_left;
          const int dim_offset_m = row_offset_1;
          const int dim_offset_k = row_offset_0;
          #pragma unroll
          for (int p = 0; p < 13; ++p) {
            reg0[p] = shrRegion0[dim_offset_m * 1] * shrRegion1[dim_offset_k * 1 + p * 8];
          }

          #pragma unroll
          for (int i = 0; i < 13; ++i) {
            glb_A[row_offset_0 * 1 + row_offset_1 * 104 + i * 8] = reg0[i] + 1.0 * glb_A[row_offset_0 * 1 + row_offset_1 * 104 + i * 8];
          }
        }
      }
    }
  }
}
void sproduct2_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48(float ** A, int A_extraOffset, const float * const * B, int B_extraOffset, const float * const * X, int X_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(384, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sproduct2_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, X, X_extraOffset, numElements, flags);
  CHECK_ERR;
}

int main(){
  constexpr size_t num_els = 122554;
  float* A = new float[4784 * num_els]{0.f};
  float* B = new float[46 * num_els]{0.f};
  float* C = new float[120 * num_els]{0.f};
  float* D = new float[195 * num_els]{0.f};
  float* E = new float[1456 * num_els]{0.f};
  float* F = new float[644 * num_els]{0.f};
  float* X = new float[104 * num_els]{0.f};
  float* R1 = new float[4784 * num_els]{0.f};
  float* R2 = new float[4784 * num_els]{0.f};
  //float* Ri1 = new float[104 * num_els]{0.f};
  //float* Ri2 = new float[4784 * num_els]{0.f};
  //float* Ri1c = new float[104 * num_els]{0.f};
  //float* Ri2c = new float[4784 * num_els]{0.f};


  float* coreA = new float[4784];
  float* coreB = new float[46];
  float* coreC = new float[120];
  float* coreD = new float[195];
  float* coreE = new float[1456];
  float* coreF = new float[644];

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<int> distribution(1, 100);
  for (size_t i = 0; i < 4784; i++){
    coreA[i] = distribution(gen);
  }
  for (size_t i = 0; i < 46; i++){
    coreB[i] = distribution(gen);
  }
  for (size_t i = 0; i < 120; i++){
    coreC[i] = distribution(gen);
  }
  for (size_t i = 0; i < 195; i++){
    coreD[i] = distribution(gen);
  }
  for (size_t i = 0; i < 1456; i++){
    coreE[i] = distribution(gen);
  }
  for (size_t i = 0; i < 644; i++){
    coreF[i] = distribution(gen);
  }

  for (size_t i = 0; i < num_els; i++){
      std::memcpy(&A[i * 4784], &coreA[0], 4784 * sizeof(float));
      std::memcpy(&B[i * 46], &coreB[0], 46 * sizeof(float));
      std::memcpy(&C[i * 120], &coreC[0], 120 * sizeof(float));
      std::memcpy(&D[i * 195], &coreD[0], 195 * sizeof(float));
      std::memcpy(&E[i * 1456], &coreE[0], 1456 * sizeof(float));
      std::memcpy(&F[i * 644], &coreF[0], 644 * sizeof(float));
  }

  float* A_dev = nullptr;
  float* B_dev = nullptr;
  float* C_dev = nullptr;
  float* D_dev = nullptr;
  float* E_dev = nullptr;
  float* F_dev = nullptr;
  float* X_dev = nullptr;

  float** A_dev_begins = new float*[num_els];
  float** B_dev_begins = new float*[num_els];
  float** C_dev_begins = new float*[num_els];
  float** D_dev_begins = new float*[num_els];
  float** E_dev_begins = new float*[num_els];
  float** F_dev_begins = new float*[num_els];
  float** X_dev_begins = new float*[num_els];

  float** A_dev_begins_dev = nullptr;
  float** B_dev_begins_dev = nullptr;
  float** C_dev_begins_dev = nullptr;
  float** D_dev_begins_dev = nullptr;
  float** E_dev_begins_dev = nullptr;
  float** F_dev_begins_dev = nullptr;
  float** X_dev_begins_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 4784 * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev, sizeof(float) * 46 * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev, sizeof(float) * 120 * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev, sizeof(float) * 195 * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev, sizeof(float) * 1456 * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev, sizeof(float) * 644 * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev, sizeof(float) * 104 * num_els); CHECK_ERR;

  hipMalloc((void **)&A_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&B_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&C_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&D_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&E_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&F_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
  hipMalloc((void **)&X_dev_begins_dev, sizeof(float*) * num_els); CHECK_ERR;
 
  hipDeviceSynchronize(); CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev, (void *)B, sizeof(float) * 46 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev, (void *)C, sizeof(float) * 120 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev, (void *)D, sizeof(float) * 195 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev, (void *)E, sizeof(float) * 1456 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev, (void *)F, sizeof(float) * 644 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 104 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  for (size_t i = 0; i < num_els; i++){
    A_dev_begins[i] = A_dev + i * 4784;
    B_dev_begins[i] = B_dev + i * 46;
    C_dev_begins[i] = C_dev + i * 120;
    D_dev_begins[i] = D_dev + i * 195;
    E_dev_begins[i] = E_dev + i * 1456;
    F_dev_begins[i] = F_dev + i * 644;
    X_dev_begins[i] = X_dev + i * 104;
  }

  hipMemcpy((void *)A_dev_begins_dev, (void *)A_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dev_begins_dev, (void *)B_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C_dev_begins_dev, (void *)C_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)D_dev_begins_dev, (void *)D_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)E_dev_begins_dev, (void *)E_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)F_dev_begins_dev, (void *)F_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev_begins_dev, (void *)X_dev_begins, sizeof(float*) * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  sloopOverGEMM_NT_NT_NT__d8_15_d8_13_d15_13__alpha_1_0_beta_0_0_p_p_p__63aca98(C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipDeviceSynchronize(); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 104 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  std::cout << "Dimensions: " << 8 << ", " << 14 << ", " << 46 << ", " << 13 << ", " << 15 << ", " << 14 << std::endl;

  float elapsedTimeT1 = 0.0;
  float elapsedTimeT2 = 0.0;
  float elapsedTimeT3 = 0.0; 
  float elapsedTimeT4 = 0.0;
  hipEvent_t startT1, stopT1;
  hipEvent_t startT2, stopT2;
  hipEvent_t startT3, stopT3;
  hipEvent_t startT4, stopT4;
  hipEventCreate(&startT1); CHECK_ERR;
  hipEventCreate(&stopT1); CHECK_ERR;
  hipEventRecord(startT1); CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d8_15_d8_13_d15_13__alpha_1_0_beta_0_0_p_p_p__63aca98(C_dev_begins_dev, 0, D_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT1); CHECK_ERR;
  hipEventSynchronize(stopT1); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT1, startT1, stopT1); CHECK_ERR;
  //hipDeviceSynchronize(); CHECK_ERR;

  //hipMemcpy(Ri1, X_dev, sizeof(float) * 104 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  
  hipEventCreate(&startT2); CHECK_ERR;
  hipEventCreate(&stopT2); CHECK_ERR;
  hipEventRecord(startT2); CHECK_ERR;
  sloopOverGEMM_NT_NT_NT__d104_46_d14_46_d104_14__alpha_1_0_beta_1_0_p_p_p__5ce9ba2(A_dev_begins_dev, 0, E_dev_begins_dev, 0, F_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT2); CHECK_ERR;
  hipEventSynchronize(stopT2); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT2, startT2, stopT2); CHECK_ERR;
  //hipDeviceSynchronize(); CHECK_ERR;

  //hipMemcpy(Ri2, A_dev, sizeof(float) * 4784 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

  hipEventCreate(&startT3); CHECK_ERR;
  hipEventCreate(&stopT3); CHECK_ERR;
  hipEventRecord(startT3); CHECK_ERR;
  sproduct_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48(A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT3); CHECK_ERR;
  hipEventSynchronize(stopT3); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT3, startT3, stopT3); CHECK_ERR;
  double elapsedTime = elapsedTimeT1 + elapsedTimeT2 + elapsedTimeT3;
  hipDeviceSynchronize(); CHECK_ERR;
  
  std::cout << "Gemmforge Tensor Contraction took: " << elapsedTime << " ms" << std::endl; 
  hipMemcpy(R1, A_dev, sizeof(float) * 4784 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  hipEventCreate(&startT4); CHECK_ERR;
  hipEventCreate(&stopT4); CHECK_ERR;
  hipEventRecord(startT4); CHECK_ERR;
  sproduct2_NT_NT_NT__d46_d8_13_46_d8_13__alpha_1_0_p_p_p__7c7cd48(A_dev_begins_dev, 0, B_dev_begins_dev, 0, X_dev_begins_dev, 0, num_els, nullptr, nullptr); CHECK_ERR;
  hipEventRecord(stopT4); CHECK_ERR;
  hipEventSynchronize(stopT4); CHECK_ERR;
  hipEventElapsedTime(&elapsedTimeT4, startT4, stopT4); CHECK_ERR;
  hipDeviceSynchronize(); CHECK_ERR;
  


  double fp_per_el = 156208;
  double ls_per_el = 48116;
  double fp_unfused_per_el = 156208;
  double ls_unfused_per_el = 87220;
  fp_per_el *= num_els;
  ls_per_el *= num_els;
  fp_unfused_per_el *= num_els;
  ls_unfused_per_el *= num_els;
  std::cout << "Gemmforge Theoretical Fused Kernel GFLOPs/s: " << fp_per_el * 1e-6 / elapsedTime << std::endl;
  std::cout << "Operational Theoretical Fused intensity: " << fp_per_el / ls_per_el << std::endl;
  std::cout << "Gemmforge GFLOPs/s: " << fp_unfused_per_el * 1e-6 / elapsedTime << std::endl;
  std::cout << "Operational intensity: " << fp_unfused_per_el / ls_unfused_per_el << std::endl;
  double peakFLOPGiven = 29767.7;
  double peakBandwidthGiven = 760.08;

  if (peakFLOPGiven > 0.1 && peakBandwidthGiven){
    double obtainable_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_per_el) / static_cast<double>(ls_per_el)));
    std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTime) / obtainable_peak << " % of roof w. respect to operational intensity achieved with Gemmforge" << std::endl;
    //std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTime) / obtainable_peak << " % of roof w. respect to operational intensity achieved with cuTensor" << std::endl;
    double obtainable_unfused_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_unfused_per_el) / static_cast<double>(ls_unfused_per_el)));
    std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTime) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with Gemmforge" << std::endl;
    //std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTime) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with cuTensor" << std::endl;
    double obtainable_unfused_peak_k1 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(3120) / static_cast<double>(1676)));
    std::cout << 100.0*(3120 * num_els  * 1e-6 / elapsedTimeT1) / obtainable_unfused_peak_k1 << " % of roof w. respect to Kernel1 intensity achieved with Gemmforge" << std::endl;
    double obtainable_unfused_peak_k2 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(138736) / static_cast<double>(46672)));
    std::cout << 100.0*(138736 * num_els  * 1e-6 / elapsedTimeT2) / obtainable_unfused_peak_k2 << " % of roof w. respect to Kernel2 intensity achieved with Gemmforge" << std::endl;
    double obtainable_unfused_peak_k3 = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(14352) / static_cast<double>(38872)));
    std::cout << 100.0*(14352 * num_els * 1e-6 / elapsedTimeT3) / obtainable_unfused_peak_k3 << " % of roof w. respect to Kernel3 intensity achieved with Gemmforge" << std::endl;
    std::cout << 100.0*(14352 * num_els * 1e-6 / elapsedTimeT4) / obtainable_unfused_peak_k3 << " % of roof w. respect to Kernel3 (Optimization Idea) intensity achieved with Gemmforge" << std::endl;
  }

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 4784 * num_els, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)X_dev, (void *)X, sizeof(float) * 104 * num_els, hipMemcpyHostToDevice); CHECK_ERR;

  if constexpr (!false){
  hiptensorHandle_t* handle;
  HANDLE_ERROR(hiptensorCreate(&handle));

  hipEvent_t startCT1, stopCT1;
  hipEvent_t startCT2, stopCT2;
  hipEvent_t startCT3, stopCT3;
  hipEventCreate(&startCT1); CHECK_ERR;
  hipEventCreate(&stopCT1); CHECK_ERR;
  hipEventCreate(&startCT2); CHECK_ERR;
  hipEventCreate(&stopCT2); CHECK_ERR;
  hipEventCreate(&startCT3); CHECK_ERR;
  hipEventCreate(&stopCT3); CHECK_ERR;
  float elapsedTimeCT1 = 0.f;
  float elapsedTimeCT2 = 0.f;
  float elapsedTimeCT3 = 0.f;

  // Kernel 1
  std::cout << "cuTensor Kernel 1" << std::endl;
  {
    float alphaK1 = 1.0f;
    float betaK1 = 0.0f;
    float alphaK2 = 1.0f;
    float betaK2 = 1.0;
    float alphaK3 = 1.0f;
    float betaK3 = 1.0;

    std::vector<int> modeA{'k', 'p', 'm', 'b'};
    std::vector<int> modeB{'m', 'b'};
    std::vector<int> modeC{'k', 'q', 'b'};
    std::vector<int> modeD{'q', 'p', 'b'};
    std::vector<int> modeE{'k', 'p', 'l', 'b'};
    std::vector<int> modeF{'l', 'm', 'b'};
    std::vector<int> modeX{'k', 'p', 'b'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();
    int nmodeD = modeD.size();
    int nmodeE = modeE.size();
    int nmodeF = modeF.size();
    int nmodeX = modeX.size();

    std::unordered_map<int, int64_t> extent;
    // Derived from the kernel
    extent['k'] = 8;
    extent['l'] = 14;
    extent['m'] = 46;
    extent['p'] = 13;
    extent['q'] = 15;
    extent['b'] = num_els;

    std::vector<int64_t> extentA;
    for (auto mode : modeA) {
        extentA.push_back(extent[mode]);
    }
    std::vector<int64_t> extentB;
    for (auto mode : modeB) {
        extentB.push_back(extent[mode]);
    }
    std::vector<int64_t> extentC;
    for (auto mode : modeC) {
        extentC.push_back(extent[mode]);
    }
    std::vector<int64_t> extentD;
    for (auto mode : modeD) {
        extentD.push_back(extent[mode]);
    }
    std::vector<int64_t> extentE;
    for (auto mode : modeE) {
        extentE.push_back(extent[mode]);
    }
    std::vector<int64_t> extentF;
    for (auto mode : modeF) {
        extentF.push_back(extent[mode]);
    }
    std::vector<int64_t> extentX;
    for (auto mode : modeX) {
        extentX.push_back(extent[mode]);
    }
    
    hipDataType typeA = HIP_R_32F;
    hipDataType typeB = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hipDataType typeD = HIP_R_32F;
    hipDataType typeE = HIP_R_32F;
    hipDataType typeF = HIP_R_32F;
    hipDataType typeX = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descA,
                    nmodeA,
                    extentA.data(),
                    NULL,
                    typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descB,
                    nmodeB,
                    extentB.data(),
                    NULL,
                    typeB, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descC,
                    nmodeC,
                    extentC.data(),
                    NULL,
                    typeC, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descD;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descD,
                    nmodeD,
                    extentD.data(),
                    NULL,
                    typeD, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descE;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(handle,
                    &descE,
                    nmodeE,
                    extentE.data(),
                    NULL,
                    typeE, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descF;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descF,
                    nmodeF,
                    extentF.data(),
                    NULL,
                    typeF, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descX;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( handle,
                    &descX,
                    nmodeX,
                    extentX.data(),
                    NULL,
                    typeX, HIPTENSOR_OP_IDENTITY));


    uint32_t alignmentRequirementA;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    A_dev,
                    &descA,
                    &alignmentRequirementA));

    uint32_t alignmentRequirementB;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    B_dev,
                    &descB,
                    &alignmentRequirementB));

    uint32_t alignmentRequirementC;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    C_dev,
                    &descC, 
                    &alignmentRequirementC));

    uint32_t alignmentRequirementD;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    D_dev,
                    &descD,
                    &alignmentRequirementD));

    uint32_t alignmentRequirementE;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    E_dev,
                    &descE,
                    &alignmentRequirementE));

    uint32_t alignmentRequirementF;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    F_dev,
                    &descF, 
                    &alignmentRequirementF));

    uint32_t alignmentRequirementX;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                    X_dev,
                    &descX, 
                    &alignmentRequirementX));

    cutensorContractionDescriptor_t desc1;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc1,
                  &descC, modeC.data(), alignmentRequirementC,
                  &descD, modeD.data(), alignmentRequirementD,
                  &descX, modeX.data(), alignmentRequirementX,
                  &descX, modeX.data(), alignmentRequirementX,
                  typeCompute));

    cutensorContractionFind_t find1;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find1, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize1 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc1,
                 &find1,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize1));

    cutensorContractionDescriptor_t desc2;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc2,
                  &descF, modeF.data(), alignmentRequirementF,
                  &descE, modeE.data(), alignmentRequirementE,
                  &descA, modeA.data(), alignmentRequirementA,
                  &descA, modeA.data(), alignmentRequirementA,
                  typeCompute));

    cutensorContractionFind_t find2;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find2, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize2 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc2,
                 &find2,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize2));


    cutensorContractionDescriptor_t desc3;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                  &desc3,
                  &descB, modeB.data(), alignmentRequirementB,
                  &descX, modeX.data(), alignmentRequirementX,
                  &descA, modeA.data(), alignmentRequirementA,
                  &descA, modeA.data(), alignmentRequirementA,
                  typeCompute));

    cutensorContractionFind_t find3;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find3, 
                 HIPTENSOR_ALGO_DEFAULT));

    uint64_t worksize3 = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc3,
                 &find3,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize3));

    uint64_t maxWorkSize = std::max(std::max(worksize1, worksize2), worksize3);
    void *work = nullptr;
    if (maxWorkSize > 0)
    {
        if (hipSuccess != hipMalloc(&work, maxWorkSize))
        {
            work = nullptr;
            maxWorkSize = 0;
            worksize1 = 0;
            worksize2 = 0;
            worksize3 = 0;
            hipGetLastError(); // Clear last error to save CHECK_ERR;
        } else {
            worksize1 = maxWorkSize;
            worksize2 = maxWorkSize;
            worksize3 = maxWorkSize;
        }
    }


    hiptensorContractionPlan_t plan1;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan1,
                 &desc1,
                 &find1,
                 worksize1));

    hiptensorContractionPlan_t plan2;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan2,
                 &desc2,
                 &find2,
                 worksize2));

    hiptensorContractionPlan_t plan3;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan3,
                 &desc3,
                 &find3,
                 worksize3));

    hipDeviceSynchronize(); CHECK_ERR;

    hipEventRecord(startCT1); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan1,
                              (void*) &alphaK1, C_dev, D_dev,
                              (void*) &betaK1,  X_dev, X_dev, 
                              work, worksize1, 0);
    hipEventRecord(stopCT1); CHECK_ERR;
    hipEventSynchronize(stopCT1); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT1, startCT1, stopCT1); CHECK_ERR;

    //hipDeviceSynchronize(); CHECK_ERR;
    //hipMemcpy(Ri1c, X_dev, sizeof(float) * 104 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipEventRecord(startCT2); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan2,
                              (void*) &alphaK2, F_dev, E_dev,
                              (void*) &betaK2,  A_dev, A_dev, 
                              work, worksize2, 0);
    hipEventRecord(stopCT2); CHECK_ERR;
    hipEventSynchronize(stopCT2); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT2, startCT2, stopCT2); CHECK_ERR;

    //hipDeviceSynchronize(); CHECK_ERR;
    //hipMemcpy(Ri2c, A_dev, sizeof(float) * 4784 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipEventRecord(startCT3); CHECK_ERR;
    hiptensorContraction(handle,
                              &plan3,
                              (void*) &alphaK3, B_dev, X_dev,
                              (void*) &betaK3,  A_dev, A_dev, 
                              work, worksize3, 0);
    hipEventRecord(stopCT3); CHECK_ERR;
    hipEventSynchronize(stopCT3); CHECK_ERR;
    hipEventElapsedTime(&elapsedTimeCT3, startCT3, stopCT3); CHECK_ERR;

    hipDeviceSynchronize(); CHECK_ERR;
    
    hipMemcpy(R2, A_dev, sizeof(float) * 4784 * num_els, hipMemcpyDeviceToHost); CHECK_ERR;

    hipFree(work);
  }

  float elapsedTimeCuTensor = elapsedTimeCT1 + elapsedTimeCT2 + elapsedTimeCT2;
  if (peakFLOPGiven > 0.1 && peakBandwidthGiven){
    double obtainable_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_per_el) / static_cast<double>(ls_per_el)));
    std::cout << 100.0*(fp_per_el * 1e-6 / elapsedTimeCuTensor) / obtainable_peak << " % of roof w. respect to operational intensity achieved with cuTensor" << std::endl;

    double obtainable_unfused_peak = std::min(static_cast<double>(peakFLOPGiven), static_cast<double>(peakBandwidthGiven * static_cast<double>(fp_unfused_per_el) / static_cast<double>(ls_unfused_per_el)));
    std::cout << 100.0*(fp_unfused_per_el * 1e-6 / elapsedTimeCuTensor) / obtainable_unfused_peak << " % of roof w. respect to unfused operational intensity achieved with cuTensor" << std::endl;
  }

  /*
  bool i1results_wrong = false;
  for (size_t i = 0; i < 104 * num_els; i++){
    if (std::abs(Ri1[i] - Ri1c[i]) > 1.0f) {
      std::cout << "Intermediate Results 1 do not match, problem first at offset " << i << " :_(" << std::endl;
      i1results_wrong = true;
      break;
    }
  }
  if (!i1results_wrong){
    std::cout << "Gemmforge and cuTensor contraction intermediate results 1 match! :)" << std::endl;
  }
  
  bool i2results_wrong = false;
  for (size_t i = 0; i < 4784 * num_els; i++){
    if (std::abs(Ri2[i] - Ri2c[i]) > 1.0f) {
      std::cout << "Intermediate Results 2 do not match, problem first at offset " << i << " :_(" << std::endl;
      i2results_wrong = true;
      break;
    }
  }
  if (!i2results_wrong){
    std::cout << "Gemmforge and cuTensor contraction intermediate results 2 match! :)" << std::endl;
  }
  */

  bool results_wrong = false;
  for (size_t i = 0; i < 4784 * num_els; i++){
    if (std::abs(R1[i] - R2[i]) > 5.0f) {
      std::cout << "Results do not match, problem first at offset " << i << " :_(" << std::endl;
      results_wrong = true;
      break;
    }
  }
  if (!results_wrong){
    std::cout << "Gemmforge and cuTensor contraction results match! :)" << std::endl;
  }
  }

  hipFree(A_dev_begins_dev);
  hipFree(B_dev_begins_dev);
  hipFree(C_dev_begins_dev);
  hipFree(D_dev_begins_dev);
  hipFree(E_dev_begins_dev);
  hipFree(F_dev_begins_dev);
  hipFree(X_dev_begins_dev);

  delete[] A;
  delete[] B;
  delete[] C;
  delete[] D;
  delete[] E;
  delete[] F;
  delete[] X;
  delete[] A_dev_begins;
  delete[] B_dev_begins;
  delete[] C_dev_begins;
  delete[] D_dev_begins;
  delete[] E_dev_begins;
  delete[] F_dev_begins;
  delete[] X_dev_begins;
  delete[] R1;
  delete[] R2;

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  hipFree(D_dev);
  hipFree(E_dev);
  hipFree(F_dev);
  hipFree(X_dev);

  delete[] coreA;
  delete[] coreB;
  delete[] coreC;
  delete[] coreD;
  delete[] coreE;
  delete[] coreF;

  return 0;
}

