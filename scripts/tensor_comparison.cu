#include "hip/hip_runtime.h"
// Example Tensor Contraction
// Number 1, Matrix Multiplication as a Tensor:
// C[ij] = A[ik] * B[kj]
// Number 2, 3D to 3D Tensors
// C[nko] = A[mko] * B[nmo]

#include <hipblas.h>
#include <iostream>
#include <random>
#include <iomanip>
#include <hiptensor.h>

#define CHECK_ERR checkErr(__FILE__, __LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line)
{
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess)
    {
        std::cout << std::endl
                  << File
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error)
                  << " (" << Error << ")"
                  << std::endl;

        if (PrevLine > 0)
            std::cout << "Previous CUDA call:" << std::endl
                      << PrevFile << ", line " << PrevLine << std::endl;
        throw;
    }
    PrevFile = File;
    PrevLine = Line;
#endif
}

void transposeMatrix(float *inputMatrix, float *outputMatrix, int numRows, int numCols)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Transpose matrix using cuBLAS geam function
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, numCols, numRows, &alpha, inputMatrix, numRows, &beta, NULL, numCols, outputMatrix, numCols);

    hipblasDestroy(handle);
}

bool compareMatrices(const float *matrixA, const float *matrixB, int numRows, int numCols, float tolerance)
{
    for (int row = 0; row < numRows; row++)
    {
        for (int col = 0; col < numCols; col++)
        {
            float diff = std::fabs(matrixA[col * numRows + row] - matrixB[col * numRows + row]);
            if (diff > tolerance)
            {
                return false;
            }
        }
    }
    return true;
}

void matrixMultiplyCPU(const float *matrixA, const float *matrixB, float *matrixC, int numRows, int numCols, int sharedDim)
{
    for (int col = 0; col < numCols; col++)
    {
        for (int row = 0; row < numRows; row++)
        {
            float sum = 0.0f;
            for (int k = 0; k < sharedDim; k++)
            {
                sum += matrixA[k * numRows + row] * matrixB[k + numRows * col];
            }
            matrixC[col * numRows + row] = sum;
        }
    }
}

// Print a matrix
void printMatrix(const float *matrix, int numRows, int numCols)
{
    std::cout << std::setprecision(4);
    for (int row = 0; row < numRows; row++)
    {
        for (int col = 0; col < numCols; col++)
        {
            std::cout << matrix[col * numRows + row] << "\t";
        }
        std::cout << std::endl;
    }
}

__global__ void
    __launch_bounds__(32)
        gemm(const float *A, const int offsetBetweenElementsAx, const int offsetBetweenElementsAy,
             const float *B, const int offsetBetweenElementsBx, const int offsetBetweenElementsBy,
             float *C, const int offsetBetweenElementsCx, const int offsetBetweenElementsCy,
             unsigned numElements, unsigned *flags)
{
    unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
    if (batchID < numElements)
    {
        bool isFlagsProvided = (flags != nullptr);
        bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
        if (allowed)
        {
            const float *const __restrict__ glb_A = &A[batchID * 64 + 0];
            const float *const __restrict__ glb_B = &B[batchID * 64 + 0];
            float *const __restrict__ glb_C = &C[batchID * 64 + 0];
            float reg0[8] = {0.0f};
            __shared__ __align__(8) float totalShrMem[64];
            float *localShrMem0 = &totalShrMem[64 * threadIdx.y];

            float *shrRegion0 = &localShrMem0[0];
            /*
                assertions that offsets are either matches 1 or a combination of dimensions like dimA or dimB or dimC*dimB etc.
            */
            // using ExtendedPatchLoader
            if (threadIdx.x < 8)
            {
#pragma unroll
                for (int i = 0; i < 8; ++i)
                {
                    shrRegion0[threadIdx.x + i * 8] = glb_B[(threadIdx.x * offsetBetweenElementsBy) + (i * offsetBetweenElementsBx)];
                }
            }
            __syncwarp();
            if (threadIdx.x < 8)
            {
                float value;

#pragma unroll
                for (int k = 0; k < 8; ++k)
                {
                    value = glb_A[(threadIdx.x * offsetBetweenElementsAy) + (k * offsetBetweenElementsAx)];

#pragma unroll
                    for (int n = 0; n < 8; ++n)
                    {
                        reg0[n] += value * shrRegion0[k + 8 * n];
                    }
                }
            }
            if (threadIdx.x < 8)
            {
#pragma unroll
                for (int n = 0; n < 8; ++n)
                {
                    glb_C[(threadIdx.x * offsetBetweenElementsCy) + (n * offsetBetweenElementsCx)] = reg0[n];
                }
            }
        }
    }
}

void gemm_launcher(const float *A, const int offsetBetweenElementsAx, const int offsetBetweenElementsAy,
                   const float *B, const int offsetBetweenElementsBx, const int offsetBetweenElementsBy,
                   float *C, const int offsetBetweenElementsCx, const int offsetBetweenElementsCy,
                   unsigned numElements, unsigned *flags, void *streamPtr)
{
    dim3 block(32, 1, 1);
    dim3 grid((numElements + 1 - 1) / 1, 1, 1);
    hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
    gemm<<<grid, block, 0, stream>>>(
        A, offsetBetweenElementsAx, offsetBetweenElementsAy,
        B, offsetBetweenElementsBx, offsetBetweenElementsBy,
        C, offsetBetweenElementsCx, offsetBetweenElementsCy,
        numElements, flags);
    CHECK_ERR;
}

int main()
{
    constexpr int numRows = 8;
    constexpr int numCols = 8;
    constexpr int sharedDim = 8;
    constexpr int numElements = numRows * numCols;
    constexpr size_t matrixSize = numElements * sizeof(float);
    constexpr float tolerance = 1e-6; // Tolerance for floating-point comparison

    // Initialize the column-major matrices A, B, and C
    float *matrixA = new float[numRows * numCols];
    float *matrixB = new float[numRows * numCols];
    float *matrixC_CPU = new float[numRows * numCols];
    float *matrixC_GPU = new float[numRows * numCols];
    float *matrixC_cuTensor = new float[numRows * numCols];
    float *matrixC_GPU_MyGemm = new float[numRows * numCols];

    // Random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    // Initialize matrices A and B with random values
    for (int i = 0; i < numRows * numCols; i++)
    {
        matrixA[i] = dist(gen);
        matrixB[i] = dist(gen);
        matrixC_CPU[i] = 0.f;
        matrixC_GPU[i] = 0.f;
        matrixC_cuTensor[i] = 0.f;
        matrixC_GPU_MyGemm[i] = 0.f;
    }

    // Mat mul CPU
    {
        matrixMultiplyCPU(matrixA, matrixB, matrixC_CPU, numRows, numCols, numRows);
    }

    // Mat mul GPU with cuBLAS
    {
        // Transpose matrices A, B, and C to row-major format
        float *deviceMatrixA;
        float *deviceMatrixB;
        float *deviceMatrixC;

        hipMalloc((void **)&deviceMatrixA, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixB, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixC, matrixSize);
        CHECK_ERR;

        // Multiply matrices A, B, and C using cuBLAS
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        constexpr float alpha = 1.0f;
        constexpr float beta = 0.0f;

        // Copy matrices A and B from the CPU to the GPU
        hipMemcpy(deviceMatrixA, matrixA, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(deviceMatrixB, matrixB, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;

        // Perform matrix multiplication C = A * B using cuBLAS
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, numRows, numCols, numRows, &alpha, deviceMatrixA, numRows, deviceMatrixB, numRows, &beta, deviceMatrixC, numRows);
        CHECK_ERR;

        // Copy the result matrix C from the GPU to the CPU
        hipMemcpy(matrixC_GPU, deviceMatrixC, matrixSize, hipMemcpyDeviceToHost);
        CHECK_ERR;

        // Compare results with CPU matrix multiplication

        bool resultsMatch = compareMatrices(matrixC_CPU, matrixC_GPU, numRows, numCols, tolerance);

        if (resultsMatch)
        {
            std::cout << "Results match! (CPU-cuBLAS)" << std::endl;
        }
        else
        {
            std::cout << "Results do not match! (CPU-cuBLAS)" << std::endl;
        }

        // Clean up resources
        hipFree(deviceMatrixA);
        CHECK_ERR;
        hipFree(deviceMatrixB);
        CHECK_ERR;
        hipFree(deviceMatrixC);
        CHECK_ERR;

        hipblasDestroy(handle);
    }

    // Mat mul with my general Gemm Implementation
    {
        // Transpose matrices A, B, and C to row-major format
        float *deviceMatrixA;
        float *deviceMatrixB;
        float *deviceMatrixC;

        hipMalloc((void **)&deviceMatrixA, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixB, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixC, matrixSize);
        CHECK_ERR;

        // Multiply matrices A, B, and C using cuBLAS
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        constexpr float alpha = 1.0f;
        constexpr float beta = 0.0f;

        // Copy matrices A and B from the CPU to the GPU
        hipMemcpy(deviceMatrixA, matrixA, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(deviceMatrixB, matrixB, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;

        // Perform matrix multiplication C = A * B using cuBLAS
        gemm_launcher(deviceMatrixA, numCols, 1, deviceMatrixB, numCols, 1, deviceMatrixC, numCols, 1, 1, nullptr, nullptr);
        CHECK_ERR;

        // Copy the result matrix C from the GPU to the CPU
        hipMemcpy(matrixC_GPU_MyGemm, deviceMatrixC, matrixSize, hipMemcpyDeviceToHost);
        CHECK_ERR;

        // Compare results with CPU matrix multiplication

        bool resultsMatch = compareMatrices(matrixC_CPU, matrixC_GPU_MyGemm, numRows, numCols, tolerance);

        if (resultsMatch)
        {
            std::cout << "Results match! (CPU-OffsetGemm)" << std::endl;
        }
        else
        {
            std::cout << "Results do not match! (CPU-OffsetGemm)" << std::endl;
        }

        // Clean up resources
        hipFree(deviceMatrixA);
        CHECK_ERR;
        hipFree(deviceMatrixB);
        CHECK_ERR;
        hipFree(deviceMatrixC);
        CHECK_ERR;

        hipblasDestroy(handle);
    }

    // Matrix multiplication with cuTensor
    {
        // cuTensor initialization
        hiptensorHandle_t handle;
        cutensorInit(&handle);
        CHECK_ERR;

        // Create vector of modes
        std::vector<int> modeA{'i', 'k'};
        std::vector<int> modeB{'k', 'j'};
        std::vector<int> modeC{'i', 'j'};
        int nmodeA = modeA.size();
        int nmodeB = modeB.size();
        int nmodeC = modeC.size();

        // Tensor descriptors
        hiptensorTensorDescriptor_t descA, descB, descC;
        const int64_t *extentA = new int64_t[2]{numRows, sharedDim};
        const int64_t *extentB = new int64_t[2]{sharedDim, numCols};
        const int64_t *extentC = new int64_t[2]{numRows, numCols};

        // size_t elementsA = numRows * sharedDim;
        // size_t elementsB = sharedDim * numCols;
        // size_t elementsC = numRows * numCols;

        float *deviceMatrixA;
        float *deviceMatrixB;
        float *deviceMatrixC;

        hipMalloc((void **)&deviceMatrixA, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixB, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixC, matrixSize);
        CHECK_ERR;

        constexpr float alpha = 1.0f;
        constexpr float beta = 0.0f;

        // Copy matrices A and B from the CPU to the GPU
        hipMemcpy(deviceMatrixA, matrixA, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(deviceMatrixB, matrixB, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;

        hiptensorInitTensorDescriptor(&handle, &descA, 2, extentA, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;
        hiptensorInitTensorDescriptor(&handle, &descB, 2, extentB, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;
        hiptensorInitTensorDescriptor(&handle, &descC, 2, extentC, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;

        uint32_t alignmentRequirementA;
        uint32_t alignmentRequirementB;
        uint32_t alignmentRequirementC;
        cutensorGetAlignmentRequirement(&handle,
                                        deviceMatrixA,
                                        &descA,
                                        &alignmentRequirementA);
        CHECK_ERR;
        cutensorGetAlignmentRequirement(&handle,
                                        deviceMatrixB,
                                        &descB,
                                        &alignmentRequirementB);
        CHECK_ERR;
        cutensorGetAlignmentRequirement(&handle,
                                        deviceMatrixC,
                                        &descC,
                                        &alignmentRequirementC);
        CHECK_ERR;

        // cuTensor contraction
        cutensorContractionDescriptor_t desc;
        cutensorInitContractionDescriptor(&handle,
                                          &desc,
                                          &descA, modeA.data(), alignmentRequirementA,
                                          &descB, modeB.data(), alignmentRequirementB,
                                          &descC, modeC.data(), alignmentRequirementC,
                                          &descC, modeC.data(), alignmentRequirementC,
                                          HIPTENSOR_COMPUTE_DESC_32F);
        CHECK_ERR;

        cutensorContractionFind_t find;
        cutensorInitContractionFind(
            &handle, &find,
            HIPTENSOR_ALGO_DEFAULT);
        CHECK_ERR;

        size_t worksize = 0;
        cutensorContractionGetWorkspaceSize(&handle,
                                            &desc,
                                            &find,
                                            CUTENSOR_WORKSPACE_RECOMMENDED, &worksize);
        CHECK_ERR;
        // Allocate workspace
        void *work = nullptr;
        if (worksize > 0)
        {
            if (hipSuccess != hipMalloc(&work, worksize)) // This is optional!
            {
                work = nullptr;
                worksize = 0;
            }
        }

        hiptensorContractionPlan_t plan;
        cutensorInitContractionPlan(&handle,
                                    &plan,
                                    &desc,
                                    &find,
                                    worksize);
        CHECK_ERR;

        hiptensorStatus_t err;

        // Execute the tensor contraction
        err = hiptensorContraction(&handle,
                                  &plan,
                                  (void *)&alpha, deviceMatrixA,
                                  deviceMatrixB,
                                  (void *)&beta, deviceMatrixC,
                                  deviceMatrixC,
                                  work, worksize, 0 /* stream */);
        CHECK_ERR;

        hipDeviceSynchronize();
        CHECK_ERR;

        hipMemcpy(matrixC_cuTensor, deviceMatrixC, matrixSize, hipMemcpyDeviceToHost);
        CHECK_ERR;

        hipFree(work);
        CHECK_ERR;
        hipFree(deviceMatrixA);
        CHECK_ERR;
        hipFree(deviceMatrixB);
        CHECK_ERR;
        hipFree(deviceMatrixC);
        CHECK_ERR;

        bool resultsMatch = compareMatrices(matrixC_cuTensor, matrixC_GPU, numRows, numCols, tolerance);

        if (resultsMatch)
        {
            std::cout << "Results match! (CPU-cuTensor)" << std::endl;
        }
        else
        {
            std::cout << "Results do not match! (CPU-cuTensor)" << std::endl;
        }
    }

    std::cout << "Matrix C (CPU Result):" << std::endl;
    printMatrix(matrixC_CPU, numRows, numCols);
    std::cout << std::endl;
    std::cout << "Matrix C (GPU Result cuBLAS):" << std::endl;
    printMatrix(matrixC_GPU, numRows, numCols);
    std::cout << std::endl;
    std::cout << "Matrix C (GPU Result Offset-Gemm):" << std::endl;
    printMatrix(matrixC_GPU_MyGemm, numRows, numCols);
    std::cout << std::endl;
    std::cout << "Matrix C (GPU Result cuTensor):" << std::endl;
    printMatrix(matrixC_cuTensor, numRows, numCols);
    std::cout << std::endl;

    delete[] matrixA;
    delete[] matrixB;
    delete[] matrixC_CPU;
    delete[] matrixC_GPU;
    delete[] matrixC_cuTensor;
}