#include "hip/hip_runtime.h"
// Example Tensor Contraction
// Number 1, tensor Multiplication as a Tensor:
// C[ij] = A[ik] * B[kj]
// Number 2, 3D to 3D Tensors
// C[nko] = A[mko] * B[nmo]

#include <hipblas.h>
#include <iostream>
#include <random>
#include <iomanip>
#include <hiptensor.h>

#define CHECK_ERR checkErr(__FILE__, __LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line)
{
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess)
    {
        std::cout << std::endl
                  << File
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error)
                  << " (" << Error << ")"
                  << std::endl;

        if (PrevLine > 0)
            std::cout << "Previous CUDA call:" << std::endl
                      << PrevFile << ", line " << PrevLine << std::endl;
        throw;
    }
    PrevFile = File;
    PrevLine = Line;
#endif
}

bool compareMatrices(const float *tensorA, const float *tensorB, size_t numElements, float tolerance)
{
    for (int i = 0; i < numElements; i++)
    {
        float diff = std::fabs(tensorA[i] - tensorB[i]);
        if (diff > tolerance)
        {
            return false;
        }
    }

    return true;
}

void matrixMultiplyCPU(const float *matrixA, const float *matrixB, float *matrixC, int numRows, int numCols, int sharedDim)
{
    for (int col = 0; col < numCols; col++)
    {
        for (int row = 0; row < numRows; row++)
        {
            float sum = 0.0f;
            for (int k = 0; k < sharedDim; k++)
            {
                sum += matrixA[k * numRows + row] * matrixB[k + numRows * col];
            }
            matrixC[col * numRows + row] = sum;
        }
    }
}

// Print a tensor
void printTensor(const float *tensor, int numRows, int numCols, int numZ)
{
    std::cout << std::setprecision(4);
    for (int z = 0; z < numZ; z++)
    {
        for (int row = 0; row < numRows; row++)
        {
            for (int col = 0; col < numCols; col++)
            {
                std::cout << tensor[col * numRows + row] << "\t";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
}

__global__ void 
__launch_bounds__(96)
 kernel_sgemm_NT_NT_m8_n8_k8_lda8_ldb8_ldc8_alpha_1_beta_0_ppp_84fcd7e(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[0 /* + MatrixOffset + */ + A_extraOffset];
      const float * const __restrict__ glb_B = &B[0 /* + MatrixOffset + */ + B_extraOffset];
      float * const __restrict__ glb_C = &C[0 /* + MatrixOffset + */ + C_extraOffset];
      float reg0[8] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[192];
      float * localShrMem0 = &totalShrMem[64 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedPatchLoader
      {
        shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        shrRegion0[threadIdx.x + 32] = glb_B[threadIdx.x + 32];
      }
      __syncwarp();
      if (threadIdx.x < 8) {
        float value;

        #pragma unroll
        for (int k = 0; k < 8; ++k) {
          value = glb_A[threadIdx.x + k * 8];

          #pragma unroll
          for (int n = 0; n < 8; ++n) {
            reg0[n] += value * shrRegion0[k + 8 * n];
          }
        }
      }
      if (threadIdx.x < 8) {
        // store.py writing things
        #pragma unroll
        for (int n = 0; n < 8; ++n) {
          glb_C[threadIdx.x + 8 * n] = reg0[n]; //u
        }
      }
    }
  }
}
void sgemm_NT_NT_m8_n8_k8_lda8_ldb8_ldc8_alpha_1_beta_0_ppp_84fcd7e(const float *A, int A_extraOffset, const float *B, int B_extraOffset, float *C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(32, 3, 1);
  dim3 grid((numElements + 3 - 1) / 3, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_NT_NT_m8_n8_k8_lda8_ldb8_ldc8_alpha_1_beta_0_ppp_84fcd7e<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}



int main()
{
    const int64_t dims[3] = {8, 8, 8};
    const int64_t numElements = dims[0] * dims[1] * dims[2];
    const int64_t tensorSize = numElements * sizeof(float);
    const float tolerance = 1e-6; // Tolerance for floating-point comparison
    const int64_t numTensors = 1000;

    // Initialize the column-major matrices A, B, and C
    float **tensorA = new float*[numTensors];
    float **tensorB = new float*[numTensors];
    float **tensorC_CPU = new float*[numTensors];
    float **tensorC_cuTensor = new float*[numTensors];
    float **tensorC_LoG = new float*[numTensors];

    for (int i = 0; i<numTensors; i++){
        tensorA[i] = new float[numElements];
        tensorB[i] = new float[numElements];
        tensorC_CPU[i] = new float[numElements];
        tensorC_cuTensor[i] = new float[numElements];
        tensorC_LoG[i] = new float[numElements];
    }

    // Random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    // Initialize matrices A and B with random values
    for (int t = 0; t < numTensors; t++){
        for (int i = 0; i < numElements; i++)
        {
            tensorA[t][i] = dist(gen);
            tensorB[t][i] = dist(gen);
            tensorC_CPU[t][i] = 0.f;
            tensorC_cuTensor[t][i] = 0.f;
            tensorC_LoG[t][i] = 0.f;
        }
    }


    // Mat mul CPU
    for (int t = 0; t < numTensors; t++)
    {
        for (size_t z = 0; z < dims[2]; z++)
        {
            matrixMultiplyCPU(tensorA[t] + z * dims[0] * dims[1],
                              tensorB[t] + z * dims[0] * dims[1],
                              tensorC_CPU[t] + z * dims[0] * dims[1],
                              dims[0], dims[1], dims[0]);
        }
    }

    // Mat mul with my general Gemm Implementation
    {
        float **tmpDevicetensorA = new float*[numTensors];
        float **tmpDevicetensorB = new float*[numTensors];
        float **tmpDevicetensorC = new float*[numTensors];
        float **devicetensorA;
        float **devicetensorB;
        float **devicetensorC;

        //hipMalloc((void **)&devicetensorA, numTensors * sizeof(float*));
        //CHECK_ERR;
        //hipMalloc((void **)&devicetensorB, numTensors * sizeof(float*));
        //CHECK_ERR;
        //hipMalloc((void **)&devicetensorC, numTensors * sizeof(float*));
        //CHECK_ERR;

        for (int t = 0; t < numTensors; t++){
            hipMalloc((void **)&tmpDevicetensorA[t], tensorSize * sizeof(float));
            CHECK_ERR;
            hipMalloc((void **)&tmpDevicetensorB[t], tensorSize * sizeof(float));
            CHECK_ERR;
            hipMalloc((void **)&tmpDevicetensorC[t], tensorSize * sizeof(float));
            CHECK_ERR;
            hipMemcpy(tmpDevicetensorA[t], tensorA[t], tensorSize * sizeof(float), hipMemcpyHostToDevice);
            CHECK_ERR;
            hipMemcpy(tmpDevicetensorB[t], tensorB[t], tensorSize * sizeof(float), hipMemcpyHostToDevice);
            CHECK_ERR;
        }

        hipMalloc((void **)&devicetensorA, numTensors * sizeof(float*));
        CHECK_ERR;
        hipMalloc((void **)&devicetensorB, numTensors * sizeof(float*));
        CHECK_ERR;
        hipMalloc((void **)&devicetensorC, numTensors * sizeof(float*));
        CHECK_ERR;
        hipMemcpy(devicetensorA, tmpDevicetensorA, numTensors * sizeof(float*), hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(devicetensorB, tmpDevicetensorB, numTensors * sizeof(float*), hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(devicetensorC, tmpDevicetensorC, numTensors * sizeof(float*), hipMemcpyHostToDevice);
        CHECK_ERR;

        // Perform tensor multiplication C = A * B using cuBLAS
        //for (int t = 0; t < numTensors; ++t) {
            for (int _o = 0; _o < 8; ++_o) {
                float const* _A = tmpDevicetensorA[t] + 64*_o;
                float const* _B =  tmpDevicetensorB[t] + 64*_o;
                float * _C =  tmpDevicetensorC[t] + 64*_o;
                sgemm_NT_NT_m8_n8_k8_lda8_ldb8_ldc8_alpha_1_beta_0_ppp_84fcd7e(const_cast<const float *>(_A), 0, const_cast<const float *>(_B), 0, _C, 0, numElements, nullptr, nullptr);
            }
        //}
        hipDeviceSynchronize();

        for (int t = 0; t < numTensors; t++){
            // Copy the result tensor C from the GPU to the CPU
            hipMemcpy(tensorC_LoG[t], tmpDevicetensorC[t], tensorSize * sizeof(float), hipMemcpyDeviceToHost);
            CHECK_ERR;
            // Compare results with CPU tensor multiplication
            bool resultsMatch = compareMatrices(tensorC_CPU[t], tensorC_LoG[t], numElements, tolerance);
            if (resultsMatch)
            {
                //std::cout << "Results match! (CPU-cuBLAS)" << std::endl;
            }
            else
            {
                std::cout << "Results do not match! (CPU <-> LoG) at " << t << std::endl;
            }
        }


        // Clean up resources
        hipFree(devicetensorA);
        CHECK_ERR;
        hipFree(devicetensorB);
        CHECK_ERR;
        hipFree(devicetensorC);
        CHECK_ERR;
    }

    // tensor multiplication with cuTensor
    {
        // cuTensor initialization
        hiptensorHandle_t handle;
        cutensorInit(&handle);
        CHECK_ERR;

        // Create vector of modes
        std::vector<int> modeA{'i', 'k', 'n'};
        std::vector<int> modeB{'k', 'j', 'n'};
        std::vector<int> modeC{'i', 'j', 'n'};
        int nmodeA = modeA.size();
        int nmodeB = modeB.size();
        int nmodeC = modeC.size();

        // Tensor descriptors
        hiptensorTensorDescriptor_t descA, descB, descC;
        const int64_t *extentA = new int64_t[3]{dims[0], dims[1], dims[2]};
        const int64_t *extentB = new int64_t[3]{dims[0], dims[1], dims[2]};
        const int64_t *extentC = new int64_t[3]{dims[0], dims[1], dims[2]};

        // size_t elementsA = numRows * sharedDim;
        // size_t elementsB = sharedDim * numCols;
        // size_t elementsC = numRows * numCols;

        float *devicetensorA;
        float *devicetensorB;
        float *devicetensorC;

        hipMalloc((void **)&devicetensorA, tensorSize);
        CHECK_ERR;
        hipMalloc((void **)&devicetensorB, tensorSize);
        CHECK_ERR;
        hipMalloc((void **)&devicetensorC, tensorSize);
        CHECK_ERR;

        constexpr float alpha = 1.0f;
        constexpr float beta = 0.0f;

        // Copy matrices A and B from the CPU to the GPU
        hipMemcpy(devicetensorA, tensorA[0], tensorSize, hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(devicetensorB, tensorB[0], tensorSize, hipMemcpyHostToDevice);
        CHECK_ERR;

        hiptensorInitTensorDescriptor(&handle, &descA, 3, extentA, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;
        hiptensorInitTensorDescriptor(&handle, &descB, 3, extentB, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;
        hiptensorInitTensorDescriptor(&handle, &descC, 3, extentC, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;

        uint32_t alignmentRequirementA;
        uint32_t alignmentRequirementB;
        uint32_t alignmentRequirementC;
        cutensorGetAlignmentRequirement(&handle,
                                        devicetensorA,
                                        &descA,
                                        &alignmentRequirementA);
        CHECK_ERR;
        cutensorGetAlignmentRequirement(&handle,
                                        devicetensorB,
                                        &descB,
                                        &alignmentRequirementB);
        CHECK_ERR;
        cutensorGetAlignmentRequirement(&handle,
                                        devicetensorC,
                                        &descC,
                                        &alignmentRequirementC);
        CHECK_ERR;

        // cuTensor contraction
        cutensorContractionDescriptor_t desc;
        cutensorInitContractionDescriptor(&handle,
                                          &desc,
                                          &descA, modeA.data(), alignmentRequirementA,
                                          &descB, modeB.data(), alignmentRequirementB,
                                          &descC, modeC.data(), alignmentRequirementC,
                                          &descC, modeC.data(), alignmentRequirementC,
                                          HIPTENSOR_COMPUTE_DESC_32F);
        CHECK_ERR;

        cutensorContractionFind_t find;
        cutensorInitContractionFind(
            &handle, &find,
            HIPTENSOR_ALGO_DEFAULT);
        CHECK_ERR;

        size_t worksize = 0;
        cutensorContractionGetWorkspaceSize(&handle,
                                            &desc,
                                            &find,
                                            CUTENSOR_WORKSPACE_RECOMMENDED, &worksize);
        CHECK_ERR;
        // Allocate workspace
        void *work = nullptr;
        if (worksize > 0)
        {
            if (hipSuccess != hipMalloc(&work, worksize)) // This is optional!
            {
                work = nullptr;
                worksize = 0;
            }
        }

        hiptensorContractionPlan_t plan;
        cutensorInitContractionPlan(&handle,
                                    &plan,
                                    &desc,
                                    &find,
                                    worksize);
        CHECK_ERR;

        hiptensorStatus_t err;

        // Execute the tensor contraction
        err = hiptensorContraction(&handle,
                                  &plan,
                                  (void *)&alpha, devicetensorA,
                                  devicetensorB,
                                  (void *)&beta, devicetensorC,
                                  devicetensorC,
                                  work, worksize, 0 /* stream */);
        CHECK_ERR;

        hipDeviceSynchronize();
        CHECK_ERR;

        hipMemcpy(tensorC_cuTensor, devicetensorC, tensorSize, hipMemcpyDeviceToHost);
        CHECK_ERR;

        hipFree(work);
        CHECK_ERR;
        hipFree(devicetensorA);
        CHECK_ERR;
        hipFree(devicetensorB);
        CHECK_ERR;
        hipFree(devicetensorC);
        CHECK_ERR;

        /*
        bool resultsMatch = compareMatrices(tensorC_cuTensor[0], tensorC_CPU[0], numElements, tolerance);

        if (resultsMatch)
        {
            std::cout << "Results match! (CPU <-> cuTensor)" << std::endl;
        }
        else
        {
            std::cout << "Results do not match! (CPU <-> cuTensor)" << std::endl;
        }
        */
    }

    std::cout << "tensor C (CPU Result):" << std::endl;
    printTensor(tensorC_CPU[0], dims[0], dims[1], dims[2]);
    std::cout << std::endl;
    std::cout << "tensor C (GPU Result cuTensor):" << std::endl;
    printTensor(tensorC_cuTensor[0], dims[0], dims[1], dims[2]);
    std::cout << std::endl;
    std::cout << "tensor C (GPU Result LoG):" << std::endl;
    printTensor(tensorC_LoG[0], dims[0], dims[1], dims[2]);
    std::cout << std::endl;

    delete[] tensorA;
    delete[] tensorB;
    delete[] tensorC_CPU;
    delete[] tensorC_cuTensor;
}