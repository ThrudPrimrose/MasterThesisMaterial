#include "hip/hip_runtime.h"
// Example Tensor Contraction
// Number 1, Matrix Multiplication as a Tensor:
// C[ij] = A[ik] * B[kj]
// Number 2, 3D to 3D Tensors
// C[nko] = A[mko] * B[nmo]

#include <hipblas.h>
#include <iostream>
#include <random>
#include <iomanip>
#include <hiptensor.h>

#define CHECK_ERR checkErr(__FILE__, __LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line)
{
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess)
    {
        std::cout << std::endl
                  << File
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error)
                  << " (" << Error << ")"
                  << std::endl;

        if (PrevLine > 0)
            std::cout << "Previous CUDA call:" << std::endl
                      << PrevFile << ", line " << PrevLine << std::endl;
        throw;
    }
    PrevFile = File;
    PrevLine = Line;
#endif
}

bool compareMatrices(const float *tensorA, const float *tensorB, size_t numElements, float tolerance)
{
    for (int i = 0; i < numElements; i++)
    {
        float diff = std::fabs(tensorA[i] - tensorB[i]);
        if (diff > tolerance)
        {
            return false;
        }
    }

    return true;
}

void matrixMultiplyCPU(const float *matrixA, const float *matrixB, float *matrixC, int numRows, int numCols, int sharedDim)
{
    for (int col = 0; col < numCols; col++)
    {
        for (int row = 0; row < numRows; row++)
        {
            float sum = 0.0f;
            for (int k = 0; k < sharedDim; k++)
            {
                sum += matrixA[k * numRows + row] * matrixB[k + numRows * col];
            }
            matrixC[col * numRows + row] = sum;
        }
    }
}

// Print a matrix
void printTensor(const float *matrix, int numRows, int numCols, int numZ)
{
    std::cout << std::setprecision(4);
    for (int z = 0; z < numZ; z++)
    {
        for (int row = 0; row < numRows; row++)
        {
            for (int col = 0; col < numCols; col++)
            {
                std::cout << matrix[col * numRows + row] << "\t";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
}

__global__ void
    __launch_bounds__(32)
        gemm(const float *A, const int offsetBetweenElementsAx, const int offsetBetweenElementsAy,
             const float *B, const int offsetBetweenElementsBx, const int offsetBetweenElementsBy,
             float *C, const int offsetBetweenElementsCx, const int offsetBetweenElementsCy,
             unsigned numElements, unsigned *flags)
{
    unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
    if (batchID < numElements)
    {
        bool isFlagsProvided = (flags != nullptr);
        bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
        if (allowed)
        {
            const float *const __restrict__ glb_A = &A[batchID * 64 + 0];
            const float *const __restrict__ glb_B = &B[batchID * 64 + 0];
            float *const __restrict__ glb_C = &C[batchID * 64 + 0];
            float reg0[8] = {0.0f};
            __shared__ __align__(8) float totalShrMem[64];
            float *localShrMem0 = &totalShrMem[64 * threadIdx.y];

            float *shrRegion0 = &localShrMem0[0];
            /*
                assertions that offsets are either matches 1 or a combination of dimensions like dimA or dimB or dimC*dimB etc.
            */
            // using ExtendedPatchLoader
            if (threadIdx.x < 8)
            {
#pragma unroll
                for (int i = 0; i < 8; ++i)
                {
                    shrRegion0[threadIdx.x + i * 8] = glb_B[(threadIdx.x * offsetBetweenElementsBy) + (i * offsetBetweenElementsBx)];
                }
            }
            __syncwarp();
            if (threadIdx.x < 8)
            {
                float value;

#pragma unroll
                for (int k = 0; k < 8; ++k)
                {
                    value = glb_A[(threadIdx.x * offsetBetweenElementsAy) + (k * offsetBetweenElementsAx)];

#pragma unroll
                    for (int n = 0; n < 8; ++n)
                    {
                        reg0[n] += value * shrRegion0[k + 8 * n];
                    }
                }
            }
            if (threadIdx.x < 8)
            {
#pragma unroll
                for (int n = 0; n < 8; ++n)
                {
                    glb_C[(threadIdx.x * offsetBetweenElementsCy) + (n * offsetBetweenElementsCx)] = reg0[n];
                }
            }
        }
    }
}

void gemm_launcher(const float *A, const int offsetBetweenElementsAx, const int offsetBetweenElementsAy,
                   const float *B, const int offsetBetweenElementsBx, const int offsetBetweenElementsBy,
                   float *C, const int offsetBetweenElementsCx, const int offsetBetweenElementsCy,
                   const int64_t *dims, const int64_t iterDimOffset, const int64_t numDims,
                   unsigned numElements, unsigned *flags, void *streamPtr)
{
    dim3 block(32, 1, 1);
    dim3 grid((numElements + 1 - 1) / 1, 1, 1);
    // hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
    int offset = 1;
    for (int i = 0; i < iterDimOffset; i++)
    {
        offset *= dims[i];
    }
    for (int iter = 0; iter < dims[iterDimOffset]; iter++)
    {
        hipStream_t stream;
        hipStreamCreate(&stream);
        CHECK_ERR;
        gemm<<<grid, block, 0, stream>>>(
            A + iter * offset, offsetBetweenElementsAx, offsetBetweenElementsAy,
            B + iter * offset, offsetBetweenElementsBx, offsetBetweenElementsBy,
            C + iter * offset, offsetBetweenElementsCx, offsetBetweenElementsCy,
            numElements, flags);
        CHECK_ERR;
    }
    CHECK_ERR;
}

int main()
{
    const int64_t dims[3] = {8, 8, 8};
    const int64_t numElements = dims[0] * dims[1] * dims[2];
    const int64_t matrixSize = numElements * sizeof(float);
    const float tolerance = 1e-6; // Tolerance for floating-point comparison

    // Initialize the column-major matrices A, B, and C
    float *matrixA = new float[numElements];
    float *matrixB = new float[numElements];
    float *matrixC_CPU = new float[numElements];
    float *matrixC_GPU = new float[numElements];
    float *matrixC_cuTensor = new float[numElements];
    float *matrixC_LoG = new float[numElements];

    // Random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    // Initialize matrices A and B with random values
    for (int i = 0; i < numElements; i++)
    {
        matrixA[i] = dist(gen);
        matrixB[i] = dist(gen);
        matrixC_CPU[i] = 0.f;
        matrixC_GPU[i] = 0.f;
        matrixC_cuTensor[i] = 0.f;
        matrixC_LoG[i] = 0.f;
    }

    // Mat mul CPU
    {
        for (size_t z = 0; z < dims[2]; z++)
        {
            matrixMultiplyCPU(matrixA + z * dims[0] * dims[1],
                              matrixB + z * dims[0] * dims[1],
                              matrixC_CPU + z * dims[0] * dims[1],
                              dims[0], dims[1], dims[0]);
        }
    }

    // Mat mul GPU with cuBLAS
    {
        // Transpose matrices A, B, and C to row-major format
        float *deviceMatrixA;
        float *deviceMatrixB;
        float *deviceMatrixC;

        hipMalloc((void **)&deviceMatrixA, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixB, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixC, matrixSize);
        CHECK_ERR;

        // Multiply matrices A, B, and C using cuBLAS
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        constexpr float alpha = 1.0f;
        constexpr float beta = 0.0f;

        // Copy matrices A and B from the CPU to the GPU
        hipMemcpy(deviceMatrixA, matrixA, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(deviceMatrixB, matrixB, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;

        // Perform matrix multiplication C = A * B using cuBLAS
        for (size_t z = 0; z < dims[2]; z++)
        {
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dims[0], dims[1], dims[0],
                        &alpha, deviceMatrixA + z * dims[0] * dims[1], dims[0],
                        deviceMatrixB + z * dims[0] * dims[1], dims[0],
                        &beta, deviceMatrixC + z * dims[0] * dims[1], dims[0]);
        }
        CHECK_ERR;
        hipDeviceSynchronize();
        CHECK_ERR;

        // Copy the result matrix C from the GPU to the CPU
        hipMemcpy(matrixC_GPU, deviceMatrixC, matrixSize, hipMemcpyDeviceToHost);
        CHECK_ERR;

        // Compare results with CPU matrix multiplication

        bool resultsMatch = compareMatrices(matrixC_CPU, matrixC_GPU, numElements, tolerance);

        if (resultsMatch)
        {
            std::cout << "Results match! (CPU-cuBLAS)" << std::endl;
        }
        else
        {
            std::cout << "Results do not match! (CPU-cuBLAS)" << std::endl;
        }

        // Clean up resources
        hipFree(deviceMatrixA);
        CHECK_ERR;
        hipFree(deviceMatrixB);
        CHECK_ERR;
        hipFree(deviceMatrixC);
        CHECK_ERR;

        hipblasDestroy(handle);
    }

    // Mat mul with my general Gemm Implementation
    {
        // Transpose matrices A, B, and C to row-major format
        float *deviceMatrixA;
        float *deviceMatrixB;
        float *deviceMatrixC;

        hipMalloc((void **)&deviceMatrixA, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixB, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixC, matrixSize);
        CHECK_ERR;

        // Multiply matrices A, B, and C using cuBLAS
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        constexpr float alpha = 1.0f;
        constexpr float beta = 0.0f;

        // Copy matrices A and B from the CPU to the GPU
        hipMemcpy(deviceMatrixA, matrixA, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(deviceMatrixB, matrixB, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;

        // Perform matrix multiplication C = A * B using cuBLAS
        gemm_launcher(deviceMatrixA, dims[0], 1,
                      deviceMatrixB, dims[0], 1,
                      deviceMatrixC, dims[0], 1,
                      &dims[0], 2, 3, 1, nullptr, nullptr);
        CHECK_ERR;
        hipDeviceSynchronize();
        CHECK_ERR;

        // Copy the result matrix C from the GPU to the CPU
        hipMemcpy(matrixC_LoG, deviceMatrixC, matrixSize, hipMemcpyDeviceToHost);
        CHECK_ERR;

        // Compare results with CPU matrix multiplication

        bool resultsMatch = compareMatrices(matrixC_CPU, matrixC_LoG, numElements, tolerance);

        if (resultsMatch)
        {
            std::cout << "Results match! (CPU-OffsetGemm)" << std::endl;
        }
        else
        {
            std::cout << "Results do not match! (CPU-OffsetGemm)" << std::endl;
        }

        // Clean up resources
        hipFree(deviceMatrixA);
        CHECK_ERR;
        hipFree(deviceMatrixB);
        CHECK_ERR;
        hipFree(deviceMatrixC);
        CHECK_ERR;

        hipblasDestroy(handle);
    }

    // Matrix multiplication with cuTensor
    {
        // cuTensor initialization
        hiptensorHandle_t handle;
        cutensorInit(&handle);
        CHECK_ERR;

        // Create vector of modes
        std::vector<int> modeA{'i', 'k', 'n'};
        std::vector<int> modeB{'k', 'j', 'n'};
        std::vector<int> modeC{'i', 'j', 'n'};
        int nmodeA = modeA.size();
        int nmodeB = modeB.size();
        int nmodeC = modeC.size();

        // Tensor descriptors
        hiptensorTensorDescriptor_t descA, descB, descC;
        const int64_t *extentA = new int64_t[3]{dims[0], dims[1], dims[2]};
        const int64_t *extentB = new int64_t[3]{dims[0], dims[1], dims[2]};
        const int64_t *extentC = new int64_t[3]{dims[0], dims[1], dims[2]};

        // size_t elementsA = numRows * sharedDim;
        // size_t elementsB = sharedDim * numCols;
        // size_t elementsC = numRows * numCols;

        float *deviceMatrixA;
        float *deviceMatrixB;
        float *deviceMatrixC;

        hipMalloc((void **)&deviceMatrixA, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixB, matrixSize);
        CHECK_ERR;
        hipMalloc((void **)&deviceMatrixC, matrixSize);
        CHECK_ERR;

        constexpr float alpha = 1.0f;
        constexpr float beta = 0.0f;

        // Copy matrices A and B from the CPU to the GPU
        hipMemcpy(deviceMatrixA, matrixA, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;
        hipMemcpy(deviceMatrixB, matrixB, matrixSize, hipMemcpyHostToDevice);
        CHECK_ERR;

        hiptensorInitTensorDescriptor(&handle, &descA, 3, extentA, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;
        hiptensorInitTensorDescriptor(&handle, &descB, 3, extentB, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;
        hiptensorInitTensorDescriptor(&handle, &descC, 3, extentC, NULL, HIP_R_32F, HIPTENSOR_OP_IDENTITY);
        CHECK_ERR;

        uint32_t alignmentRequirementA;
        uint32_t alignmentRequirementB;
        uint32_t alignmentRequirementC;
        cutensorGetAlignmentRequirement(&handle,
                                        deviceMatrixA,
                                        &descA,
                                        &alignmentRequirementA);
        CHECK_ERR;
        cutensorGetAlignmentRequirement(&handle,
                                        deviceMatrixB,
                                        &descB,
                                        &alignmentRequirementB);
        CHECK_ERR;
        cutensorGetAlignmentRequirement(&handle,
                                        deviceMatrixC,
                                        &descC,
                                        &alignmentRequirementC);
        CHECK_ERR;

        // cuTensor contraction
        cutensorContractionDescriptor_t desc;
        cutensorInitContractionDescriptor(&handle,
                                          &desc,
                                          &descA, modeA.data(), alignmentRequirementA,
                                          &descB, modeB.data(), alignmentRequirementB,
                                          &descC, modeC.data(), alignmentRequirementC,
                                          &descC, modeC.data(), alignmentRequirementC,
                                          HIPTENSOR_COMPUTE_DESC_32F);
        CHECK_ERR;

        cutensorContractionFind_t find;
        cutensorInitContractionFind(
            &handle, &find,
            HIPTENSOR_ALGO_DEFAULT);
        CHECK_ERR;

        size_t worksize = 0;
        cutensorContractionGetWorkspaceSize(&handle,
                                            &desc,
                                            &find,
                                            CUTENSOR_WORKSPACE_RECOMMENDED, &worksize);
        CHECK_ERR;
        // Allocate workspace
        void *work = nullptr;
        if (worksize > 0)
        {
            if (hipSuccess != hipMalloc(&work, worksize)) // This is optional!
            {
                work = nullptr;
                worksize = 0;
            }
        }

        hiptensorContractionPlan_t plan;
        cutensorInitContractionPlan(&handle,
                                    &plan,
                                    &desc,
                                    &find,
                                    worksize);
        CHECK_ERR;

        hiptensorStatus_t err;

        // Execute the tensor contraction
        err = hiptensorContraction(&handle,
                                  &plan,
                                  (void *)&alpha, deviceMatrixA,
                                  deviceMatrixB,
                                  (void *)&beta, deviceMatrixC,
                                  deviceMatrixC,
                                  work, worksize, 0 /* stream */);
        CHECK_ERR;

        hipDeviceSynchronize();
        CHECK_ERR;

        hipMemcpy(matrixC_cuTensor, deviceMatrixC, matrixSize, hipMemcpyDeviceToHost);
        CHECK_ERR;

        hipFree(work);
        CHECK_ERR;
        hipFree(deviceMatrixA);
        CHECK_ERR;
        hipFree(deviceMatrixB);
        CHECK_ERR;
        hipFree(deviceMatrixC);
        CHECK_ERR;

        bool resultsMatch = compareMatrices(matrixC_cuTensor, matrixC_GPU, numElements, tolerance);

        if (resultsMatch)
        {
            std::cout << "Results match! (CPU-cuTensor)" << std::endl;
        }
        else
        {
            std::cout << "Results do not match! (CPU-cuTensor)" << std::endl;
        }
    }

    std::cout << "Matrix C (CPU Result):" << std::endl;
    printTensor(matrixC_CPU, dims[0], dims[1], dims[2]);
    std::cout << std::endl;
    std::cout << "Matrix C (GPU Result cuBLAS):" << std::endl;
    printTensor(matrixC_GPU, dims[0], dims[1], dims[2]);
    std::cout << std::endl;
    std::cout << "Matrix C (GPU Result cuTensor):" << std::endl;
    printTensor(matrixC_cuTensor, dims[0], dims[1], dims[2]);
    std::cout << std::endl;
    std::cout << "Matrix C (GPU Result LoG):" << std::endl;
    printTensor(matrixC_LoG, dims[0], dims[1], dims[2]);
    std::cout << std::endl;

    delete[] matrixA;
    delete[] matrixB;
    delete[] matrixC_CPU;
    delete[] matrixC_GPU;
    delete[] matrixC_cuTensor;
}