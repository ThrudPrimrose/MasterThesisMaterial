#include <iostream>
#include <hip/hip_runtime.h>

constexpr size_t N = static_cast<size_t>(1e9 / static_cast<float>(sizeof(float)));

#define CHECK_ERR checkErr(__FILE__,__LINE__)

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {{
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess) {{
        std::cout << std::endl << File
                << ", line " << Line
                << ": " << hipGetErrorString(Error)
                << " (" << Error << ")"
                << std::endl;

        if (PrevLine > 0)
        std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
        throw;
    }}
    PrevFile = File;
    PrevLine = Line;
#endif
}}

__global__ void vectorAdd(float* a, float* b, float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    // Declare and initialize host vectors
    float* host_a = new float[N];
    float* host_b = new float[N];
    float* host_c = new float[N];
    for (int i = 0; i < N; ++i)
    {
        host_a[i] = i;
        host_b[i] = 2 * i;
    }

    // Declare and allocate device vectors
    float* dev_a, * dev_b, * dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(float)); CHECK_ERR;
    hipMalloc((void**)&dev_b, N * sizeof(float)); CHECK_ERR;
    hipMalloc((void**)&dev_c, N * sizeof(float)); CHECK_ERR;

    // Copy host vectors to device
    hipMemcpy(dev_a, host_a, N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
    hipMemcpy(dev_b, host_b, N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;

    // Define kernel launch configuration
    int blockSize, gridSize;
    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, vectorAdd, 0, N); CHECK_ERR;
    // int blockSize = 256;
    gridSize = (N + blockSize - 1) / blockSize;

    // Fire first kernel and discard
    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c); CHECK_ERR;
    hipDeviceSynchronize();

    // Start timer
    hipEvent_t start, stop;
    hipEventCreate(&start); CHECK_ERR;
    hipEventCreate(&stop); CHECK_ERR;
    hipEventRecord(start); CHECK_ERR;

    // Launch kernel
    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c); CHECK_ERR;

    // Stop timer and calculate execution duration
    hipEventRecord(stop); CHECK_ERR;
    hipEventSynchronize(stop); CHECK_ERR;
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); CHECK_ERR;

    // Copy result from device to host
    hipMemcpy(host_c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost); CHECK_ERR;
    hipDeviceSynchronize(); CHECK_ERR;

    for (int i = 0; i < N; ++i)
    {
        if (host_c[i] > 1.001f * (3.0f * static_cast<float>(i)) ||
            host_c[i] < 0.999f * (3.0f * static_cast<float>(i))){
            throw std::runtime_error("Results different from expected " + std::to_string(host_c[i]) + " != " + std::to_string(3.0f * static_cast<float>(i)));
        }
    }

    // Print execution duration
    std::cout << "Kernel execution duration: " << milliseconds << " ms" << std::endl;

    size_t numFloatingPointOps = N;
    size_t numBytesAccessed = 3 * N * sizeof(float);
    float opsPerByte = static_cast<float>(numFloatingPointOps) / static_cast<float>(numBytesAccessed);

    std::cout << "Floating-point operations per byte: " << opsPerByte << std::endl;

    float executionTimeSeconds = milliseconds / 1e3;
    float numGFLOPs = static_cast<float>(numFloatingPointOps) / 1e9;
    float GFLOPs = numGFLOPs / executionTimeSeconds;

    std::cout << "GFLOP/s: " << GFLOPs << std::endl;

    float peakMemoryBandwidthTheo = 176.032; // GB /s
    float peakGFLOPTheo  = 4329.47; // GFlop /s
    float peakGFLOPforIntensity = std::min(peakMemoryBandwidthTheo * opsPerByte, peakGFLOPTheo);

    float achievedPeak = (static_cast<float>(GFLOPs) / peakGFLOPforIntensity) * 100.0f;
    std::string strAchievedPeak(6, '\0');
    std::sprintf(&strAchievedPeak[0], "%.2f", achievedPeak);
    std::cout << "Percentage of Peak Performance: " << strAchievedPeak << "%" << std::endl;

    float GBPerSecond = (static_cast<float>(numBytesAccessed) * 1e-9) / executionTimeSeconds;
    std::cout << "GB per Second: " << GBPerSecond << std::endl;

    // Cleanup
    hipFree(dev_a); CHECK_ERR;
    hipFree(dev_b); CHECK_ERR;
    hipFree(dev_c); CHECK_ERR;
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;

    return 0;
}
