#include <iostream>
#include <hip/hip_runtime.h>

#define N 200000

__global__ void vectorAdd(float* a, float* b, float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    // Declare and initialize host vectors
    float* host_a = new float[N];
    float* host_b = new float[N];
    float* host_c = new float[N];
    for (int i = 0; i < N; ++i)
    {
        host_a[i] = i;
        host_b[i] = 2 * i;
    }

    // Declare and allocate device vectors
    float* dev_a, * dev_b, * dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, N * sizeof(float));

    // Copy host vectors to device
    hipMemcpy(dev_a, host_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Define kernel launch configuration
    int blockSize, gridSize;
    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, vectorAdd, 0, N);

    // Start timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c);

    // Stop timer and calculate execution duration
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result from device to host
    hipMemcpy(host_c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Print execution duration
    std::cout << "Kernel execution duration: " << milliseconds << " ms" << std::endl;

    int numFloatingPointOps = N;
    int numBytesAccessed = 3 * N * sizeof(float);
    float opsPerByte = static_cast<float>(numFloatingPointOps) / static_cast<float>(numBytesAccessed);

    std::cout << "Floating-point operations per byte: " << opsPerByte << std::endl;

    float executionTimeSeconds = milliseconds / 1e3;
    float numGFLOPs = static_cast<float>(numFloatingPointOps) / 1e9;
    float GFLOPs = numGFLOPs / executionTimeSeconds;

    std::cout << "GFLOP/s: " << GFLOPs << std::endl;

    // Cleanup
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;

    return 0;
}